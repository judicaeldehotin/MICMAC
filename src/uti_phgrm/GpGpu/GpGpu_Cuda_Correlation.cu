#include "hip/hip_runtime.h"
#include "GpGpu/GpGpu_ParamCorrelation.cuh"
#include "GpGpu/GpGpu_TextureTools.cuh"
#include "GpGpu/GpGpu_TextureCorrelation.cuh"
#include "GpGpu/SData2Correl.h"


/// \file       GpGpuCudaCorrelation.cu
/// \brief      Kernel
/// \author     GC
/// \version    0.2
/// \date       mars 2013

static __constant__ invParamCorrel  invPc;

extern "C" void CopyParamInvTodevice( pCorGpu param )
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(invPc), &param.invPC, sizeof(invParamCorrel)));
}


/// \fn template<int TexSel> __global__ void correlationKernel( uint *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
/// \brief Kernel fonction GpGpu Cuda
/// Calcul les vignettes de correlation pour toutes les images
///
template<int TexSel> __global__ void correlationKernel( uint *dev_NbImgOk, float* cachVig, uint2 nbActThrd,HDParamCorrel HdPc)
{

  extern __shared__ float cacheImg[];

  // Coordonn�es du terrain global avec bordure // __umul24!!!! A voir

  const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);

  // Si le processus est hors du terrain, nous sortons du kernel

  if (oSE(ptHTer,HdPc.dimDTer)) return;

  const float2 ptProj = GetProjection<TexSel>(ptHTer,invPc.sampProj,blockIdx.z);

  uint pitZ,modZ;

  if (oI(ptProj,0))

      return;

  else
  {
      pitZ  = blockIdx.z / invPc.nbImages;

      modZ  = blockIdx.z - pitZ * invPc.nbImages;

      cacheImg[threadIdx.y*BLOCKDIM + threadIdx.x] = GetImageValue(ptProj,modZ);
  }

  __syncthreads();

  const int2 ptTer = make_int2(ptHTer) - make_int2(invPc.rayVig);

  // Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel

  // Simplifier!!!
  if (oSE(threadIdx, nbActThrd + invPc.rayVig) || oI(threadIdx , invPc.rayVig) || oSE( ptTer, HdPc.dimTer) || oI(ptTer, 0))
    return;

  if(tex2D(TexS_MaskGlobal, ptTer.x + HdPc.rTer.pt0.x , ptTer.y + HdPc.rTer.pt0.y) == 0) return;

  const short2 c0	= make_short2(threadIdx) - invPc.rayVig;
  const short2 c1	= make_short2(threadIdx) + invPc.rayVig;

  // Intialisation des valeurs de calcul
  float aSV = 0.0f, aSVV = 0.0f;
  short2 pt;

  #pragma unroll // ATTENTION PRAGMA FAIT AUGMENTER LA quantit� MEMOIRE des registres!!!
  for (pt.y = c0.y ; pt.y <= c1.y; pt.y++)
  {
        const int pic = pt.y*BLOCKDIM;
      #pragma unroll
      for (pt.x = c0.x ; pt.x <= c1.x; pt.x++)
      {
          const float val = cacheImg[pic+ pt.x];	// Valeur de l'image
          //        if (val ==  cH.floatDefault) return;
          aSV  += val;          // Somme des valeurs de l'image cte
          aSVV += (val*val);	// Somme des carr�s des vals image cte
      }
  }

  aSV   = fdividef(aSV,(float)invPc.sizeVig );

  aSVV  = fdividef(aSVV,(float)invPc.sizeVig );

  aSVV -=	(aSV * aSV);

  if ( aSVV <= invPc.mAhEpsilon) return;

  aSVV =	rsqrtf(aSVV); // racine carre inverse

  const uint pitchCache = blockIdx.z * HdPc.sizeCach + ptTer.x * invPc.dimVig.x;

  const uint pitchCachY = ptTer.y * invPc.dimVig.y ;

#pragma unroll
  for ( pt.y = c0.y ; pt.y <= c1.y; pt.y++)
    {
      float* cImg = cacheImg + pt.y*BLOCKDIM;
      float* cVig = cachVig + pitchCache + (pitchCachY + (pt.y - c0.y))* HdPc.dimCach.x - c0.x;
#pragma unroll
      for ( pt.x = c0.x ; pt.x <= c1.x; pt.x++)
        cVig[ pt.x ] = (cImg[pt.x] -aSV)*aSVV;

    }

  const int ZPitch	= pitZ * HdPc.sizeTer;

  const int idN		= ZPitch + to1D(ptTer,HdPc.dimTer);

  atomicAdd( &dev_NbImgOk[idN], 1U);

}

/// \brief Fonction qui lance les kernels de correlation
extern "C" void	 LaunchKernelCorrelation(const int s,hipStream_t stream,pCorGpu &param,SData2Correl &data2cor)
{

    dim3	threads( BLOCKDIM, BLOCKDIM, 1);
    uint2	thd2D		= make_uint2(threads);
    uint2	nbActThrd	= thd2D - 2 * param.invPC.rayVig;
    uint2	block2D		= iDivUp(param.HdPc.dimDTer,nbActThrd);
    dim3	blocks(block2D.x , block2D.y, param.invPC.nbImages * param.ZCInter);

  switch (s)
    {
    case 0:
      correlationKernel<0><<<blocks, threads, BLOCKDIM * BLOCKDIM * sizeof(float), stream>>>( data2cor.DeviVolumeNOK(0), data2cor.DeviVolumeCache(0), nbActThrd,param.HdPc);
      getLastCudaError("Basic Correlation kernel failed stream 0");
      break;
    case 1:
      correlationKernel<1><<<blocks, threads, BLOCKDIM * BLOCKDIM* sizeof(float), stream>>>( data2cor.DeviVolumeNOK(1), data2cor.DeviVolumeCache(1), nbActThrd,param.HdPc);
      getLastCudaError("Basic Correlation kernel failed stream 1");
      break;
    }
}

/// \brief Kernel Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens n utilisant pas des fonctions atomiques
__global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, uint* dev_NbImgOk, uint2 nbActThr,HDParamCorrel HdPc)
{

  __shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];          // Somme des valeurs
  __shared__ float aSVV[ SBLOCKDIM  ][ SBLOCKDIM ];         // Somme des carr�s des valeurs
  __shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];		// resultat
  __shared__ ushort nbIm[ SBLOCKDIM/2][ SBLOCKDIM/2 ];		// nombre d'images correcte

  // coordonn�es des threads
  const uint2 t = make_uint2(threadIdx);

  aSV [t.y][t.x]        = 0.0f;

  aSVV[t.y][t.x]        = 0.0f;

  resu[t.y/2][t.x/2]	= 0.0f;

  nbIm[t.y/2][t.x/2]	= 0;

  if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

  // Coordonn�es 2D du cache vignette
  const uint2 ptCach = make_uint2(blockIdx) * nbActThr + t;

  // Si le thread est en dehors du cache
  if ( oSE(ptCach, HdPc.dimCach))	return;

  const uint2	ptTer	= ptCach / invPc.dimVig; // Coordonn�es 2D du terrain

  if(!tex2D(TexS_MaskGlobal, ptTer.x + HdPc.rTer.pt0.x , ptTer.y + HdPc.rTer.pt0.y)) return;

  const uint	iTer	= blockIdx.z * HdPc.sizeTer + to1D(ptTer, HdPc.dimTer);     // Coordonn�es 1D dans le terrain avec prise en compte des differents Z

  const uint2   thTer	= t / invPc.dimVig;                                        // Coordonn�es 2D du terrain dans le repere des threads

  //if(aEq(t,thTer * cH.dimVig))
  nbIm[thTer.y][thTer.x] = (ushort)dev_NbImgOk[iTer];

  //__syncthreads();

  if ( nbIm[thTer.y][thTer.x]  < 2) return;

  const uint pitLayerCache  = blockIdx.z * HdPc.sizeCachAll + to1D( ptCach, HdPc.dimCach );	// Taille du cache vignette pour une image
  //const uint pit  = blockIdx.z * cH.nbImages;

 #pragma unroll
  for(uint i = 0;i< HdPc.sizeCachAll;i+=HdPc.sizeCach)
  //for(uint l = pit ;l< pit + cH.nbImages;l++)
    {
      const float val  = cacheVign[pitLayerCache  + i];
      //const float val  = tex2DLayered( TexL_Cache,ptCach.x , ptCach.y,l);

      //if(val!= cH.floatDefault) A verifier si pas d'influence
        //{
          // Coordonn�es 1D du cache vignette

          aSV[t.y][t.x]   += val;
          aSVV[t.y][t.x]  += val * val;
        //}
    }

  __syncthreads();

  atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - fdividef(aSV[t.y][t.x] * aSV[t.y][t.x],(float)nbIm[thTer.y][thTer.x]));

  if (!aEq(t - thTer*invPc.dimVig,0)) return;

  __syncthreads();

  // Normalisation pour le ramener a un equivalent de 1-Correl
  const float cost = fdividef( resu[thTer.y][thTer.x], (float)( nbIm[thTer.y][thTer.x] -1.0f) * (invPc.sizeVig));

  dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));

}


/// \brief Fonction qui lance les kernels de multi-Correlation n'utilisant pas des fonctions atomiques
extern "C" void LaunchKernelMultiCorrelation(hipStream_t stream, pCorGpu &param, SData2Correl &dataCorrel)
{

    //-------------	calcul de dimension du kernel de multi-correlation NON ATOMIC ------------
    uint2	nbActThr	= SBLOCKDIM - make_uint2( SBLOCKDIM % param.invPC.dimVig.x, SBLOCKDIM % param.invPC.dimVig.y);
    dim3	threads(SBLOCKDIM, SBLOCKDIM, 1);
    uint2	block2D	= iDivUp(param.HdPc.dimCach,nbActThr);
    dim3	blocks(block2D.x,block2D.y,param.ZCInter);

    multiCorrelationKernel<<<blocks, threads, 0, stream>>>(dataCorrel.DeviVolumeCost(0), dataCorrel.DeviVolumeCache(0), dataCorrel.DeviVolumeNOK(0), nbActThr,param.HdPc);
    getLastCudaError("Multi-Correlation NON ATOMIC kernel failed");

}
