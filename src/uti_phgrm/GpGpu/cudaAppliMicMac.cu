#include "hip/hip_runtime.h"
#include "GpGpu/cudaAppliMicMac.cuh"
#include "GpGpu/cudaTextureTools.cuh"
#include "GpGpu/CudaRefTexture.cuh"


/// \file       cudaAppliMicMac.cu
/// \brief      Kernel
/// \author     GC
/// \version    0.1
/// \date       mars 2013


static __constant__ pCorGpu cH;

/// \fn extern "C" void CopyParamTodevice( pCorGpu param )
/// \brief copie les paramteres du host vers le device
extern "C" void CopyParamTodevice( pCorGpu param )
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cH), &param, sizeof(pCorGpu)));
}

/// \fn template<int TexSel> __global__ void correlationKernel( uint *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
/// \brief Kernel fonction GpGpu Cuda
/// Calcul les vignettes de correlation pour toutes les images
template<int TexSel> __global__ void correlationKernel( uint *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
{
  __shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];


  // Coordonn�es du terrain global avec bordure // __umul24!!!! A voir
  const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);

  // Si le processus est hors du terrain, nous sortons du kernel
  if (oSE(ptHTer,cH.dimDTer)) return;

  const float2 ptProj = GetProjection<TexSel>(ptHTer,cH.sampProj,blockIdx.z);

  uint pitZ,modZ;

  if (oI(ptProj,0))
    {
      //cacheImg[threadIdx.y][threadIdx.x]  = cH.floatDefault;
      return;
    }
  else
    {
      pitZ  = blockIdx.z / cH.nbImages;
      modZ  = blockIdx.z - pitZ * cH.nbImages;
      cacheImg[threadIdx.y][threadIdx.x] = GetImageValue(ptProj,modZ);
    }
  __syncthreads();

  const int2 ptTer = make_int2(ptHTer) - make_int2(cH.rayVig);
  // Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel
  if (oSE(threadIdx, nbActThrd + cH.rayVig) || oI(threadIdx , cH.rayVig) || oSE( ptTer, cH.dimTer) || oI(ptTer, 0))
    return;

  if(tex2D(TexS_MaskTer, ptTer.x, ptTer.y) == 0) return;

  const short2 c0	= make_short2(threadIdx) - cH.rayVig;
  const short2 c1	= make_short2(threadIdx) + cH.rayVig;

  // Intialisation des valeurs de calcul
  float aSV = 0.0f, aSVV = 0.0f;
  short2 pt;

#pragma unroll // ATTENTION PRAGMA FAIT AUGMENTER LA quantit� MEMOIRE des registres!!!
  for (pt.y = c0.y ; pt.y <= c1.y; pt.y++)
#pragma unroll
    for (pt.x = c0.x ; pt.x <= c1.x; pt.x++)
      {
        const float val = cacheImg[pt.y][pt.x];	// Valeur de l'image
//        if (val ==  cH.floatDefault) return;
        aSV  += val;		// Somme des valeurs de l'image cte
        aSVV += (val*val);	// Somme des carr�s des vals image cte
      }

#ifdef FLOATMATH
  aSV   = fdividef(aSV,(float)cH.sizeVig );
  aSVV  = fdividef(aSVV,(float)cH.sizeVig );
  aSVV -=	(aSV * aSV);
#else
  aSV	/=	cH.sizeVig;
  aSVV  /=	cH.sizeVig;
  aSVV  -=	(aSV * aSV);
#endif

  if ( aSVV <= cH.mAhEpsilon) return;

  aSVV =	rsqrtf(aSVV); // racine carre inverse

  const uint pitchCache = blockIdx.z * cH.sizeCach + ptTer.x * cH.dimVig.x;
  const uint pitchCachY = ptTer.y * cH.dimVig.y ;
#pragma unroll
  for ( pt.y = c0.y ; pt.y <= c1.y; pt.y++)
    {
      const int _py	= (pitchCachY + (pt.y - c0.y))* cH.dimCach.x;
#pragma unroll
      for ( pt.x = c0.x ; pt.x <= c1.x; pt.x++)
        cachVig[ pitchCache + _py  + (pt.x - c0.x)] = (cacheImg[pt.y][pt.x] -aSV)*aSVV;

    }

  const int ZPitch	= pitZ * cH.sizeTer;
  const int idN		= ZPitch + to1D(ptTer,cH.dimTer);
  atomicAdd( &dev_NbImgOk[idN], 1U);

}

/// \brief Fonction qui lance les kernels de correlation
extern "C" void	 KernelCorrelation(const int s,hipStream_t stream, dim3 blocks, dim3 threads, uint *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
{

  switch (s)
    {
    case 0:
      correlationKernel<0><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 0");
      break;
    case 1:
      correlationKernel<1><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 1");
      break;
    case 2:
      correlationKernel<2><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 2");
      break;
    case 3:
      correlationKernel<3><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 3");
      break;
    case 4:
      correlationKernel<4><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 4");
      break;
    case 5:
      correlationKernel<5><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 5");
      break;
    case 6:
      correlationKernel<6><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 6");
      break;
    case 7:
      correlationKernel<7><<<blocks, threads, 0, stream>>>( dev_NbImgOk, cachVig, nbActThrd);
      getLastCudaError("Basic Correlation kernel failed stream 7");
      break;
    }

}
/// \brief Kernel Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens utilisant des fonctions atomiques
template<int sNbTh> __global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, int* dev_NbImgOk, uint2 nbActThr)
{

  const ushort BB = ( 4 - sNbTh ) * SBLOCKDIM / 3;
  __shared__ float aSV [ BB ][ BB ];		// Somme des valeurs
  __shared__ float aSVV[ BB  ][ BB ];		// Somme des carr�s des valeurs
  __shared__ float resu[ BB/2 ][ BB/2 ];		// resultat
  __shared__ ushort nbIm[ BB/2][ BB/2 ];		// nombre d'images correcte

  // coordonn�es des threads
  const uint2 t = make_uint2(threadIdx);

  if ( threadIdx.z == 0)
    {
      aSV [t.y][t.x]		= 0.0f;
      aSVV[t.y][t.x]		= 0.0f;
      resu[t.y/2][t.x/2]	= 0.0f;
      nbIm[t.y/2][t.x/2]	= 0;
    }

  __syncthreads();

  if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

  // Coordonn�es 2D du cache vignette
  const uint2 ptCach = make_uint2(blockIdx) * nbActThr  + t;

  // Si le thread est en dehors du cache
  if ( oSE(ptCach, cH.dimCach))	return;

  const uint2	ptTer	= ptCach / cH.dimVig;						// Coordonn�es 2D du terrain

  if(tex2D(TexS_MaskTer, ptTer.x, ptTer.y) == 0) return;

  const uint	iTer	= blockIdx.z * cH.sizeTer + to1D(ptTer, cH.dimTer);	// Coordonn�es 1D dans le terrain
  const uint2   thTer	= t / cH.dimVig;									// Coordonn�es 2D du terrain dans le repere des threads
  const bool	mThrd	= aEq(t - thTer*cH.dimVig,0) && threadIdx.z == 0;

  if (mThrd)
    nbIm[thTer.y][thTer.x] = (ushort)dev_NbImgOk[iTer];

  __syncthreads();

  if (nbIm[thTer.y][thTer.x] < 2) return;

  const uint sizLayer = (blockIdx.z * cH.nbImages + threadIdx.z) * cH.sizeCach;	// Taille du cache vignette pour une image

  const uint2 cc    = ptTer * cH.dimVig;					// coordonn�es 2D 1er pixel de la vignette
  const int iCC     = sizLayer + to1D( cc, cH.dimCach );			// coordonn�es 1D 1er pixel de la vignette

  if (cacheVign[iCC]== cH.floatDefault) return;					// sortir si la vignette incorrecte

  const uint iCach  = sizLayer + to1D( ptCach, cH.dimCach );		// Coordonn�es 1D du cache vignette
  const float val   = cacheVign[iCach];

  atomicAdd( &(aSV[t.y][t.x]), val);
  atomicAdd(&(aSVV[t.y][t.x]), val * val);
  __syncthreads();

  if ( threadIdx.z != 0) return;

#ifdef FLOATMATH
  atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - fdividef(aSV[t.y][t.x] * aSV[t.y][t.x],(float)nbIm[thTer.y][thTer.x]));
#else
  atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - ((aSV[t.y][t.x] * aSV[t.y][t.x])/ nbIm[thTer.y][thTer.x]));
#endif

  if ( !mThrd ) return;
  __syncthreads();

  // Normalisation pour le ramener a un equivalent de 1-Correl
#ifdef FLOATMATH
  const float cost = fdividef( resu[thTer.y][thTer.x], (float)( nbIm[thTer.y][thTer.x] -1.0f) * (cH.sizeVig));
#else
  const float cost = resu[thTer.y][thTer.x]/ (( nbIm[thTer.y][thTer.x] -1.0f) * ((float)cH.sizeVig));
#endif

  dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));
}

/// \brief Kernel Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens n utilisant pas des fonctions atomiques
__global__ void multiCorrelationKernelNA(float *dTCost, float* cacheVign, int* dev_NbImgOk, uint2 nbActThr)
{

  __shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];		// Somme des valeurs
  __shared__ float aSVV[ SBLOCKDIM  ][ SBLOCKDIM ];		// Somme des carr�s des valeurs
  __shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];		// resultat
  __shared__ ushort nbIm[ SBLOCKDIM/2][ SBLOCKDIM/2 ];		// nombre d'images correcte

  // coordonn�es des threads
  const uint2 t = make_uint2(threadIdx);

  aSV [t.y][t.x]	= 0.0f;
  aSVV[t.y][t.x]	= 0.0f;
  resu[t.y/2][t.x/2]	= 0.0f;
  nbIm[t.y/2][t.x/2]	= 0;

  if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

  // Coordonn�es 2D du cache vignette
  const uint2 ptCach = make_uint2(blockIdx) * nbActThr + t;

  // Si le thread est en dehors du cache
  if ( oSE(ptCach, cH.dimCach))	return;

  const uint2	ptTer	= ptCach / cH.dimVig; // Coordonn�es 2D du terrain

  if(tex2D(TexS_MaskTer, ptTer.x, ptTer.y) == 0) return;

  const uint	iTer	= blockIdx.z * cH.sizeTer + to1D(ptTer, cH.dimTer);     // Coordonn�es 1D dans le terrain avec prise en compte des differents Z
  const uint2   thTer	= t / cH.dimVig;                                        // Coordonn�es 2D du terrain dans le repere des threads

  //if(aEq(t,thTer * cH.dimVig))
  nbIm[thTer.y][thTer.x] = (ushort)dev_NbImgOk[iTer];

  //__syncthreads();

  if ( nbIm[thTer.y][thTer.x]  < 2) return;

  const uint pitLayerCache  = blockIdx.z * cH.sizeCachAll + to1D( ptCach, cH.dimCach );	// Taille du cache vignette pour une image
  //const uint pit  = blockIdx.z * cH.nbImages;

 #pragma unroll
  for(uint i = 0;i< cH.sizeCachAll;i+=cH.sizeCach)
  //for(uint l = pit ;l< pit + cH.nbImages;l++)
    {
      const uint iCach = pitLayerCache  + i;
      const float val  = cacheVign[iCach];
      //const float val  = tex2DLayered( TexL_Cache,ptCach.x , ptCach.y,l);

      if(val!= cH.floatDefault)
        {
          // Coordonn�es 1D du cache vignette

          aSV[t.y][t.x]   += val;
          aSVV[t.y][t.x]  += val * val;
        }
    }

  __syncthreads();

  atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - fdividef(aSV[t.y][t.x] * aSV[t.y][t.x],(float)nbIm[thTer.y][thTer.x]));

  if (!aEq(t - thTer*cH.dimVig,0)) return;

  __syncthreads();

  // Normalisation pour le ramener a un equivalent de 1-Correl
  const float cost = fdividef( resu[thTer.y][thTer.x], (float)( nbIm[thTer.y][thTer.x] -1.0f) * (cH.sizeVig));

  dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));

}

/// \brief Fonction qui lance les kernels de multi-Correlation utilisant des fonctions atomiques
extern "C" void KernelmultiCorrelation(hipStream_t stream, dim3 blocks, dim3 threads, float *dTCost, float* cacheVign, int * dev_NbImgOk, uint2 nbActThr, ushort divideNThreads)
{

  switch (divideNThreads)
    {
    case 1:
      multiCorrelationKernel<1><<<blocks, threads, 0, stream>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
      break;
    case 2:
      multiCorrelationKernel<2><<<blocks, threads, 0, stream>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
      break;
    case 3:
      multiCorrelationKernel<3><<<blocks, threads, 0, stream>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
      break;
    default :
      multiCorrelationKernel<3><<<blocks, threads, 0, stream>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
    }

  getLastCudaError("Multi-Correlation kernel failed");

}

/// \brief Fonction qui lance les kernels de multi-Correlation n'utilisant pas des fonctions atomiques
extern "C" void KernelmultiCorrelationNA(hipStream_t stream, dim3 blocks, dim3 threads, float *dTCost, float* cacheVign, int * dev_NbImgOk, uint2 nbActThr)
{
  multiCorrelationKernelNA<<<blocks, threads, 0, stream>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
  getLastCudaError("Multi-Correlation NON ATOMIC kernel failed");

}
