#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_Z_H_
#define _OPTIMISATION_KERNEL_Z_H_

#include "GpGpu/GpGpu_StreamData.cuh"
#include "GpGpu/SData2Optimize.h"

// On pourrait imaginer un buffer des tailles calculer en parallel
// SIZEBUFFER[threadIdx.x] = count(lI[threadIdx.x]);


__device__ void GetConeZ(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    aDz.x =   aZ_Prev.x-aZ;
    if (aZ != aZ_Next.x)
        aDz.x = max(aDz.x,-MaxDeltaZ);

    aDz.y = aZ_Prev.y-1-aZ;
    if (aZ != aZ_Next.y-1)
        aDz.y = min(aDz.y,MaxDeltaZ);

    if (aDz.x > aDz.y)
        if (aDz.y <0)
            aDz.x = aDz.y;
        else
            aDz.y = aDz.x;
}

template<bool sens> __device__
void RunLine(   SimpleStream<short2>    &streamIndex,
                SimpleStream<uint>      &streamFCost,
                SimpleStream<ushort>    &streamICost,
                short2     *S_Bf_Index,
                ushort     *ST_Bf_ICost,
                uint       *S_FCost[2],
                ushort     &sId_ICost,
                uint        penteMax,
                uint        lenghtLine,
                short2     &prevIndex,
                int        &id_Line,
                ushort     &idSeg,
                bool       &idBuf)
{
    const ushort  tid   = threadIdx.x;
    short2* ST_Bf_Index = S_Bf_Index + sgn(tid);
    short2  ConeZ;
    uint    segLine     = 0;

    __shared__ uint globMinFCost;

    while(id_Line < lenghtLine)
    {

        uint  segLine = min(lenghtLine-id_Line,WARPSIZE);

        while(idSeg < segLine)
        {

            const short2 index  = S_Bf_Index[idSeg];
            const ushort dZ     = count(index); // creer buffer de count
            ushort       z      = 0;
            globMinFCost        = max_cost;

            while( z < dZ)
            {           
                if(sId_ICost > NAPPEMAX)
                {
                    streamICost.read<sens>(ST_Bf_ICost);
                    streamFCost.incre<sens>();
                    sId_ICost = 0;
                }

                uint fCostMin           = max_cost;
                const ushort costInit   = ST_Bf_ICost[sId_ICost];
                const ushort tZ         = z + tid;
                const short  Z          = index.x + tZ;
                const short prZ         = Z - prevIndex.x;

                GetConeZ(ConeZ,Z,penteMax,index,prevIndex);

                uint* prevFCost = S_FCost[idBuf] + prZ;

                ConeZ.y = min(NAPPEMAX - prZ,ConeZ.y );

                for (int i = ConeZ.x; i <= ConeZ.y; ++i)
                        fCostMin = min(fCostMin, costInit + prevFCost[i]);

                const uint fcost    =  fCostMin;// + sens * (streamFCost.GetValue(s_idCur_ICost) - costInit);

                if( tZ < NAPPEMAX)
                {
                    S_FCost[!idBuf][tZ] = fcost;
                    streamFCost.SetValue(sId_ICost, fcost);

                    if(!sens)
                        atomicMin(&globMinFCost,fcost);
                }

                const ushort pIdCost = sId_ICost;
                sId_ICost += min(dZ - z,WARPSIZE);
                z         += min(WARPSIZE,NAPPEMAX-pIdCost);
            }

            prevIndex = index;
            idSeg++;
            idBuf =!idBuf;
        }

        streamIndex.read<sens>(ST_Bf_Index);
        id_Line += segLine;
        idSeg   = 0;
    }

    idSeg = segLine - 1;
}

template<class T> __global__
void Run(ushort* g_ICost, short2* g_Index, uint* g_FCost, uint3* g_RecStrParam, uint penteMax)
{
    __shared__ short2   S_BuffIndex[WARPSIZE];
    __shared__ ushort   S_BuffICost0[NAPPEMAX + 2*WARPSIZE];
    __shared__ uint     S_BuffFCost0[NAPPEMAX + 2*WARPSIZE];
    __shared__ uint     S_BuffFCost1[NAPPEMAX + 2*WARPSIZE];
    __shared__ uint     pit_Id;
    __shared__ uint     pit_Stream;
    __shared__ uint     lenghtLine;

    const ushort    tid     = threadIdx.x;

    uint*    S_BuffFCost[2] = {S_BuffFCost0 + WARPSIZE,S_BuffFCost1 + WARPSIZE};
    ushort*  S_BuffICost    = S_BuffICost0 + WARPSIZE + tid;

    bool            idBuf   = false;
    ushort          s_id_Icost;

    if(!threadIdx.x)
    {
        uint3 recStrParam   = g_RecStrParam[blockIdx.x];
        pit_Stream          = recStrParam.x;
        pit_Id              = recStrParam.y;
        lenghtLine          = recStrParam.z;
    }

    __syncthreads();

    SimpleStream<ushort>    streamICost(g_ICost + pit_Stream,NAPPEMAX);
    SimpleStream<uint>      streamFCost(g_FCost + pit_Stream,NAPPEMAX);
    SimpleStream<short2>    streamIndex(g_Index + pit_Id    ,WARPSIZE);

    streamICost.read<eAVANT>(S_BuffICost);

    uint*   locFCost = S_BuffFCost[idBuf] + tid;
    for (ushort i = 0; i < NAPPEMAX; i+=WARPSIZE)
        locFCost[i] = S_BuffICost[i];

    streamIndex.read<eAVANT>(S_BuffIndex + tid);

    short2  prevIndex   = S_BuffIndex[0];
    int     id_Line     = 0;
    ushort  idSeg       = 1;

    s_id_Icost   = count(prevIndex);

    RunLine<eAVANT>(streamIndex,streamFCost,streamICost,S_BuffIndex,S_BuffICost,S_BuffFCost,s_id_Icost,penteMax,lenghtLine,prevIndex,id_Line,idSeg,idBuf);

//    streamFCost.reverse<eARRIERE>();
//    streamIndex.reverse<eARRIERE>();
//    streamICost.reverse<eARRIERE>();

//    S_BuffFCost[0]  += NAPPEMAX;
//    S_BuffFCost[1]  += NAPPEMAX;
//    S_BuffICost     += NAPPEMAX - 2*tid;

//    streamICost.readFrom<eARRIERE>(S_BuffFCost[idBuf] - tid, s_id_Icost - NAPPEMAX);

//    streamICost.incre<eARRIERE>();
//    streamFCost.incre<eARRIERE>();
//    streamIndex.incre<eARRIERE>();

//    prevIndex       = S_BuffIndex[idSeg];
//    idSeg           = WARPSIZE - idSeg;
//    id_Line         = -idSeg;

//    const short nonRead   = count(prevIndex) - s_id_Icost;

//    if(count(prevIndex) < s_id_Icost)
//        s_id_Icost = NAPPEMAX + nonRead;
//    else
//    {
//        streamICost.read<eARRIERE>(S_BuffICost);
//        streamFCost.incre<eARRIERE>();
//        s_id_Icost = nonRead;
//    }

//    RunLine<eARRIERE>(  streamIndex,
//                        streamFCost,
//                        streamICost,
//                        S_BuffIndex + WARPSIZE,
//                        S_BuffICost - tid,
//                        S_BuffFCost,
//                        s_id_Icost,
//                        penteMax,
//                        lenghtLine,
//                        prevIndex,
//                        id_Line,
//                        idSeg,
//                        idBuf);

}

extern "C" void OptimisationOneDirectionZ(Data2Optimiz<CuDeviceData3D> &d2O)
{
    uint deltaMax = 3;
    dim3 Threads(WARPSIZE,1,1);
    dim3 Blocks(d2O.NBlines(),1,1);

    Run< uint ><<<Blocks,Threads>>>
                                    (
                                        d2O.pInitCost(),
                                        d2O.pIndex(),
                                        d2O.pForceCostVol(),
                                        d2O.pParam(),
                                        deltaMax
                                        );
    getLastCudaError("kernelOptiOneDirection failed");
}

#endif //_OPTIMISATION_KERNEL_Z_H_

