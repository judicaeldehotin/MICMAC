#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_Z_H_
#define _OPTIMISATION_KERNEL_Z_H_

#include "GpGpu/GpGpu_StreamData.cuh"
#include "GpGpu/SData2Optimize.h"

__device__ inline void readDepthCoor(short2 *GBuffer, short2 *SBuffer, uint& id)
{
    SBuffer[threadIdx.x] = GBuffer[threadIdx.x + id];
    // On pourrait imaginer un buffer des tailles calculer en parallel
    // SIZEBUFFER[threadIdx.x] = count(lI[threadIdx]);
    id += WARPSIZE;
}

template<class T, bool sens> __device__
void RunLine(short2 *GBuffer,ushort *GInitCost)
{

    const uint lLine    = 256;
    uint  idRun         = 0;
    ushort tid          = threadIdx.x;
    __shared__ short2 depthCoor[WARPSIZE];
    __shared__ ushort initCost[NAPPEMAX];

    uint   G_idIdex  = 0;
    uint   G_idCost  = 0;
    ushort S_idCost  = 0;

    while(idRun < lLine)
    {
        readDepthCoor(GBuffer,depthCoor,G_idIdex);

        const uint Z2Comp = min(lLine-idRun,WARPSIZE);

        ushort l = 0;

        while(l < Z2Comp)
        {
            const short2 lI     = GBuffer[l];
            const ushort sLI    = count(lI);

            if(S_idCost + sLI > NAPPEMAX)
            {
                initCost[threadIdx.x] = initCost[S_idCost + tid];
                S_idCost = 0;
                for(ushort i = 0;i<NAPPEMAX-WARPSIZE;i+=WARPSIZE)
                    initCost[i + tid] = GInitCost[G_idCost + i + tid];
            }

            l++;
        }

        idRun += WARPSIZE;
    }

}


#endif //_OPTIMISATION_KERNEL_Z_H_
