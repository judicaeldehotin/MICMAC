#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_Z_H_
#define _OPTIMISATION_KERNEL_Z_H_

#include "GpGpu/GpGpu_StreamData.cuh"
#include "GpGpu/SData2Optimize.h"

// On pourrait imaginer un buffer des tailles calculer en parallel
// SIZEBUFFER[threadIdx.x] = count(lI[threadIdx.x]);


__device__ void GetConeZ(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    aDz.x =   aZ_Prev.x-aZ;
    if (aZ != aZ_Next.x)
        aDz.x = max(aDz.x,-MaxDeltaZ);

    aDz.y = aZ_Prev.y-1-aZ;
    if (aZ != aZ_Next.y-1)
        aDz.y = min(aDz.y,MaxDeltaZ);

    if (aDz.x > aDz.y)
        if (aDz.y <0)
            aDz.x = aDz.y;
        else
            aDz.y = aDz.x;
}

template<bool sens>
__device__ inline void ReadIndex(short2 *g_BuffIdex, short2 *s_BuffIdex, uint& g_id,ushort* dZ)
{
    *(s_BuffIdex) = *(g_BuffIdex + g_id);
    *(dZ) = count(*(s_BuffIdex));
    g_id += sgn(WARPSIZE);
}

template<bool sens>
__device__ inline void ReadInitCost(ushort *g__ICst, ushort* s__ICst, ushort& s_id, uint& g_id)
{    
    for(ushort i = 0; i < sgn(NAPPEMAX); i+= sgn(WARPSIZE))
        *(s__ICst + i) = *(g__ICst + i);

    s_id  = 0;
    g_id += sgn(NAPPEMAX);
}

template<class T, bool sens> __device__
void RunLine(SimpleStream<short2> &streamIndex, SimpleStream<uint> streamFCost, SimpleStream<ushort> &streamICost,short2* S_Bf_Index,ushort *ST_Bf_ICost, uint S_FCost[][NAPPEMAX + WARPSIZE], uint penteMax, uint lenghtLine,bool &idBuf)
{
    const ushort  tid       = threadIdx.x;
    short2* ST_Bf_Index     = S_Bf_Index + tid;

    __shared__ uint globMinFCost;

    short2 ConeZ;
    short2 prevIndex;

    streamICost.read<sens>(ST_Bf_ICost);

    for (ushort i = 0; i < NAPPEMAX; i+=WARPSIZE)
        S_FCost[idBuf][i +tid] = ST_Bf_ICost[i];

    streamIndex.read<sens>(ST_Bf_Index);
    ushort sId_ICost = count(S_Bf_Index[0]);

    uint  id_Line = 1;

    while(id_Line < lenghtLine)
    {

        const uint  segLine = min(lenghtLine-id_Line,WARPSIZE);
        ushort      idSeg   = 0;

        while(idSeg < segLine)
        {

            const short2 index  = S_Bf_Index[idSeg];
            const ushort dZ     = count(index); // creer buffer de count pre calculer en Multi threading lors de l'aquisition des index

            ushort       z      = 0;
            globMinFCost        = max_cost;

            while( z < dZ)
            {           

                if(sId_ICost > NAPPEMAX)
                {
                    if(z + NAPPEMAX < dZ )
                    {                 
                        streamICost.read<sens>(ST_Bf_ICost); /// ERREUR DE DEPASSEMENT!!!
                        streamFCost.incre<sens>();
                    }
                    sId_ICost = 0;
                }

                uint fCostMin           = max_cost;
                const ushort costInit   = ST_Bf_ICost[sId_ICost];
                const ushort tZ         = z + tid;
                const short  Z          = index.x + tZ;

                GetConeZ(ConeZ,Z,penteMax,index,prevIndex);

                uint* prevFCost = S_FCost[idBuf] + Z - prevIndex.x;

                #pragma unroll
                for (int i = ConeZ.x; i < ConeZ.y; ++i)
                    fCostMin = min(fCostMin, costInit + *(prevFCost+i));

                const uint fcost    =  fCostMin;// + sens * (streamFCost.GetValue(s_idCur_ICost) - costInit);

                S_FCost[!idBuf][tZ] = fcost;

                if(tZ < dZ)
                streamFCost.SetValue(sId_ICost, fcost);

                if(!sens)
                    atomicMin(&globMinFCost,fcost);

                z         += WARPSIZE;
                sId_ICost += WARPSIZE;
            }

            prevIndex = index;
            idSeg++;
            idBuf =!idBuf;
        }

        streamIndex.read<sens>(ST_Bf_Index);
        id_Line += segLine;
    }

 //   if(blockIdx.x == 35 && !tid)
//        printf(" Count : %d/%d", counter,compareCount);
}

template<class T> __global__
void Run(ushort* g_ICost, short2* g_Index, uint* g_FCost, uint3* g_RecStrParam, uint penteMax)
{

    __shared__ short2   S_BuffIndex[WARPSIZE];
    __shared__ ushort   S_BuffICost[NAPPEMAX + WARPSIZE];
    __shared__ uint     S_BuffFCost[2][NAPPEMAX + WARPSIZE];
    __shared__ uint     pit_Id;
    __shared__ uint     pit_Stream;
    __shared__ uint     lenghtLine;

    bool                idBuf       = false;

    if(!threadIdx.x)
    {
        uint3 recStrParam   = g_RecStrParam[blockIdx.x];
        pit_Stream          = recStrParam.x;
        pit_Id              = recStrParam.y;
        lenghtLine          = recStrParam.z;
    }

    __syncthreads();

    SimpleStream<ushort>    streamICost(g_ICost + pit_Stream,NAPPEMAX);
    SimpleStream<uint>      streamFCost(g_FCost + pit_Stream,NAPPEMAX);
    SimpleStream<short2>    streamIndex(g_Index + pit_Id    ,WARPSIZE);

    RunLine<T,true>(streamIndex,streamFCost,streamICost,S_BuffIndex,S_BuffICost + threadIdx.x,S_BuffFCost,penteMax,lenghtLine,idBuf);
    //RunLine<T,true>(streamIndex,streamFCost,streamICost,S_BuffIndex,S_BuffICost,S_BuffFCost,penteMax,lenghtLine,idBuf);

//    g_idIX -= WARPSIZE;
//    g_idICO-= NAPPEMAX;

   // RunLine<T,false>(streamIndex,streamFCost,streamICost,S_BuffIndex,S_BuffICost,S_BuffFCost,penteMax,lenghtLine,idBuf);
}

extern "C" void OptimisationOneDirectionZ(Data2Optimiz<CuDeviceData3D> &d2O)
{
    uint deltaMax = 3;
    dim3 Threads(WARPSIZE,1,1);
    dim3 Blocks(d2O.NBlines(),1,1);

    Run< uint ><<<Blocks,Threads>>>
                                    (
                                        d2O.pInitCost(),
                                        d2O.pIndex(),
                                        d2O.pForceCostVol(),
                                        d2O.pParam(),
                                        deltaMax
                                        );
    getLastCudaError("kernelOptiOneDirection failed");
}

#endif //_OPTIMISATION_KERNEL_Z_H_

