#include "hip/hip_runtime.h"

#include "GpGpu/GpGpu_CommonHeader.h"
//#include "GpGpu/GpGpu_TextureTools.cuh"
#include "GpGpu/GpGpu_Interface_CorMultiScale.h"

// Algorithme Correlation multi echelle sur ligne epipolaire

// Donn�es :
//  - 2 images avec diff�rents niveaux de floutage

//
// * Pr�-calcul et param�tres                                   |       GPU
// -------------------------------------------------------------|----------------------------------
// - Tableau de parcours des vignettes                          >>      constant 3d data short2
// - poids des echelles                                         >>      constant 2d data ???
// - Tableau du ZMin et ZMax de chaque coordonn�es terrain      >>      global 2D data short2
// - les offsets Terrain <--> Image Epi                         >>      constant 2 x int2
// - le masque erod� de l'image 1                               >>      1 texture pixel
// - 2 images x N echelles                                      >>      2 textures layered float


//
// Phase mNbByPix // pas utilise en GPU


/*  CPU
 *
 *  pour chaque
 *      - calcul des images interpol�es pour l'image 1
 *      - mise en vecteur des images interpol�es
 *      - Precalcul somme et somme quad
 *      - Parcour du terrain
 *      - Calcul des images interpol�
 *      - Parcours des Z
 *          - Calcul de la projection image 1
 *          - Calcul de la correlation Quick_MS_CorrelBasic_Center
 *              - pour chaque echelle
 *                  - Calcul de correlation
 *
 *      - set cost dans la matrice de regularisation
 */


///
static __constant__ const_Param_Cor_MS     cstP_CorMS;

extern "C" void paramCorMultiScale2Device( const_Param_Cor_MS &param )
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cstP_CorMS), &param, sizeof(const_Param_Cor_MS)));
}

texture< float,	hipTextureType2DLayered >      texture_ImageEpi_00;
texture< float,	hipTextureType2DLayered >      texture_ImageEpi_01;
texture< pixel,	hipTextureType2D >             Texture_Masq_Erod_00;
texture< pixel,	hipTextureType2D >             Texture_Masq_Erod_01;

extern "C" textureReference& texture_ImageEpi(int nEpi){return nEpi == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01;}

extern "C" textureReference* pTexture_ImageEpi(int nEpi){return nEpi == 0 ? &texture_ImageEpi_00 : &texture_ImageEpi_01;}

extern "C" textureReference* ptexture_Masq_Erod(int nEpi){return nEpi == 0 ? &Texture_Masq_Erod_00 : &Texture_Masq_Erod_01;}

__device__
inline    bool GET_Val_BIT(const U_INT1 * aData,int anX)
{
    return (aData[anX/8] >> (7-anX %8) ) & 1;
}

__device__
inline    texture< pixel,hipTextureType2D>  getMask(ushort iDi)
{
    return iDi == 0 ? Texture_Masq_Erod_00 : Texture_Masq_Erod_01;
}


__device__
inline    bool IsOkErod(uint3 pt)
{
    // TODO peut etre simplifier % et division

    const int ptxBy8 = pt.x >> 3;           // pt.x >> 3 Division par 8
    const int modulo = pt.x - (ptxBy8 << 3)  ;// (ptxBy8<<3) multiplication par 8

    pixel mask8b = tex2D(getMask(pt.z),(float)(ptxBy8) + 0.5f,(float)pt.y + 0.5f);

    return (mask8b >> (7-modulo ) ) & 1;
}

__device__
inline    bool IsOkErod(uint2 pt,ushort idi)
{
    return IsOkErod(make_uint3(pt.x,pt.y,idi));
}

__device__
inline    texture< float,	hipTextureType2DLayered >  getTexture(ushort iDi)
{
    return iDi == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01;
}

__device__
inline    float getValImage(float2 pt,ushort iDi,ushort nScale)
{
    return tex2DLayered(getTexture(iDi),pt.x + 0.5f,pt.y + 0.5f ,nScale);
}

/*
//Algorithme de precalcul de corr�lation
__device__
inline    void correl(float2 pt,ushort iDi, float* mdata1, float* mdata2)
{
    float aGlobSom1 = 0;
    float aGlobSom2 = 0;
    float aGlobPds  = 0;

    for (int aKS=0 ; aKS< cPCencus.aNbScale ; aKS++)
    {
        float   aSom1   = 0;
        float   aSom2   = 0;
        short2 *aVP     = cPCencus.aVV[aKS];
        ushort  aNbP    = cPCencus.size_aVV[aKS];
        float   aPdsK   = cPCencus.aVPds[aKS];

        for (int aKP=0 ; aKP<aNbP ; aKP++)
        {
            const short2 aP = aVP[aKP];
            //const uint ptV  = make_uint2(pt.x+aP.x,)
            float aV = getValImage(pt+aP,iDi,aKS);
            aSom1 += aV;
            aSom2 += aV*aV;
        }

        aGlobSom1 += aSom1 * aPdsK;
        aGlobSom2 += aSom2 * aPdsK;
        aGlobPds += aPdsK * aNbP;

//        mData1[aKS][aYGlob][aXGlob] = aGlobSom1 / aGlobPds;
//        mData2[aKS][aYGlob][aXGlob] = aGlobSom2 / aGlobPds;

    }
}
*/


// calcul rapide de la correlation multi-echelles centre sur une vignette
__device__
inline    float Quick_MS_CorrelBasic_Center(

    const float2 & aPG0,
    const float2 & aPG1,

//    float ***  aSom1,
//    float ***  aSom11,
//    float ***  aSom2,
//    float ***  aSom22,
    float*  aSom1,
    float*  aSom11,
    float*  aSom2,
    float*  aSom22,
    int     aPx2,
    bool    ModeMax,
    ushort  aPhase)
{
    float aMaxCor = -1;
    float aCovGlob = 0;
    float aPdsGlob = 0;

    const float2 aPG1_x2 = make_float2(aPG1.x + aPx2,aPG1.y);

    int aNbScale = cstP_CorMS.aNbScale;
    for (int aKS=0 ; aKS< aNbScale ; aKS++)
    {
         bool   aLast   = (aKS==(aNbScale-1));
         short2*aVP     = cstP_CorMS.aVV[aKS];
         float  aPds    = cstP_CorMS.aVPds[aKS];
         float  aCov    = 0;
         ushort aNbP    = cstP_CorMS.size_aVV[aKS];

//         float ** anIm1= aVBOI1[aKS]->data();
//         float ** anIm2= aVBOI2[aKS]->data();

         aPdsGlob += aPds * aNbP;
         for (int aKP=0 ; aKP<aNbP ; aKP++)
         {
             const short2 aP = aVP[aKP];

             const float valima_0 = getValImage(aPG0 + aP,0,aKS);
             const float valima_1 = getValImage(aPG1_x2 + aP,1,aKS); // ATTENTION LE PAS SUB PIXELAIRE !!!

             aCov += valima_0*valima_1;
             //aCov += anIm1[aP.y][aP.x]*anIm2[aP.y][aP.x+aPx2];
         }

         aCovGlob += aCov * aPds;

         if (ModeMax || aLast)
         {
//             float aM1  = aSom1 [aKS][aPG0.y][aPG0.x];
//             float aM2  = aSom2 [aKS][aPG1.y][aPG1.x];
//             float aM11 = aSom11[aKS][aPG0.y][aPG0.x] - aM1*aM1;
//             float aM22 = aSom22[aKS][aPG1.y][aPG1.x] - aM2*aM2;

             const uint3 pt0    =   make_uint3(aPG0.x,aPG0.y,aKS);
             const uint3 pt1    =   make_uint3(aPG1_x2.x,aPG1_x2.y,aKS + aNbScale*aPhase);
             const uint3 dim    =   make_uint3(cstP_CorMS._dimTerrain.x,cstP_CorMS._dimTerrain.x,1);

             const float aM1    =   aSom1 [to1D(pt0,dim)];
             const float aM2    =   aSom2 [to1D(pt1,dim)];

             const float aM11   =   aSom11[to1D(pt0,dim)] - aM1*aM1;
             const float aM22   =   aSom22[to1D(pt1,dim)] - aM2*aM2;

             const float aM12   =   aCovGlob / aPdsGlob   - aM1 * aM2;

             if (ModeMax)
             {
                float aCor = (aM12 * abs(aM12)) /max(cstP_CorMS.anEpsilon,aM11*aM22);
                aMaxCor = max(aMaxCor,aCor);
             }
             else
                return aM12 / sqrt(max(cstP_CorMS.anEpsilon,aM11*aM22));
        }

    }
    return (aMaxCor > 0) ? sqrt(aMaxCor) : - sqrt(-aMaxCor) ;
}

__global__
void projectionMasqImage(float * dataPixel,uint3 dTer)
{

    if(blockIdx.x > cstP_CorMS._dimTerrain.x || blockIdx.y > cstP_CorMS._dimTerrain.y)
        return;

    const uint3 pt = make_uint3(blockIdx.x,blockIdx.y,blockIdx.z);

    float valImage = tex2DLayered(pt.z == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01 ,pt.x + 0.5f,pt.y + 0.5f ,0);

    dataPixel[to1D(pt,dTer)] = IsOkErod(pt) ? valImage/(32768.f) : 0;
}

__global__
void KernelDoCorrelMultiScale(float* aSom1,float*  aSom11,float* aSom2,float*  aSom22,short2 *nappe, float *cost)
{

    // ??? TODO � cabler
    bool    DoMixte     = false;
    bool    aModeMax    = false;
    float   aSeuilHC    = 1.0;
    float   aSeuilBC    = 1.0;
    // ???

    int2    pt  =   make_int2(blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);
    uint    tZ  =   blockIdx.z*blockDim.z + threadIdx.z;

    if(oSE(pt,cstP_CorMS._dimTerrain))
        return;

    const int2  aPIm0   =   pt.x+cstP_CorMS.anOff0; // TODO Attention au unsigned
    const bool  OkIm0   =   IsOkErod(make_uint2(aPIm0),0);
    const short2 iZ     =   nappe[to1D(pt,cstP_CorMS._dimTerrain)];
    int aZ0             =   iZ.x;
    const int aZ1       =   iZ.y;
    const int DeltaZ    =   abs(aZ1-aZ0);

    if(tZ>DeltaZ)
        return; // TODO on pourrait eventuellement affacter la valeur du cout par defaut.... mais bof

    int aZI = aZ0 + tZ;

    const int2 aIm1SsPx =  pt + cstP_CorMS.anOff1;

    // float aGlobCostGraphe = 0;
    float aGlobCostBasic  = 0;
    float aGlobCostCorrel = 0;

    float aCost = cstP_CorMS.mAhDefCost;

    if (OkIm0)
    {
        //
        // anOffset calcul de anOffset
        ///
        int aPhase = tZ%cstP_CorMS.mNbByPix;

        while ((aZ0%cstP_CorMS.mNbByPix) != aPhase) aZ0++;

        int anOffset    = aZ0 / cstP_CorMS.mNbByPix;
        anOffset        = anOffset - ((anOffset * cstP_CorMS.mNbByPix) > aZ0);
        int sOff        = abs(aZI-aZ0)/cstP_CorMS.mNbByPix; // --> doit tomber juste
        anOffset       += sOff;

        const uint2 aPIm1 = make_uint2(aIm1SsPx.x+anOffset,aIm1SsPx.y);

        if (IsOkErod(aPIm1,1))
        {

            // TODO � cabler avec correl(uint2 pt,ushort iDi)
            //float*  aSom1;  // ---> peut precalculer dans un kernel precedent!
            //float*  aSom11; // ---> peut precalculer dans un kernel precedent!

            // TODO � cabler avec correl(uint2 pt,ushort iDi)
            //float*  aSom2; // ---> peut-etre precalculer dans un kernel precedent! A VERIFIER!!!
            //float*  aSom22;// ---> peut-etre precalculer dans un kernel precedent! A VERIFIER!!!

            const float2 faPIm0 = make_float2((float)aPIm0.x,(float)aPIm0.y); // TODO ajouter le pas sub pixelaire            
            const float2 faPIm1 = make_float2((float)aPIm1.x,(float)aPIm1.y); // TODO ajouter le pas sub pixelaire


            // FAUX !!!!
            const int    aPx2   = aPhase*cstP_CorMS.aStepPix;
            // FAUX !!!!

            aCost = Quick_MS_CorrelBasic_Center(faPIm0,faPIm1,aSom1,aSom11,aSom2,aSom22,aPx2,aModeMax,aPhase);

            aGlobCostCorrel = aCost;

            if (DoMixte)
            {
               if(aGlobCostCorrel>aSeuilHC)

                    aCost = aGlobCostCorrel;

               else if (aGlobCostCorrel>aSeuilBC)
               {
                    float aPCor =  (aGlobCostCorrel - aSeuilBC) / (aSeuilHC-aSeuilBC);
                    aCost       =  aPCor * aGlobCostCorrel + (1-aPCor) * aSeuilBC *  aGlobCostBasic;
               }
               else
                    aCost =  aSeuilBC *  aGlobCostBasic;
            }

//            aCost = 1.f-aCost;

            const uint3 ptCost  = make_uint3(pt.x,pt.y,tZ);
            const uint3 dimCost = make_uint3(cstP_CorMS._dimTerrain.x,cstP_CorMS._dimTerrain.y,1);

            cost[to1D(ptCost,dimCost)] = 1.f-aCost;

        }
        else return;
    }
    else
        return;

}

extern "C" void LaunchKernelCorrelationMultiScalePreview(dataCorrelMS &data,const_Param_Cor_MS &param)
{
    dim3	threads( 1, 1, 1);
    dim3	blocks(param._dimTerrain.x , param._dimTerrain.y, 2);

    CuHostData3D<float>     hData;
    CuDeviceData3D<float>   dData;

    uint3 dTer  = make_uint3(param._dimTerrain.x , param._dimTerrain.y,2);
    uint2 dTer2 = make_uint2(dTer);

    hData.Malloc(dTer2,2);
    dData.Malloc(dTer2,2);
    hData.Fill(0.f);
    dData.Memset(0);

    projectionMasqImage<<<blocks, threads>>>(dData.pData(),dTer);

    dData.CopyDevicetoHost(hData);

    GpGpuTools::Array1DtoImageFile(hData.pData()    ,"ET_HOP_0.pmg",hData.GetDimension());
    GpGpuTools::Array1DtoImageFile(hData.pLData(1)  ,"ET_HOP_1.pmg",hData.GetDimension());
}

__global__
void KernelPrepareCorrel(ushort idImage,float aStepPix, ushort mNbByPix, float* mSom, float* mSomSqr)
{

    // point image
    const uint2     pt          =   make_uint2(blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);

    if(oSE(pt,cstP_CorMS._dimTerrain))
        return;

    // indice de l'etape sub pixelaire, le maximum �tant cPCencus.mNbByPix
    const ushort    etapeSub    =   (ushort)blockIdx.z;

    // la dimension du cache, la cache stocke des precaluls pour la corr�lation
    const uint3     dimCache    =   make_uint3(cstP_CorMS._dimTerrain.x,cstP_CorMS._dimTerrain.y,mNbByPix*cstP_CorMS.aNbScale);

    // le d�calage sub pixelaire
    const float     cStepPix    =   ((float)etapeSub)*aStepPix;

    // point de l'image pour cette etape sub pixelaire
    const float2    ptImage     =   make_float2((float)pt.x + cStepPix,(float)pt.y);

    float aGlobSom = 0;
    float aGlobSomSqr = 0;
    float aGlobPds  = 0;

    // pour toutes les echelles
    for (int aKS=0 ; aKS< cstP_CorMS.aNbScale ; aKS++)
    {
        float   aSom    = 0;
        float   aSomSqr = 0;
        short2 *aVP     = cstP_CorMS.aVV[aKS];
        ushort  aNbP    = cstP_CorMS.size_aVV[aKS];
        float   aPdsK   = cstP_CorMS.aVPds[aKS];

        // pour les �l�ments de la vignettes
        for (int aKP=0 ; aKP<aNbP ; aKP++)
        {
            const short2 aP = aVP[aKP];
            float aV = getValImage(ptImage+aP,idImage,aKS);
            aSom += aV;
            aSomSqr += aV*aV;
        }

        aGlobSom    += aSom     * aPdsK;
        aGlobSomSqr += aSomSqr  * aPdsK;
        aGlobPds    += aPdsK    * aNbP;

        // indice dans le cache
        const uint3     p3d        =   make_uint3(pt.x,pt.y,etapeSub*mNbByPix + aKS);

        // Ecriture dans le cache des
        mSom    [to1D(p3d,dimCache)] = aGlobSom    / aGlobPds;
        mSomSqr [to1D(p3d,dimCache)] = aGlobSomSqr / aGlobPds;

    }
}
inline __device__ int dElise_div(int a,int b)
{
       int res = a / b;
       return res - ((res * b) > a);
}

__global__
void Kernel__DoCorrel_MultiScale_Global(float* aSom1,float*  aSom11,float* aSom2,float*  aSom22,short2 *nappe, float *cost)
{

    // point image
    const uint2  an  =   make_uint2(blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);

    // Z relatif au thread
    const ushort thZ   =   blockIdx.z*blockDim.z + threadIdx.z;

    const uint   pit =   to1D(make_uint3(an.x,an.y,thZ),make_uint3(cstP_CorMS._dimTerrain));

//    float&          _cost    =   cost[pit];
    const short2    _nappe   =   nappe[pit];
    short aZ0                = _nappe.x;

    // z Absolu
    const short aZ = (short)thZ + aZ0;

    // calcul de la phase
    const ushort aPhase = (ushort)(((int)aZ)%cstP_CorMS.mNbByPix);

    /// peut etre precalcul  -- voir simplifier
    while (aZ0%cstP_CorMS.mNbByPix != aPhase) aZ0++;
    int anOffset = dElise_div((int)_nappe.x,cstP_CorMS.mNbByPix);

    const uint2 aIm1SsPx    =   make_uint2((int)an.x + cstP_CorMS.anOff1.x,(int)an.y + cstP_CorMS.anOff1.y);
    const uint2 aPIm0       =   make_uint2((int)an.x + cstP_CorMS.anOff0.x,(int)an.y + cstP_CorMS.anOff0.y);
    const uint2 aPIm1       =   make_uint2(aIm1SsPx.x+anOffset,aIm1SsPx.y);



//    while (mod(aZ0,mNbByPix) != aPhase) aZ0++;
//    int anOffset = Elise_div(aZ0,mNbByPix);

}

extern "C" void LaunchKernel__Correlation_MultiScale(dataCorrelMS &data,const_Param_Cor_MS &parCMS)
{
    // Cache device
    //CuUnifiedData3D<float>  aSom1;

    CuDeviceData3D<float>  aSom_0;
    CuDeviceData3D<float>  aSomSqr_0;

    CuDeviceData3D<float>  aSom_1;
    CuDeviceData3D<float>  aSomSqr_1;

    aSom_0   .Malloc (parCMS._dimTerrain,parCMS.aNbScale); //  pas de sous echantillonnage
    aSomSqr_0.Malloc (parCMS._dimTerrain,parCMS.aNbScale);

    aSom_1   .Malloc (parCMS._dimTerrain,parCMS.aNbScale*parCMS.mNbByPix); // avec sous echantillonnage
    aSomSqr_1.Malloc (parCMS._dimTerrain,parCMS.aNbScale*parCMS.mNbByPix);

    dim3	threads( 32, 32, 1);

    uint    divDTerX = iDivUp32(parCMS._dimTerrain.x);
    uint    divDTerY = iDivUp32(parCMS._dimTerrain.y);

    dim3	blocks_00(divDTerX,divDTerY, 1);
    dim3	blocks_01(divDTerX,divDTerY, parCMS.mNbByPix);

    /// Les donn�es sont structur�es par calques
    /// les echelles (du m�me subpixel) sont regroup�es par calques cons�cutifs
    KernelPrepareCorrel<<<blocks_00,threads>>>(0,1,1,aSom_0.pData(),aSomSqr_0.pData());
    KernelPrepareCorrel<<<blocks_01,threads>>>(1,parCMS.aStepPix,parCMS.mNbByPix,aSom_1.pData(),aSomSqr_1.pData());

    ushort  modThreadZ = 8;

    dim3	threads_CorMS( 32, 32, modThreadZ);

    uint    bC =  iDivUp(data._maxDeltaZ,modThreadZ);

    dim3    blocks__CorMS(divDTerX,divDTerY,bC);


    /// calcul des couts de correlation multi-echelles
    Kernel__DoCorrel_MultiScale_Global<<<threads_CorMS,blocks__CorMS>>>(
                                                        aSom_0   .pData(),
                                                        aSomSqr_0.pData(),
                                                        aSom_1   .pData(),
                                                        aSomSqr_1.pData(),
                                                        data._uInterval_Z   .pData(),
                                                        data._uCost         .pData());

//    aSom1.syncHost();
//    aSom1.hostData.OutputValues();

    aSom_0   .Dealloc();
    aSomSqr_0.Dealloc();
    aSom_1   .Dealloc();
    aSomSqr_1.Dealloc();

}
