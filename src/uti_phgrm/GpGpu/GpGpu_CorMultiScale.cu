#include "hip/hip_runtime.h"

#include "GpGpu/GpGpu_CommonHeader.h"
//#include "GpGpu/GpGpu_TextureTools.cuh"
#include "GpGpu/GpGpu_Interface_CorMultiScale.h"

// Algorithme Correlation multi echelle sur ligne epipolaire

// Donn�es :
//  - 2 images avec diff�rents niveaux de floutage

//
// * Pr�-calcul et param�tres                                   |       GPU
// -------------------------------------------------------------|----------------------------------
// - Tableau de parcours des vignettes                          >>      constant 3d data short2
// - poids des echelles                                         >>      constant 2d data ???
// - Tableau du ZMin et ZMax de chaque coordonn�es terrain      >>      global 2D data short2
// - les offsets Terrain <--> Image Epi                         >>      constant 2 x int2
// - le masque erod� de l'image 1                               >>      1 texture pixel
// - 2 images x N echelles                                      >>      2 textures layered float


//
// Phase mNbByPix // pas utilise en GPU


/*  CPU
 *
 *  pour chaque
 *      - calcul des images interpol�es pour l'image 1
 *      - mise en vecteur des images interpol�es
 *      - Precalcul somme et somme quad
 *      - Parcour du terrain
 *      - Calcul des images interpol�
 *      - Parcours des Z
 *          - Calcul de la projection image 1
 *          - Calcul de la correlation Quick_MS_CorrelBasic_Center
 *              - pour chaque echelle
 *                  - Calcul de correlation
 *
 *      - set cost dans la matrice de regularisation
 */


///
static __constant__ const_Param_Cor_MS     cstP_CorMS;

extern "C" void paramCorMultiScale2Device( const_Param_Cor_MS &param )
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cstP_CorMS), &param, sizeof(const_Param_Cor_MS)));
}

texture< float,	hipTextureType2DLayered >      texture_ImageEpi_00;
texture< float,	hipTextureType2DLayered >      texture_ImageEpi_01;
texture< pixel,	hipTextureType2D >             Texture_Masq_Erod_00;
texture< pixel,	hipTextureType2D >             Texture_Masq_Erod_01;

extern "C" textureReference& texture_ImageEpi(int nEpi){return nEpi == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01;}

extern "C" textureReference* pTexture_ImageEpi(int nEpi){return nEpi == 0 ? &texture_ImageEpi_00 : &texture_ImageEpi_01;}

extern "C" textureReference* ptexture_Masq_Erod(int nEpi){return nEpi == 0 ? &Texture_Masq_Erod_00 : &Texture_Masq_Erod_01;}

__device__
inline    bool GET_Val_BIT(const U_INT1 * aData,int anX)
{
    return (aData[anX/8] >> (7-anX %8) ) & 1;
}

__device__
inline    texture< pixel,hipTextureType2D>  getMask(ushort iDi)
{
    return iDi == 0 ? Texture_Masq_Erod_00 : Texture_Masq_Erod_01;
}

inline __device__ int dElise_div(int a,int b)
{
       int res = a / b;
       return res - ((res * b) > a);
}

__device__
inline    bool IsOkErod(int3 pt)
{
    // TODO peut etre simplifier % et division

    const int ptxBy8 = pt.x >> 3;           // pt.x >> 3 Division par 8
    const int modulo = pt.x - (ptxBy8 << 3)  ;// (ptxBy8<<3) multiplication par 8

    pixel mask8b = tex2D(getMask(pt.z),(float)(ptxBy8) + 0.5f,(float)pt.y + 0.5f);

    return (mask8b >> (7-modulo ) ) & 1;
}

__device__
inline    bool IsOkErod(int2 pt,ushort idi)
{
    return IsOkErod(make_int3(pt.x,pt.y,idi));
}

__device__
inline    texture< float,	hipTextureType2DLayered >  getTexture(ushort iDi)
{
    return iDi == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01;
}

__device__
inline    float getValImage(float2 pt,ushort iDi,ushort nScale)
{
    return tex2DLayered(getTexture(iDi),pt.x + 0.5f,pt.y + 0.5f ,nScale);
}

template<class T>
__device__ float getValImage(T pt,ushort iDi,ushort nScale)
{
    return tex2DLayered(getTexture(iDi),(float)pt.x + 0.5f,(float)pt.y + 0.5f ,nScale);
}

__global__
void projectionMasqImage(float * dataPixel,uint3 dTer)
{

    if(blockIdx.x > cstP_CorMS._dimTerrain.x || blockIdx.y > cstP_CorMS._dimTerrain.y)
        return;

    const int3 pt = make_int3(blockIdx.x,blockIdx.y,blockIdx.z);

    float valImage = tex2DLayered(pt.z == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01 ,pt.x + 0.5f,pt.y + 0.5f ,0);

    dataPixel[to1D(pt,dTer)] = IsOkErod(pt) ? valImage/(32768.f) : 0;
}

extern "C" void LaunchKernelCorrelationMultiScalePreview(dataCorrelMS &data,const_Param_Cor_MS &param)
{
    dim3	threads( 1, 1, 1);
    dim3	blocks(param._dimTerrain.x , param._dimTerrain.y, 2);

    CuHostData3D<float>     hData;
    CuDeviceData3D<float>   dData;

    uint3 dTer  = make_uint3(param._dimTerrain.x , param._dimTerrain.y,2);
    uint2 dTer2 = make_uint2(dTer);

    hData.Malloc(dTer2,2);
    dData.Malloc(dTer2,2);
    hData.Fill(0.f);
    dData.Memset(0);

    projectionMasqImage<<<blocks, threads>>>(dData.pData(),dTer);

    dData.CopyDevicetoHost(hData);

    GpGpuTools::Array1DtoImageFile(hData.pData()    ,"ET_HOP_0.pmg",hData.GetDimension());
    GpGpuTools::Array1DtoImageFile(hData.pLData(1)  ,"ET_HOP_1.pmg",hData.GetDimension());
}

//  pre-calcul pour la correlation multi echelle et mise en cache dans mSom et mSomSqr
__global__
void KernelPrepareCorrel(ushort idImage,float aStepPix, ushort mNbByPix, float* mSom, float* mSomSqr)
{

    // point image
    const uint2     pt          =   make_uint2(blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);

    if(oSE(pt,cstP_CorMS._dimTerrain))
        return;

    // indice de l'etape sub pixelaire, le maximum �tant cPCencus.mNbByPix
    const ushort    etapeSub    =   (ushort)blockIdx.z;

    // la dimension du cache, la cache stocke des precaluls pour la corr�lation
    const uint3     dimCache    =   make_uint3(cstP_CorMS._dimTerrain.x,cstP_CorMS._dimTerrain.y,mNbByPix*cstP_CorMS.aNbScale);

    // le d�calage sub pixelaire
    const float     cStepPix    =   ((float)etapeSub)*aStepPix;

    // point de l'image pour cette etape sub pixelaire
    const float2    ptImage     =   make_float2((float)pt.x + cStepPix,(float)pt.y);

    float aGlobSom = 0;
    float aGlobSomSqr = 0;
    float aGlobPds  = 0;

    // pour toutes les echelles
    for (int aKS=0 ; aKS< cstP_CorMS.aNbScale ; aKS++)
    {
        float   aSom    = 0;
        float   aSomSqr = 0;
        short2 *aVP     = cstP_CorMS.aVV[aKS];
        ushort  aNbP    = cstP_CorMS.size_aVV[aKS];
        float   aPdsK   = cstP_CorMS.aVPds[aKS];

        // pour les �l�ments de la vignettes
        for (int aKP=0 ; aKP<aNbP ; aKP++)
        {
            const short2 aP = aVP[aKP];
            float aV = getValImage(ptImage+aP,idImage,aKS);
            aSom += aV;
            aSomSqr += aV*aV;
        }

        aGlobSom    += aSom     * aPdsK;
        aGlobSomSqr += aSomSqr  * aPdsK;
        aGlobPds    += aPdsK    * aNbP;

        // indice dans le cache
        const uint3     p3d        =   make_uint3(pt.x,pt.y,etapeSub*mNbByPix + aKS);

        // Ecriture dans le cache des
        mSom    [to1D(p3d,dimCache)] = aGlobSom    / aGlobPds;
        mSomSqr [to1D(p3d,dimCache)] = aGlobSomSqr / aGlobPds;

    }
}


// calcul rapide de la correlation multi-echelles centre sur une vignette
__device__
inline    float Quick_MS_CorrelBasic_Center(

    const int2 & aPG0,
    const int2 & aPG1,

//    float ***  aSom1,
//    float ***  aSom11,
//    float ***  aSom2,
//    float ***  aSom22,
    float*  aSom1,
    float*  aSom11,
    float*  aSom2,
    float*  aSom22,
    int     aPx2,
    bool    ModeMax,
    ushort  aPhase)
{
    float aMaxCor = -1;
    float aCovGlob = 0;
    float aPdsGlob = 0;


    // pt float dans l'image 1
    const float2      aFG1      =   f2X(cstP_CorMS.aStepPix*(float)aPhase + (float)dElise_div(aPx2,cstP_CorMS.mNbByPix))+  aPG1;

    int aNbScale = cstP_CorMS.aNbScale;
    for (int aKS=0 ; aKS< aNbScale ; aKS++)
    {
         bool   aLast   = (aKS==(aNbScale-1));
         short2*aVP     = cstP_CorMS.aVV[aKS];
         float  aPds    = cstP_CorMS.aVPds[aKS];
         float  aCov    = 0;
         ushort aNbP    = cstP_CorMS.size_aVV[aKS];

         aPdsGlob += aPds * aNbP;
         for (int aKP=0 ; aKP<aNbP ; aKP++)
         {
             const short2 aP = aVP[aKP];

             const float valima_0 = getValImage(aPG0 + aP,0,aKS);
             const float valima_1 = getValImage(aFG1 + aP,1,aKS);

             aCov += valima_0*valima_1;
         }

         aCovGlob += aCov * aPds;

         if (ModeMax || aLast)
         {
             const uint  pit0   =   to1D(make_uint3(aPG0.x,aPG0.y,aKS),cstP_CorMS._dimTerrain);
             const uint  pit1   =   to1D(make_uint3(aPG1.x,aPG1.y,aKS + aNbScale*aPhase),cstP_CorMS._dimTerrain);

             const float aM1    =   aSom1 [pit0];
             const float aM2    =   aSom2 [pit1];

             const float aM11   =   aSom11[pit0] - aM1*aM1;
             const float aM22   =   aSom22[pit1] - aM2*aM2;

             const float aM12   =   aCovGlob / aPdsGlob   - aM1 * aM2;

             if (ModeMax)
             {
                float aCor = (aM12 * abs(aM12)) /max(cstP_CorMS.anEpsilon,aM11*aM22);
                aMaxCor = max(aMaxCor,aCor);
             }
             else
                return aM12 / sqrt(max(cstP_CorMS.anEpsilon,aM11*aM22));
        }

    }
    return (aMaxCor > 0) ? sqrt(aMaxCor) : - sqrt(-aMaxCor) ;
}

__global__
void Kernel__DoCorrel_MultiScale_Global(float* aSom1,float*  aSom11,float* aSom2,float*  aSom22,short2 *nappe, float *cost)
{

    // ??? TODO � cabler
    bool    DoMixte     = false;
    bool    aModeMax    = true;
    float   aSeuilHC    = 1.0;
    float   aSeuilBC    = 1.0;

    // point image
    const   int2  an  =   make_int2(blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);

    // sortir si le point est en dehors du terrain
    if(oSE(an,cstP_CorMS._dimTerrain))
        return;

    //      pt int dans l'image 0
    const   int2     aPIm0       =   an + cstP_CorMS.anOff0;

    // si dans le masque de l'image 0
    const bool  OkIm0   =   IsOkErod(aPIm0,0);

    if (OkIm0)
    {

        // Z relatif au thread
        const ushort thZ   =   blockIdx.z*blockDim.z + threadIdx.z;

//        if(thZ+1 >= cstP_CorMS.maxDeltaZ)
//        {
//            return;
//        }

        // pitch de decalage
        const uint   pit    =   to1D(an,thZ,cstP_CorMS._dimTerrain);
        const uint   pit2d  =   to1D(an,cstP_CorMS._dimTerrain);

        float&          _cost   =  cost[pit];
        const short2    _nappe  =  nappe[pit2d];
        short           aZ0     =  _nappe.x;
        const int       DeltaZ  =  abs(_nappe.y-aZ0);

        if(thZ>=DeltaZ)
            return; // TODO on pourrait eventuellement affacter la valeur du cout par defaut.... mais bof

        // z Absolu
        const short aZ = (short)thZ + aZ0;

        // calcul de la phase
        // Attention probleme avec valeur negative et le modulo
        const ushort aPhase = (ushort)((abs((int)aZ))%cstP_CorMS.mNbByPix);

        /// peut etre precalcul  -- voir simplifier
        ///
//        while ((abs((int)aZ0))%cstP_CorMS.mNbByPix != aPhase)
//            aZ0++;

        int gpu_anOffset = dElise_div((int)aZ,cstP_CorMS.mNbByPix);


//        if( aEq(an,10) && aPhase == 0 && thZ < cstP_CorMS.mNbByPix)
//            DUMP(gpu_anOffset)

//int anOffset = dElise_div((int)aZ0,cstP_CorMS.mNbByPix);
       //
        const   int2     aIm1SsPx     =   an + cstP_CorMS.anOff1;
        //      pt int dans l'image 1
        const   int2     aPIm1        =   aIm1SsPx + i2X(gpu_anOffset);

        if (IsOkErod(aPIm1,1))
        {
            float aCost             = cstP_CorMS.mAhDefCost;
            float aGlobCostBasic    = 0;
            float aGlobCostCorrel   = 0;

            aCost = Quick_MS_CorrelBasic_Center(aPIm0,aPIm1,aSom1,aSom11,aSom2,aSom22,gpu_anOffset,aModeMax,aPhase);

            aGlobCostCorrel = aCost;

            if (DoMixte)
            {
               if(aGlobCostCorrel>aSeuilHC)

                    aCost = aGlobCostCorrel;

               else if (aGlobCostCorrel>aSeuilBC)
               {
                    float aPCor =  (aGlobCostCorrel - aSeuilBC) / (aSeuilHC-aSeuilBC);
                    aCost       =  aPCor * aGlobCostCorrel + (1-aPCor) * aSeuilBC *  aGlobCostBasic;
               }
               else
                    aCost =  aSeuilBC *  aGlobCostBasic;
            }

            _cost = 1.f-aCost;

        }   
    }
}
#include <stdio.h>
extern "C" void LaunchKernel__Correlation_MultiScale(dataCorrelMS &data,const_Param_Cor_MS &parCMS)
{
    // Cache device
    //CuUnifiedData3D<float>  aSom1;

    CuDeviceData3D<float>  aSom_0;
    CuDeviceData3D<float>  aSomSqr_0;

    CuDeviceData3D<float>  aSom_1;
    CuDeviceData3D<float>  aSomSqr_1;

    aSom_0   .Malloc (parCMS._dimTerrain,parCMS.aNbScale); //  pas de sous echantillonnage
    aSomSqr_0.Malloc (parCMS._dimTerrain,parCMS.aNbScale);

    aSom_1   .Malloc (parCMS._dimTerrain,parCMS.aNbScale*parCMS.mNbByPix); // avec sous echantillonnage
    aSomSqr_1.Malloc (parCMS._dimTerrain,parCMS.aNbScale*parCMS.mNbByPix);

    dim3	threads( 32, 32, 1);

    uint    divDTerX = iDivUp32(parCMS._dimTerrain.x);
    uint    divDTerY = iDivUp32(parCMS._dimTerrain.y);

    dim3	blocks_00(divDTerX,divDTerY, 1);
    dim3	blocks_01(divDTerX,divDTerY, parCMS.mNbByPix);

    /// Les donn�es sont structur�es par calques
    /// les echelles (du m�me subpixel) sont regroup�es par calques cons�cutifs
    KernelPrepareCorrel<<<blocks_00,threads>>>(0,1,1,aSom_0.pData(),aSomSqr_0.pData());
    KernelPrepareCorrel<<<blocks_01,threads>>>(1,parCMS.aStepPix,parCMS.mNbByPix,aSom_1.pData(),aSomSqr_1.pData());

    ushort  modThreadZ = 8;

    dim3	threads_CorMS( 32, 32, modThreadZ);

    uint    bC =  iDivUp(data._maxDeltaZ,modThreadZ);

    dim3    blocks__CorMS(divDTerX,divDTerY,bC);

    /// calcul des couts de correlation multi-echelles    

    Kernel__DoCorrel_MultiScale_Global<<<threads_CorMS,blocks__CorMS>>>(
                                                        aSom_0   .pData(),
                                                        aSomSqr_0.pData(),
                                                        aSom_1   .pData(),
                                                        aSomSqr_1.pData(),
                                                        data._uInterval_Z   .pData(),
                                                        data._uCost         .pData());

//    aSom1.syncHost();
//    aSom1.hostData.OutputValues();

    data._uCost.syncHost();

   data._uCost.hostData.OutputValues();

    aSom_0   .Dealloc();
    aSomSqr_0.Dealloc();
    aSom_1   .Dealloc();
    aSomSqr_1.Dealloc();

}
