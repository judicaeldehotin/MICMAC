#include "hip/hip_runtime.h"

#include "GpGpu/GpGpu_CommonHeader.h"
//#include "GpGpu/GpGpu_TextureTools.cuh"
#include "GpGpu/GpGpu_Interface_Census.h"

// Algorithme Correlation multi echelle sur ligne epipolaire

// Donn�es :
//  - 2 images avec diff�rents niveaux de floutage

//
// * Pr�-calcul et param�tres                                   |       GPU
// -------------------------------------------------------------|----------------------------------
// - Tableau de parcours des vignettes                          >>      constant 3d data short2
// - poids des echelles                                         >>      constant 2d data ???
// - Tableau du ZMin et ZMax de chaque coordonn�es terrain      >>      global 2D data short2
// - les offsets Terrain <--> Image Epi                         >>      constant 2 x int2
// - le masque erod� de l'image 1                               >>      1 texture pixel
// - 2 images x N echelles                                      >>      2 textures layered float


//
// Phase mNbByPix // pas utilise en GPU


/*  CPU
 *
 *  pour chaque
 *      - calcul des images interpol�es pour l'image 1
 *      - mise en vecteur des images interpol�es
 *      - Precalcul somme et somme quad
 *      - Parcour du terrain
 *      - Calcul des images interpol�
 *      - Parcours des Z
 *          - Calcul de la projection image 1
 *          - Calcul de la correlation Quick_MS_CorrelBasic_Center
 *              - pour chaque echelle
 *                  - Calcul de correlation
 *
 *      - set cost dans la matrice de regularisation
 */


///
static __constant__ constantParameterCensus     cParamCencus;

extern "C" void paramCencus2Device( constantParameterCensus &param )
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cParamCencus), &param, sizeof(constantParameterCensus)));
}

texture< float,	hipTextureType2DLayered >      texture_ImageEpi_00;
texture< float,	hipTextureType2DLayered >      texture_ImageEpi_01;
texture< pixel,	hipTextureType2DLayered >      Texture_Masq_Erod;

extern "C" textureReference& texture_ImageEpi(int nEpi){return nEpi == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01;}

extern "C" textureReference* pTexture_ImageEpi(int nEpi){return nEpi == 0 ? &texture_ImageEpi_00 : &texture_ImageEpi_01;}

extern "C" textureReference& texture_Masq_Erod(){return Texture_Masq_Erod;}

__device__
inline    bool GET_Val_BIT(const U_INT1 * aData,int anX)
{
    return (aData[anX/8] >> (7-anX %8) ) & 1;
}

__device__
inline    bool okErod(uint3 pt)
{
    // TODO peut etre simplifier % et division

    pixel mask8b = tex2DLayered(Texture_Masq_Erod,pt.x/8 + 0.5f,pt.y + 0.5f ,pt.z);

    return (mask8b >> (7-pt.x %8) ) & 1;
}

__global__ void projectionMasq(float * dataPixel,uint3 dTer)
{

    if(blockIdx.x > cParamCencus._dimTerrain.y || blockIdx.y > cParamCencus._dimTerrain.x)
        return;

    const uint3 pt = make_uint3(blockIdx.x,blockIdx.y,blockIdx.z);

    float valImage = tex2DLayered(pt.z == 0 ? texture_ImageEpi_00 : texture_ImageEpi_01 ,pt.x + 0.5f,pt.y + 0.5f ,0);

    dataPixel[to1D(pt,dTer)] = okErod(pt) ? valImage/(32768.f/4.f) : 0;
}

extern "C" void LaunchKernelCorrelationCensus(dataCorrelMS &data,constantParameterCensus &param)
{
    dim3	threads( 1, 1, 1);
    dim3	blocks(param._dimTerrain.y , param._dimTerrain.x, 2);

    CuHostData3D<float>     hData;
    CuDeviceData3D<float>   dData;

    uint3 dTer  = make_uint3(param._dimTerrain.y , param._dimTerrain.x,2);
    uint2 dTer2 = make_uint2(dTer);

    hData.Malloc(dTer2,2);
    dData.Malloc(dTer2,2);
    hData.Fill(0.f);
    dData.Memset(0);

    DUMP_INT2(param._offset0)
    DUMP_INT2(param._offset1 )

    projectionMasq<<<blocks, threads>>>(dData.pData(),dTer);

    dData.CopyDevicetoHost(hData);

    GpGpuTools::Array1DtoImageFile(hData.pData()    ,"ET_HOP_0.pmg",hData.GetDimension());
    GpGpuTools::Array1DtoImageFile(hData.pLData(1)  ,"ET_HOP_1.pmg",hData.GetDimension());
}
