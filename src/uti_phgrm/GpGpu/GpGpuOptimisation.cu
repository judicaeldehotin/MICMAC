#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_H_
/// \brief ....
#define _OPTIMISATION_KERNEL_H_

/// \file       GpGpuOptimisation.cu
/// \brief      Kernel optimisation
/// \author     GC
/// \version    0.01
/// \date       Avril 2013

#include "GpGpu/GpGpuStreamData.cuh"

/// brief Calcul le Z min et max.
__device__ void ComputeIntervaleDelta(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    aDz.x =   aZ_Prev.x-aZ;
    if (aZ != aZ_Next.x)
        aDz.x = max(aDz.x,-MaxDeltaZ);

    aDz.y = aZ_Prev.y-1-aZ;
    if (aZ != aZ_Next.y-1)
        aDz.y = min(aDz.y,MaxDeltaZ);

    if (aDz.x > aDz.y)
        if (aDz.y <0)
            aDz.x = aDz.y;
        else
            aDz.y = aDz.x;
}

template<class T, bool sens > __device__ void ReadOneSens(CDeviceDataStream<T> &costStream, uint lenghtLine, T pData[][NAPPEMAX], bool& idBuffer, T* gData, ushort penteMax, uint3 dimBlockTer)
{
    const ushort    tid     = threadIdx.x;

    for(int idParLine = 0; idParLine < lenghtLine;idParLine++)
    {
        const short2 uZ = costStream.read(pData[0],tid,sens,0);
        short z = uZ.x;

        while( z < uZ.y )
        {
            int Z       = z + tid - uZ.x;
            if(Z < NAPPEMAX )
                gData[idParLine * dimBlockTer.z + Z]    = pData[0][Z];
            z          += min(uZ.y - z,WARPSIZE);
        }
    }
}

template<class T, bool sens > __device__ void ScanOneSens(CDeviceDataStream<T> &costStream, uint lenghtLine, T pData[][NAPPEMAX], bool& idBuffer, T* g_ForceCostVol, ushort penteMax, uint& idStreamOut )
{
    const ushort    tid     = threadIdx.x;
    short2          uZ_Prev = costStream.read(pData[idBuffer],tid, sens,0);
    short           z       = uZ_Prev.x;
    __shared__ T    minCost;

    if(sens)
        while( z < uZ_Prev.y )
        {
            int Z       = z + tid - uZ_Prev.x;
            g_ForceCostVol[Z]    = pData[idBuffer][Z];
            z += min(uZ_Prev.y - z,WARPSIZE);
        }

    for(int idParLine = 1; idParLine < lenghtLine;idParLine++)
    {

        idStreamOut = idStreamOut + (sens? 1 : -1) * abs(uZ_Prev.y - uZ_Prev.x);

        const short2 uZ_Next = costStream.read(pData[2],tid,sens,0);
        short2 aDz;
        short z = uZ_Next.x;

        if(!tid) minCost = 1e9;

        while( z < uZ_Next.y )
        {
            int Z = z + tid;

            if( Z < uZ_Next.y)
            {
                ComputeIntervaleDelta(aDz,Z,penteMax,uZ_Next,uZ_Prev);
                T costMin   = 1e9;

                short ZId = Z - uZ_Next.x;

                T costInit  = ZId < NAPPEMAX ? pData[2][Z - uZ_Next.x] : 0;

                for(short i = aDz.x ; i <= aDz.y; i++)
                {
                    short idZprev = Z - uZ_Prev.x + i;
                    if(idZprev < NAPPEMAX)
                        costMin = min(costMin, costInit + pData[idBuffer][Z - uZ_Prev.x + i]);
                }

                if(ZId < NAPPEMAX)
                    pData[!idBuffer][Z - uZ_Next.x] = costMin;

                //int idGData     = (sens ? idParLine : lenghtLine -  idParLine - 1) * dimBlockTer.z + Z - uZ_Next.x;
                int idGData     = idStreamOut + Z - uZ_Next.x;
                int cost        = sens ? costMin : costMin + g_ForceCostVol[idGData] - costInit;

                g_ForceCostVol[idGData]  = cost;

                if(!sens)
                    atomicMin(&minCost,cost);

            }


            z += min(uZ_Next.y - z,WARPSIZE);
        }

        if(!sens)
        {
            z = uZ_Next.x;
            while( z < uZ_Next.y )
            {
                int Z = z + tid;
                //int idGData     = (lenghtLine -  idParLine - 1) * dimBlockTer.z + Z - uZ_Next.x;
                int idGData     = idStreamOut + Z - uZ_Next.x;
                g_ForceCostVol[idGData]  -= minCost;
                z += min(uZ_Next.y - z,WARPSIZE);
            }
        }

        idBuffer    = !idBuffer;
        uZ_Prev     = uZ_Next;
    }
}

template<class T> __global__ void kernelOptiOneDirection(T* gStream, short2* gStreamId, T* g_ForceCostVol, uint3* g_RecStrParam, uint penteMax)
{
    __shared__ T        bufferData[WARPSIZE];
    __shared__ short2   bufferIndex[WARPSIZE];
    __shared__ T        pdata[3][NAPPEMAX];
    __shared__ uint     pit_Id;
    __shared__ uint     pit_Stream;
    __shared__ uint     sizeLine;

    uint                idStreamOut;

    if(!threadIdx.x)
    {
        uint3 recStrParam   = g_RecStrParam[blockIdx.x];
        pit_Stream          = recStrParam.x;
        pit_Id              = recStrParam.y;
        sizeLine            = recStrParam.z;
    }

    __syncthreads();

    bool idBuf      =   false;

    CDeviceDataStream<T> costStream(bufferData, gStream + pit_Stream,bufferIndex, gStreamId + pit_Id, sizeLine * NAPPEMAX, sizeLine);

    ScanOneSens<T,eAVANT>   (costStream, sizeLine, pdata,idBuf,g_ForceCostVol + pit_Stream,penteMax, idStreamOut);
    ScanOneSens<T,eARRIERE> (costStream, sizeLine, pdata,idBuf,g_ForceCostVol + pit_Stream,penteMax, idStreamOut);

}

/// \brief Lance le kernel d optimisation pour une direction
template <class T> void LaunchKernelOptOneDirection(CuHostData3D<T> &hInputStream, CuHostData3D<short2> &hInputindex, uint nBLine, CuHostData3D<T> &h_ForceCostVol, CuHostData3D<uint3>  rStrPar)
{

    uint    deltaMax    =   3;

    uint    dimDeltaMax =   deltaMax * 2 + 1;
    dim3    Threads(32,1,1);
    dim3    Blocks(nBLine,1,1);

    float   hPen[PENALITE];
    ushort  hMapIndex[WARPSIZE];

    for(int i=0 ; i < WARPSIZE; i++)
        hMapIndex[i] = i / dimDeltaMax;

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //================ Copie des penalites dans le device ============================

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite),    hPen,       sizeof(float)   * PENALITE));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dMapIndex),   hMapIndex,  sizeof(ushort)  * WARPSIZE));

    //======================== Variables Device ======================================

    CuDeviceData3D<T>       d_InputStream    ( hInputStream.GetSize(),  "d_InputStream"  );
    CuDeviceData3D<short2>  d_InputIndex     ( hInputindex.GetSize(),   "d_InputIndex"   );
    CuDeviceData3D<uint3>   d_RecStrParam    ( rStrPar.GetSize(),       "d_InputStream"  );
    CuDeviceData3D<T>       d_ForceCostVol   ( hInputStream.GetSize(),  "d_ForceCostVol" );

    //  ============== Copie du volume de couts dans le device  ======================

    d_InputStream.CopyHostToDevice(  hInputStream.pData());
    d_InputIndex .CopyHostToDevice(  hInputindex .pData());
    d_RecStrParam.CopyHostToDevice(  rStrPar     .pData());

    //================================================================================

    kernelOptiOneDirection<T><<<Blocks,Threads>>>
                                                (
                                                    d_InputStream    .pData(),
                                                    d_InputIndex     .pData(),
                                                    d_ForceCostVol   .pData(),
                                                    d_RecStrParam    .pData(),
                                                    deltaMax
                                                    );

    getLastCudaError("kernelOptiOneDirection failed");

    //============================================================================--
    d_ForceCostVol.CopyDevicetoHost(h_ForceCostVol.pData());

    d_ForceCostVol  .Dealloc();
    d_InputStream   .Dealloc();
    d_InputIndex    .Dealloc();
    d_RecStrParam   .Dealloc();
}

/// \brief Appel exterieur du kernel d optimisation
extern "C" void OptimisationOneDirection(CuHostData3D<uint> &data,CuHostData3D<short2> &index, uint nBLine, CuHostData3D<uint> & h_ForceCostVol, CuHostData3D<uint3>  rStrPar)
{
    LaunchKernelOptOneDirection(data,index,nBLine, h_ForceCostVol, rStrPar);
}

/// \brief Appel exterieur du kernel
extern "C" void Launch()
{
    uint    prof        = 40;
    uint3   dimVolCost  = make_uint3(80,4,prof );

    CuHostData3D<uint>      H_StreamCost      ( make_uint3( dimVolCost.z, dimVolCost.y, dimVolCost.x) );
    CuHostData3D<uint>      H_ForceCostVol    ( make_uint3( dimVolCost.z, dimVolCost.y, dimVolCost.x) );
    CuHostData3D<short2>    H_StreamIndex     ( make_uint2( dimVolCost.y, dimVolCost.x ));

    H_StreamCost  .SetName("streamCost");
    H_StreamIndex .SetName("streamIndex");

    uint si = 0 , sizeStreamCost = 0;

    srand (time(NULL));

    for(int i = 0 ; i < dimVolCost.x ; i++)
    {
        int pit         = i     * dimVolCost.y;
        int pitLine     = pit   * dimVolCost.z;
        si =  sizeStreamCost = 0;

        while (si < dimVolCost.y){

            int min                         =  -CData<int>::GetRandomValue(prof / 4,prof / 2 -1);
            int max                         =   CData<int>::GetRandomValue(prof / 4,prof / 2 -1);
            int dim                         =   max - min + 1;
            H_StreamIndex[pit + si]         =   make_short2(min,max);

            for(int i = 0 ; i < dim; i++)
                H_StreamCost[pitLine + sizeStreamCost + i] = i+1;//CData<uint>::GetRandomValue(16,128);

            si++;
            sizeStreamCost += dim;

        }
    }

    int id = 0;
    H_StreamCost.OutputValues(id);

    //LaunchKernelOptOneDirection(H_StreamCost,H_StreamIndex,H_ForceCostVol);

    H_ForceCostVol.OutputValues(id);

    H_StreamCost.Dealloc();
    H_StreamIndex.Dealloc();
}

#endif
