#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_H_
/// \brief ....
#define _OPTIMISATION_KERNEL_H_

/// \file       GpGpuOptimisation.cu
/// \brief      Kernel optimisation
/// \author     GC
/// \version    0.01
/// \date       Avril 2013

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "GpGpu/GpGpuTools.h"
#include "GpGpu/helper_math_extented.cuh"

using namespace std;


/// \brief Tableau des penalites pre-calculees
#define PENALITE 7
#define WARPSIZE 32
#define NAPPEMAX 256

#define eAVANT      1
#define eARRIERE   -1

static __constant__ float   penalite[PENALITE];
static __constant__ ushort  dMapIndex[WARPSIZE];

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type

/// \struct SharedMemory
/// \brief  Structure de donnees partagees pour un block.
///         Allocation dynamique de la memoire lors du lancement du kernel
template<class T>
struct SharedMemory
{
    /// \brief ...
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    /// \brief ...
    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

/// \brief Opere une reduction d un tableau en Cpu
template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    return sum;
}

/// \brief Opere une reduction d un tableau en Gpu
template<class T> __global__ void kernelReduction(T* g_idata,T* g_odata,  int n)
{

    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockDim.x < n)
        mySum += g_idata[i+blockDim.x];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/// \brief  Fonction Gpu d optimisation
template<class T> __global__ void kernelOptiOneDirection(T* g_idata,T* g_odata,int* g_oPath, uint2 dimPlanCost, uint2 delta, float defaultValue)
{
    __shared__ T    sdata[32];

    const int   tid = threadIdx.x;
    const uint  pit = blockIdx.x * blockDim.x;
    uint        i0  = pit + tid;
    sdata[tid]      = g_idata[i0];
    bool        defV= sdata[tid] == defaultValue;
    g_odata[i0]     = defV ? 0 : sdata[tid];
    g_oPath[i0]     = tid;

    T minCost, cost;

    for(int l=1;l<dimPlanCost.y;l++)
    {
        uint        i1   = i0 + dimPlanCost.x;
        int         iL   = tid;

        if(i1<size(dimPlanCost))
        {
            cost = g_idata[i1];

            if(cost!=defaultValue)

                minCost = defV ? cost : cost + sdata[tid] + penalite[0];

            __syncthreads();

            if(cost!=defaultValue)
                for(int t = -((int)(delta.x)); t < ((int)(delta.y));t++)
                {
                    int Tl = tid + t;
                    if( t!=0 && Tl >= 0 && Tl < blockDim.x && sdata[Tl] != defaultValue)
                    {
                        T Cost = cost + sdata[Tl] + penalite[abs(t)];
                        if(Cost < minCost || defV)
                        {
                            minCost = Cost;
                            iL      = Tl;
                        }
                    }
                }

            else
                minCost = defV ? 0 : sdata[tid];

            i0 = l * dimPlanCost.x + pit + tid;

            g_odata[i0] = minCost;
            sdata[tid]  = minCost;
            defV        = minCost == defaultValue;
            g_oPath[i0] = iL;
        }
    }
}

/// brief Calcul le Z min et max.

__device__ void ComputeIntervaleDelta
(
        int & aDzMin,
        int & aDzMax,
        int aZ,
        int MaxDeltaZ,
        int aZ1Min,
        int aZ1Max,
        int aZ0Min,
        int aZ0Max
        )
{
    aDzMin =   aZ0Min-aZ;
    if (aZ != aZ1Min)
        aDzMin = max(aDzMin,-MaxDeltaZ);

    aDzMax = aZ0Max-1-aZ;
    if (aZ != aZ1Max-1)
        aDzMax = min(aDzMax,MaxDeltaZ);

    if (aDzMin > aDzMax)
        if (aDzMax <0)
            aDzMin = aDzMax;
        else
            aDzMax = aDzMin;
}


template< class T >
class CDeviceStream

{
public:

    __device__ CDeviceStream(T* buf,T* stream):
        _bufferData(buf),
        _streamData(stream),
        _curSteamId(0),
        _curBuffeId(WARPSIZE)
    {}

    __device__ virtual short getLengthRead(short2 &index)
    {
        index = make_short2(0,0);
        return 1;
    }

    __device__ short2 read(T* destData, ushort tid, short sens, T def, bool waitSync = true)
    {
        short2  index;
        ushort  NbCopied = 0 , NbTotalToCopy = getLengthRead(index);

        while(NbCopied < NbTotalToCopy)
        {
            ushort NbToCopy = min(NbTotalToCopy - NbCopied , WARPSIZE - _curBuffeId);

            if(NbToCopy == 0)
            {
                _bufferData[threadIdx.x] = _streamData[_curSteamId + threadIdx.x];
                _curBuffeId = 0;
                /*if (!tid)*/
                _curSteamId  += WARPSIZE;
                NbToCopy = min(NbTotalToCopy - NbCopied ,WARPSIZE);
                __syncthreads();
            }

            if (tid < NbToCopy)
                destData[NbCopied + tid] =  _bufferData[_curBuffeId + tid];
            else
                destData[NbCopied + tid] =  def;

            if(waitSync)
                __syncthreads();

            _curBuffeId += NbToCopy;
            NbCopied    += NbToCopy;
        }
       return index;
    }

private:

    T*                          _bufferData;
    T*                          _streamData;
    uint                        _curSteamId;
    ushort                      _curBuffeId;

};

template< class T >
class CDeviceDataStream : public CDeviceStream<T>
{
public:

    __device__ CDeviceDataStream(T* buf,T* stream,short2* bufId,short2* streamId):
        CDeviceStream<T>(buf,stream),
        _streamIndex(bufId,streamId)
    {}

    __device__ short getLengthRead(short2 &index)
    {
        _streamIndex.read(&index,0,0,make_short2(0,0),false);
        const short leng = diffYX(index) + 1;
        if(threadIdx.x ==0)printf("longueur %d\n",leng);
        return leng;
    }

private:
    CDeviceStream<short2>     _streamIndex;
};


template<class T> __device__ short2 readStream(T* destData,T* bufferData, short2* bufferIndex, T* streamData, short2* streamIndex, int tid, int& bufIdId, int& bufDaId, int& idCel, int& idStm, short sens)

{

    bufIdId ++;
    ushort elCopied = 0;

    if(bufIdId >= WARPSIZE)
    {
        int pit = idCel;
        bufferIndex[tid] = streamIndex[pit + tid];
        bufIdId = 0;
        __syncthreads();
    }

    const short2 Z      = bufferIndex[bufIdId];
    const ushort dimZ   = diffYX(Z);

    while(elCopied < dimZ)
    {
        ushort elToCopy = min(dimZ - elCopied , WARPSIZE - bufDaId);

        if(elToCopy == 0)
        {
            bufferData[tid] = streamData[idStm + tid];
            bufDaId = 0;

            if (!tid) idStm  += WARPSIZE;

            elToCopy = min(dimZ - elCopied ,WARPSIZE);
            __syncthreads();
        }

        destData[elCopied + tid] = (tid <= elToCopy) ? bufferData[bufDaId + tid] : -1;

        __syncthreads();
        bufDaId += elToCopy;
        elCopied  += elToCopy;
    }

    idCel++;

    return Z;
}

template<class T> __global__ void kernelOptiOneDirection2(T* gInputStream, short2* gInputIndex, T* g_odata, uint3 dimBlockTer, uint penteMax )

{

    __shared__ T        bufferData[WARPSIZE];
    __shared__ short2   bufferindex[WARPSIZE];
    __shared__ T        pdata[3][NAPPEMAX];

    int             idStm   =   0;
    const ushort    tid     =   threadIdx.x;
    int             bufIdId =   WARPSIZE;
    int             bufDaId =   WARPSIZE;
    int             idCel   =   0;
    bool            idBuf   =   0;
    const int       pit     =   blockIdx.x * dimBlockTer.y;
    const int       pitStr  =   pit * dimBlockTer.z;
    const int       pitId   =   pit * 2;

    short2 uZ_P = readStream(pdata[idBuf], bufferData , bufferindex , gInputStream + pitStr, gInputIndex + pitId, tid,bufIdId, bufDaId, idCel, idStm,1);

    g_odata[pitStr + tid] = pdata[idBuf][tid]; // ATTENTION  Faible bande passante

    for(int l=1;l<dimBlockTer.y;l++)
    {
        const short2 uZ_N = readStream(pdata[2],bufferData,bufferindex , gInputStream + pitStr, gInputIndex + pitId, tid,bufIdId, bufDaId, idCel, idStm,1);

        int aDzMin,aDzMax;
        short z = uZ_N.x;

        while( z < uZ_N.y )
        {
            int Z = z + tid;

            if( Z < uZ_N.y)
            {
                ComputeIntervaleDelta(aDzMin,aDzMax,Z,penteMax,uZ_N.x,uZ_N.y,uZ_P.x,uZ_P.y);
                int costMin = 1e9;
                for(int i = aDzMin ; i < aDzMax; i++)
                    costMin = min(costMin,pdata[2][Z - uZ_N.x] + pdata[idBuf][Z - uZ_P.x+ i]);

                pdata[!idBuf][Z - uZ_N.x]           = costMin;
                g_odata[pitStr + l*WARPSIZE + Z - uZ_N.x]    = costMin; // ATTENTION  Faible bande passante
            }

            z += min(uZ_N.y - z,WARPSIZE);
        }

        idBuf = !idBuf;
        uZ_P = uZ_N;
    }

}

template<class T> __global__ void kernelOptiOneDirection3(T* gStream, short2* gStreamId, T* g_odata, uint3 dimBlockTer, uint penteMax)
{

    __shared__ T        bufferData[WARPSIZE];
    __shared__ short2   bufferIndex[WARPSIZE];
    __shared__ T        pdata[3][NAPPEMAX];

    const ushort    tid     =   threadIdx.x;
    const int       pit     =   blockIdx.x * dimBlockTer.y;
    const int       pitStr  =   pit * dimBlockTer.z;
    bool            idBuf   =   false;

    CDeviceDataStream<T> costStream(bufferData, gStream + pitStr,bufferIndex, gStreamId + pit);

    short2 uZ_P = costStream.read(pdata[idBuf],tid, eAVANT,0);

    g_odata[tid] = pdata[idBuf][tid];

    __syncthreads();

    costStream.read(pdata[idBuf],tid,eAVANT,0);

    g_odata[WARPSIZE + tid] = pdata[idBuf][tid];

    __syncthreads();

    costStream.read(pdata[idBuf],tid,eAVANT,0);

    g_odata[WARPSIZE * 2+ tid] = pdata[idBuf][tid];

/*
    for(int l=1;l<dimBlockTer.y;l++)
    {
        const short2 uZ_N = costStream.read(pdata[2],tid,eAVANT);

        int aDzMin,aDzMax;
        short z = uZ_N.x;

        while( z < uZ_N.y )
        {
            int Z = z + tid;

            if( Z < uZ_N.y)
            {
                ComputeIntervaleDelta(aDzMin,aDzMax,Z,penteMax,uZ_N.x,uZ_N.y,uZ_P.x,uZ_P.y);
                int costMin = 1e9;
                for(int i = aDzMin ; i < aDzMax; i++)
                    costMin = min(costMin,pdata[2][Z - uZ_N.x] + pdata[idBuf][Z - uZ_P.x+ i]);

                pdata[!idBuf][Z - uZ_N.x]           = costMin;
                g_odata[pitStr + l*WARPSIZE + Z - uZ_N.x]    = costMin; // ATTENTION  Faible bande passante
            }

            z += min(uZ_N.y - z,WARPSIZE);
        }

        idBuf = !idBuf;
        uZ_P = uZ_N;
    }
*/
}

/// \brief Lance le kernel d optimisation pour une direction
template <class T> void LaunchKernelOptOneDirection2(CuHostData3D<T> &hInputStream, CuHostData3D<short2> &hInputindex, uint3 dimVolCost,float defaultValue, int sizeVolumeCost)
{

    int     nBLine      =   dimVolCost.x;
    int     si          =   dimVolCost.z * nBLine;
    int     dimLine     =   dimVolCost.y;
    uint2   diPlanCost  =   make_uint2(si,dimLine);
    uint    deltaMax    =   3;
    uint    dimDeltaMax =   deltaMax * 2 + 1;
    dim3    Threads(32,1,1);
    dim3    Blocks(nBLine,1,1);

    float   hPen[PENALITE];
    ushort  hMapIndex[WARPSIZE];


    for(int i=0 ; i < WARPSIZE; i++)
        hMapIndex[i] = i / dimDeltaMax;

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //-------- Copie des penalites dans le device ----------

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite), hPen, sizeof(float)*PENALITE));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dMapIndex), hMapIndex, sizeof(ushort)*WARPSIZE));

    //------------------------------------------------------

    uint2   sizeInput   =   make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y);
    uint2   sizeIndex   =   make_uint2(dimVolCost.y,dimVolCost.x);

    //----------- Declaration des variables Host -----------

    CuHostData3D<T>         hOutputValue(sizeInput,1);
    hOutputValue.SetName("hOutputValue");

    //----------------- Variables Device -------------------

    CuDeviceData3D<T>       dInputStream(sizeInput,1,"dInputStream");
    CuDeviceData3D<short2>     dInputIndex(sizeIndex,1,"dInputIndex");
    CuDeviceData3D<T>       dOutputData(sizeInput,1,"dOutputData");

    //--------- Initialisation des Variables Device ---------

    dOutputData.Memset(0); //???

    //------- Copie du volume de couts dans le device  -------

    dInputStream.CopyHostToDevice(hInputStream.pData());
    dInputIndex.CopyHostToDevice(hInputindex.pData());

    kernelOptiOneDirection3<T><<<Blocks,Threads>>>(dInputStream.pData(),dInputIndex.pData(),dOutputData.pData(),dimVolCost,deltaMax);
    getLastCudaError("kernelOptiOneDirection failed");

    dOutputData.CopyDevicetoHost(hOutputValue.pData());
    hipDeviceSynchronize();
    hOutputValue.OutputValues(0,XY,NEGARECT,3,-1);
//    hInputindex.OutputValues();

    dInputStream.Dealloc();
    dOutputData.Dealloc();

}


/// \brief Lance le kernel d optimisation pour une direction

template <class T> void LaunchKernelOptOneDirection(CuHostData3D<T> &hInputValue, uint3 dimVolCost,float defaultValue = 0)
{
    //nZ      = 32 doit etre en puissance de 2
    int     nBLine      =   dimVolCost.x;
    uint2   dimTer      =   make_uint2(dimVolCost.x,dimVolCost.y);
    int     si          =   dimVolCost.z * nBLine;
    int     dimLine     =   dimVolCost.y;
    uint2   diPlanCost  =   make_uint2(si,dimLine);
    uint2   delta       =   make_uint2(5);
    dim3    Threads(dimVolCost.z,1,1);
    dim3    Blocks(nBLine,1,1);

    float hPen[PENALITE];

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //-------- Copie des penalites dans le device ----------

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite), hPen, sizeof(float)*PENALITE));

    //----------- Declaration des variables Host -----------

    CuHostData3D<T>         hOutputValue(diPlanCost);
    CuHostData3D<int>       hPath(diPlanCost);
    CuHostData3D<float>     hMinCostId(dimTer);

    //----------------- Variables Device -------------------

    CuDeviceData3D<T>       dInputData(diPlanCost,1,"dInputData");
    CuDeviceData3D<T>       dOutputData(diPlanCost,1,"dOutputData");
    CuDeviceData3D<int>     dPath(diPlanCost,1,"dPath");
    CuDeviceData3D<float>   dMinCostId(make_uint2(dimVolCost.x,1),1,"minCostId");

    //--------- Initialisation des Variables Device ---------

    dOutputData.Memset(0);
    dPath.Memset(0);
    dMinCostId.Memset(0);

    //------- Copie du volume de couts dans le device  -------

    dInputData.CopyHostToDevice(hInputValue.pData());

    kernelOptiOneDirection<T><<<Blocks,Threads>>>(dInputData.pData(),dOutputData.pData(),dPath.pData(),diPlanCost, delta,/*dMinCostId.pData(),*/defaultValue);
    getLastCudaError("kernelOptimisation failed");

    dOutputData.CopyDevicetoHost(hOutputValue.pData());
    dPath.CopyDevicetoHost(hPath.pData());
    dMinCostId.CopyDevicetoHost(hMinCostId.pData());

/*

    uint2   ptTer;
    uint2   prev = make_uint2(0,1);
    for ( ptTer.x = 0; ptTer.x < dimTer.x; ptTer.x++)
        for(ptTer.y = 1; ptTer.y < dimTer.y ; ptTer.y++)
        {
            uint2 pt = make_uint2(ptTer.x * dimVolCost.z + (uint)hMinCostId[ptTer - prev],ptTer.y);
            hMinCostId[ptTer] =  (float)hPath[pt];
        }
    for (ptTer.x = 0; ptTer.x < dimTer.x; ptTer.x++)
        for(ptTer.y = 0; ptTer.y < dimTer.y ; ptTer.y++)
            if (defaultValue == hInputValue[ptTer])
                hMinCostId[ptTer] = 0.0f;
    hMinCostId.OutputValues();
    hInputValue.OutputValues(0,XY,Rect(0,0,32,dimVolCost.y));
    hPath.OutputValues(0,XY,Rect(0,0,dimVolCost.z,dimVolCost.y));
    hOutputValue.OutputValues(0,XY,Rect(0,0,dimVolCost.z,dimVolCost.y),4);
    GpGpuTools::Array1DtoImageFile(GpGpuTools::MultArray(hMinCostId.pData(),dimTer,1.0f/32.0f),"ZMap.pgm",dimTer);

*/

    hOutputValue.Dealloc();
    hPath.Dealloc();
    hMinCostId.Dealloc();
    dInputData.Dealloc();
    dOutputData.Dealloc();
    dPath.Dealloc();
    dMinCostId.Dealloc();

}

/// \brief Appel exterieur du kernel d optimisation
extern "C" void OptimisationOneDirection(CuHostData3D<float> &data, uint3 dimVolCost, float defaultValue)
{
    LaunchKernelOptOneDirection(data,dimVolCost,defaultValue);
}

/// \brief Appel exterieur du kernel
extern "C" void Launch()
{
    uint3 dimVolCost  = make_uint3(1,3,32);

    CuHostData3D<int>       streamCost(make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y));
    CuHostData3D<short2>    streamIndex(make_uint2(dimVolCost.y,dimVolCost.x));

    streamCost.SetName("streamCost");
    streamIndex.SetName("streamIndex");

    uint si = 0 , sizeStreamCost = 0;

    srand (time(NULL));

    for(int i = 0 ; i < dimVolCost.x ; i++)
    {
        int pit         = i * dimVolCost.y;
        int pitLine     = pit * dimVolCost.z;

        while (si < dimVolCost.y){

            int min                         =  -CData<int>::GetRandomValue(10,16);
            int max                         =   CData<int>::GetRandomValue(10,16);
            int dim                         =   max - min + 1;
            printf("Dim cpu  : %d\n",dim);
            streamIndex[pit + si]           =   make_short2(min,max);

            for(int i = 0 ; i < dim; i++)
                streamCost[pitLine + sizeStreamCost+i] = sizeStreamCost+i;//CData<int>::GetRandomValue(4,10);

            si++;
            sizeStreamCost += dim;

        }
    }
    streamCost.OutputValues();

    LaunchKernelOptOneDirection2(streamCost,streamIndex,dimVolCost,5.0f, sizeStreamCost);

    streamCost.Dealloc();
    streamIndex.Dealloc();

}

#endif
