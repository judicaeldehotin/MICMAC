#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_H_
/// \brief ....
#define _OPTIMISATION_KERNEL_H_

/// \file       GpGpuOptimisation.cu
/// \brief      Kernel optimisation
/// \author     GC
/// \version    0.01
/// \date       Avril 2013

#include "GpGpu/GpGpuStreamData.cuh"

/// brief Calcul le Z min et max.
__device__ void ComputeIntervaleDelta
(
        short2 & aDz,
        int aZ,
        int MaxDeltaZ,
        short2 aZ1,
        short2 aZ0
        )
{
    aDz.x =   aZ0.x-aZ;
    if (aZ != aZ1.x)
        aDz.x = max(aDz.x,-MaxDeltaZ);

    aDz.y = aZ0.y-1-aZ;
    if (aZ != aZ1.y-1)
        aDz.y = min(aDz.y,MaxDeltaZ);

    if (aDz.x > aDz.y)
        if (aDz.y <0)
            aDz.x = aDz.y;
        else
            aDz.y = aDz.x;
}

template<class T> __device__ void ScanOneSens(CDeviceDataStream<T> &costStream, bool sens, uint lenghtLine, T pData[][NAPPEMAX], bool& idBuffer, T* gData, ushort penteMax)
{
    const ushort    tid     =   threadIdx.x;

    short2 uZ_Prev = costStream.read(pData[idBuffer],tid, sens,0);

    for(int idCurLine = 1; idCurLine < lenghtLine;idCurLine++)
    {
        const short2 uZ_Next = costStream.read(pData[2],tid,sens,0);

        short2 aDz;
        short z = uZ_Next.x;

        while( z < uZ_Next.y )
        {
            int Z = z + tid;

            if( Z < uZ_Next.y)
            {
                ComputeIntervaleDelta(aDz,Z,penteMax,uZ_Next,uZ_Prev);
                int costMin = 1e9;
                for(int i = aDz.x ; i < aDz.y; i++)
                    costMin = min(costMin,pData[2][Z - uZ_Next.x] + pData[idBuffer][Z - uZ_Prev.x+ i]);

                pData[!idBuffer][Z - uZ_Next.x] = costMin;
                gData[costStream.getStartIndex() + idCurLine * WARPSIZE + Z - uZ_Next.x] = costMin;
            }

            z += min(uZ_Next.y - z,WARPSIZE);
        }

        idBuffer = !idBuffer;
        uZ_Prev = uZ_Next;
    }

}

template<class T> __global__ void kernelOptiOneDirection(T* gStream, short2* gStreamId, T* g_odata, uint3 dimBlockTer, uint penteMax)
{
    __shared__ T        bufferData[WARPSIZE];
    __shared__ short2   bufferIndex[WARPSIZE];
    __shared__ T        pdata[3][NAPPEMAX];

    const int       pit     =   blockIdx.x * dimBlockTer.y;
    const int       pitStr  =   pit * dimBlockTer.z;
    bool            idBuf   =   false;

    CDeviceDataStream<T> costStream(bufferData, gStream,bufferIndex, gStreamId + pit,pitStr);

    ScanOneSens<T>(costStream,eAVANT,dimBlockTer.y, pdata,idBuf,g_odata,penteMax);
    ScanOneSens<T>(costStream,eARRIERE,dimBlockTer.y, pdata,idBuf,g_odata,penteMax);
}

/// \brief Lance le kernel d optimisation pour une direction
template <class T> void LaunchKernelOptOneDirection(CuHostData3D<T> &hInputStream, CuHostData3D<short2> &hInputindex, uint3 dimVolCost)
{

    int     nBLine      =   dimVolCost.x;
    uint    deltaMax    =   3;
    uint    dimDeltaMax =   deltaMax * 2 + 1;
    dim3    Threads(32,1,1);
    dim3    Blocks(nBLine,1,1);

    float   hPen[PENALITE];
    ushort  hMapIndex[WARPSIZE];

    for(int i=0 ; i < WARPSIZE; i++)
        hMapIndex[i] = i / dimDeltaMax;

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //---------------------- Copie des penalites dans le device --------------------------------------- //
                                                                                                        //
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite), hPen, sizeof(float)*PENALITE));                        //
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dMapIndex), hMapIndex, sizeof(ushort)*WARPSIZE));                 //
                                                                                                        //
    //------------------------------------------------------------------------------------------------- //

    uint2   sizeInput   =   make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y);
    uint2   sizeIndex   =   make_uint2(dimVolCost.y,dimVolCost.x);

    //---------------------------- Declaration des variables Host -------------------------------------- //

    CuHostData3D<T> hOutputValue(sizeInput,1);
    hOutputValue.SetName("hOutputValue");

    //----------------- Variables Device -------------------------------------------------------------- //

    CuDeviceData3D<T>       dInputStream(sizeInput,1,"dInputStream");
    CuDeviceData3D<short2>     dInputIndex(sizeIndex,1,"dInputIndex");
    CuDeviceData3D<T>       dOutputData(sizeInput,1,"dOutputData");

    //--------- Initialisation des Variables Device ---------------------------------------------------- //

    dOutputData.Memset(0); //???

    //------- Copie du volume de couts dans le device  ------------------------------------------------- //

    dInputStream.CopyHostToDevice(hInputStream.pData());
    dInputIndex.CopyHostToDevice(hInputindex.pData());

    //------------------------------------------------------------------------------------------------- //

    kernelOptiOneDirection<T><<<Blocks,Threads>>>(dInputStream.pData(),dInputIndex.pData(),dOutputData.pData(),dimVolCost,deltaMax);
    getLastCudaError("kernelOptiOneDirection failed");

    //------------------------------------------------------------------------------------------------- //

    dOutputData.CopyDevicetoHost(hOutputValue.pData());
    //hipDeviceSynchronize();
    hOutputValue.OutputValues(0,XY,NEGARECT,3,-1);
    dInputStream.Dealloc();
    dOutputData.Dealloc();
}

/// \brief Appel exterieur du kernel d optimisation
extern "C" void OptimisationOneDirection(CuHostData3D<float> &data, uint3 dimVolCost, float defaultValue)
{
    //LaunchKernelOptOneDirection(data,dimVolCost,defaultValue);
}

/// \brief Appel exterieur du kernel
extern "C" void Launch()
{
    uint3 dimVolCost  = make_uint3(1,10,32);

    CuHostData3D<int>       streamCost(make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y));
    CuHostData3D<short2>    streamIndex(make_uint2(dimVolCost.y,dimVolCost.x));

    streamCost.SetName("streamCost");
    streamIndex.SetName("streamIndex");

    uint si = 0 , sizeStreamCost = 0;

    srand (time(NULL));

    for(int i = 0 ; i < dimVolCost.x ; i++)
    {
        int pit         = i * dimVolCost.y;
        int pitLine     = pit * dimVolCost.z;

        while (si < dimVolCost.y){

            int min                         =  -CData<int>::GetRandomValue(5,16);
            int max                         =   CData<int>::GetRandomValue(5,16);
            int dim                         =   max - min + 1;            
            streamIndex[pit + si]           =   make_short2(min,max);

            for(int i = 0 ; i < dim; i++)
                streamCost[pitLine + sizeStreamCost+i] =  CData<int>::GetRandomValue(16,128);

            si++;
            sizeStreamCost += dim;

        }
    }
    //streamCost.OutputValues();
    LaunchKernelOptOneDirection(streamCost,streamIndex,dimVolCost);

    streamCost.Dealloc();
    streamIndex.Dealloc();
}

#endif
