#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_H_
/// \brief ....
#define _OPTIMISATION_KERNEL_H_

/// \file       GpGpuOptimisation.cu
/// \brief      Kernel optimisation
/// \author     GC
/// \version    0.01
/// \date       Avril 2013

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "GpGpu/GpGpuTools.h"
#include "GpGpu/helper_math_extented.cuh"

using namespace std;


/// \brief Tableau des penalites pre-calculees
#define PENALITE 7
#define WARPSIZE 32
#define NAPPEMAX 256

static __constant__ float   penalite[PENALITE];
static __constant__ ushort  dMapIndex[WARPSIZE];

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type

/// \struct SharedMemory
/// \brief  Structure de donnees partagees pour un block.
///         Allocation dynamique de la memoire lors du lancement du kernel
template<class T>
struct SharedMemory
{
    /// \brief ...
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    /// \brief ...
    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

/// \brief Opere une reduction d un tableau en Cpu
template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    return sum;
}

/// \brief Opere une reduction d un tableau en Gpu
template<class T> __global__ void kernelReduction(T* g_idata,T* g_odata,  int n)
{

    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockDim.x < n)
        mySum += g_idata[i+blockDim.x];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/// \brief  Fonction Gpu d optimisation
template<class T> __global__ void kernelOptiOneDirection(T* g_idata,T* g_odata,int* g_oPath, uint2 dimPlanCost, uint2 delta, float defaultValue)
{
    __shared__ T    sdata[32];

    const int   tid = threadIdx.x;
    const uint  pit = blockIdx.x * blockDim.x;
    uint        i0  = pit + tid;
    sdata[tid]      = g_idata[i0];
    bool        defV= sdata[tid] == defaultValue;
    g_odata[i0]     = defV ? 0 : sdata[tid];
    g_oPath[i0]     = tid;

    T minCost, cost;

    for(int l=1;l<dimPlanCost.y;l++)
    {
        uint        i1   = i0 + dimPlanCost.x;
        int         iL   = tid;

        if(i1<size(dimPlanCost))
        {
            cost = g_idata[i1];

            if(cost!=defaultValue)

                minCost = defV ? cost : cost + sdata[tid] + penalite[0];

            __syncthreads();

            if(cost!=defaultValue)
                for(int t = -((int)(delta.x)); t < ((int)(delta.y));t++)
                {
                    int Tl = tid + t;
                    if( t!=0 && Tl >= 0 && Tl < blockDim.x && sdata[Tl] != defaultValue)
                    {
                        T Cost = cost + sdata[Tl] + penalite[abs(t)];
                        if(Cost < minCost || defV)
                        {
                            minCost = Cost;
                            iL      = Tl;
                        }
                    }
                }

            else
                minCost = defV ? 0 : sdata[tid];

            i0 = l * dimPlanCost.x + pit + tid;

            g_odata[i0] = minCost;
            sdata[tid]  = minCost;
            defV        = minCost == defaultValue;
            g_oPath[i0] = iL;
        }
    }
}

/// brief Calcul le Z min et max.
__device__ void ComputeIntervaleDelta
(
        int & aDzMin,
        int & aDzMax,
        int aZ,
        int MaxDeltaZ,
        int aZ1Min,
        int aZ1Max,
        int aZ0Min,
        int aZ0Max
        )
{
    aDzMin =   aZ0Min-aZ;
    if (aZ != aZ1Min)
        aDzMin = max(aDzMin,-MaxDeltaZ);

    aDzMax = aZ0Max-1-aZ;
    if (aZ != aZ1Max-1)
        aDzMax = min(aDzMax,MaxDeltaZ);

    if (aDzMin > aDzMax)
        if (aDzMax <0)
            aDzMin = aDzMax;
        else
            aDzMax = aDzMin;

}

template<class T> __device__ short2 readStream(T* destData,T* bufferData, int* bufferIndex, T* streamData, int* streamIndex, int tid, int& bufIdId, int& bufDaId, int& idCel, int& idStm, short sens)
{

    bufIdId += 2;
    ushort elCopied = 0;

    if(bufIdId >= WARPSIZE)
    {
        int pit = idCel*2;
        bufferIndex[tid] = streamIndex[pit + tid];

        bufIdId = 0;
        __syncthreads();
    }

    const short Z0      = bufferIndex[bufIdId];
    const short Z1      = bufferIndex[bufIdId+1];
    const ushort dimZ   = Z1 - Z0;

    while(elCopied < dimZ)
    {
        ushort elToCopy = min(dimZ - elCopied , WARPSIZE - bufDaId);

        if(elToCopy == 0)
        {
            bufferData[tid] = streamData[idStm + tid];
            bufDaId = 0;


            if (!tid) idStm  += WARPSIZE;

            elToCopy = min(dimZ - elCopied ,WARPSIZE);
            __syncthreads();
        }

        destData[elCopied + tid] = (tid <= elToCopy) ? bufferData[bufDaId + tid] : -1;

        __syncthreads();
        bufDaId += elToCopy;
        elCopied  += elToCopy;
    }

    idCel++;

    return make_short2(Z0,Z1);
}

template<class T> __global__ void kernelOptiOneDirection2(T* gInputStream, int* gInputIndex, T* g_odata, uint3 dimBlockTer, uint penteMax, float defaultValue)
{

    __shared__ T    bufferData[WARPSIZE];
    __shared__ T    bufferindex[WARPSIZE];
    __shared__ T    pdata[3][NAPPEMAX];
    int             idStm   =   0;
    const ushort    tid     =   threadIdx.x;
    int             bufIdId =   WARPSIZE;
    int             bufDaId =   WARPSIZE;
    int             idCel   =   0;
    bool            idBuf   =   0;
    const int       pit     =   blockIdx.x * dimBlockTer.y;
    const int       pitStr  =   pit * dimBlockTer.z;
    const int       pitId   =   pit * 2;

    short2 uZ_P = readStream(pdata[idBuf], bufferData , bufferindex , gInputStream + pitStr, gInputIndex + pitId, tid,bufIdId, bufDaId, idCel, idStm,1);

    g_odata[pitStr + tid] = pdata[idBuf][tid]; // ATTENTION  Faible bande passante

    for(int l=1;l<dimBlockTer.y;l++)
    {
        const short2 uZ_N = readStream(pdata[2],bufferData,bufferindex , gInputStream + pitStr, gInputIndex + pitId, tid,bufIdId, bufDaId, idCel, idStm,1);

        int aDzMin,aDzMax;
        short z = uZ_N.x;

        while( z < uZ_N.y )
        {
            int Z = z + tid;

            if( Z < uZ_N.y)
            {
                ComputeIntervaleDelta(aDzMin,aDzMax,Z,penteMax,uZ_N.x,uZ_N.y,uZ_P.x,uZ_P.y);
                int costMin = 1e9;
                for(int i = aDzMin ; i < aDzMax; i++)
                    costMin = min(costMin,pdata[2][Z - uZ_N.x] + pdata[idBuf][Z - uZ_P.x+ i]);

                pdata[!idBuf][Z - uZ_N.x]           = costMin;
                g_odata[pitStr + l*WARPSIZE + Z - uZ_N.x]    = costMin; // ATTENTION  Faible bande passante
            }

            z += min(uZ_N.y - z,WARPSIZE);
        }

        idBuf = !idBuf;
        uZ_P = uZ_N;
    }

}

/// \brief Lance le kernel d optimisation pour une direction
template <class T> void LaunchKernelOptOneDirection2(CuHostData3D<T> &hInputStream, CuHostData3D<int> &hInputindex, uint3 dimVolCost,float defaultValue, int sizeVolumeCost)
{

    int     nBLine      =   dimVolCost.x;
    int     si          =   dimVolCost.z * nBLine;
    int     dimLine     =   dimVolCost.y;
    uint2   diPlanCost  =   make_uint2(si,dimLine);
    uint    deltaMax    =   3;
    uint    dimDeltaMax =   deltaMax * 2 + 1;
    dim3    Threads(32,1,1);
    dim3    Blocks(nBLine,1,1);

    float   hPen[PENALITE];
    ushort  hMapIndex[WARPSIZE];


    for(int i=0 ; i < WARPSIZE; i++)
        hMapIndex[i] = i / dimDeltaMax;

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //-------- Copie des penalites dans le device ----------

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite), hPen, sizeof(float)*PENALITE));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dMapIndex), hMapIndex, sizeof(ushort)*WARPSIZE));

    //------------------------------------------------------

    uint2   sizeInput   =   make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y);
    uint2   sizeIndex   =   make_uint2(dimVolCost.y*2,dimVolCost.x);

    //----------- Declaration des variables Host -----------

    CuHostData3D<T>         hOutputValue(sizeInput,1);
    hOutputValue.SetName("hOutputValue");

    //----------------- Variables Device -------------------

    CuDeviceData3D<T>       dInputStream(sizeInput,1,"dInputStream");
    CuDeviceData3D<int>     dInputIndex(sizeIndex,1,"dInputIndex");
    CuDeviceData3D<T>       dOutputData(sizeInput,1,"dOutputData");

    //--------- Initialisation des Variables Device ---------

    dOutputData.Memset(0); //???

    //------- Copie du volume de couts dans le device  -------

    dInputStream.CopyHostToDevice(hInputStream.pData());
    dInputIndex.CopyHostToDevice(hInputindex.pData());

    kernelOptiOneDirection2<T><<<Blocks,Threads>>>(dInputStream.pData(),dInputIndex.pData(),dOutputData.pData(),dimVolCost, deltaMax,defaultValue);
    getLastCudaError("kernelOptiOneDirection failed");

    dOutputData.CopyDevicetoHost(hOutputValue.pData());
    hipDeviceSynchronize();
//    hOutputValue.OutputValues(0,XY,NEGARECT,3,-1);
//    hInputindex.OutputValues();

    dInputStream.Dealloc();
    dOutputData.Dealloc();

}


/// \brief Lance le kernel d optimisation pour une direction

template <class T> void LaunchKernelOptOneDirection(CuHostData3D<T> &hInputValue, uint3 dimVolCost,float defaultValue = 0)
{
    //nZ      = 32 doit etre en puissance de 2
    int     nBLine      =   dimVolCost.x;
    uint2   dimTer      =   make_uint2(dimVolCost.x,dimVolCost.y);
    int     si          =   dimVolCost.z * nBLine;
    int     dimLine     =   dimVolCost.y;
    uint2   diPlanCost  =   make_uint2(si,dimLine);
    uint2   delta       =   make_uint2(5);
    dim3    Threads(dimVolCost.z,1,1);
    dim3    Blocks(nBLine,1,1);

    float hPen[PENALITE];

    for(int i=0;i<PENALITE;i++)
        hPen[i] = ((float)(1 / 10.0f));

    //-------- Copie des penalites dans le device ----------

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(penalite), hPen, sizeof(float)*PENALITE));

    //----------- Declaration des variables Host -----------

    CuHostData3D<T>         hOutputValue(diPlanCost);
    CuHostData3D<int>       hPath(diPlanCost);
    CuHostData3D<float>     hMinCostId(dimTer);

    //----------------- Variables Device -------------------

    CuDeviceData3D<T>       dInputData(diPlanCost,1,"dInputData");
    CuDeviceData3D<T>       dOutputData(diPlanCost,1,"dOutputData");
    CuDeviceData3D<int>     dPath(diPlanCost,1,"dPath");
    CuDeviceData3D<float>   dMinCostId(make_uint2(dimVolCost.x,1),1,"minCostId");

    //--------- Initialisation des Variables Device ---------

    dOutputData.Memset(0);
    dPath.Memset(0);
    dMinCostId.Memset(0);

    //------- Copie du volume de couts dans le device  -------

    dInputData.CopyHostToDevice(hInputValue.pData());

    kernelOptiOneDirection<T><<<Blocks,Threads>>>(dInputData.pData(),dOutputData.pData(),dPath.pData(),diPlanCost, delta,/*dMinCostId.pData(),*/defaultValue);
    getLastCudaError("kernelOptimisation failed");

    dOutputData.CopyDevicetoHost(hOutputValue.pData());
    dPath.CopyDevicetoHost(hPath.pData());
    dMinCostId.CopyDevicetoHost(hMinCostId.pData());

/*

    uint2   ptTer;
    uint2   prev = make_uint2(0,1);
    for ( ptTer.x = 0; ptTer.x < dimTer.x; ptTer.x++)
        for(ptTer.y = 1; ptTer.y < dimTer.y ; ptTer.y++)
        {
            uint2 pt = make_uint2(ptTer.x * dimVolCost.z + (uint)hMinCostId[ptTer - prev],ptTer.y);
            hMinCostId[ptTer] =  (float)hPath[pt];
        }
    for (ptTer.x = 0; ptTer.x < dimTer.x; ptTer.x++)
        for(ptTer.y = 0; ptTer.y < dimTer.y ; ptTer.y++)
            if (defaultValue == hInputValue[ptTer])
                hMinCostId[ptTer] = 0.0f;
    hMinCostId.OutputValues();
    hInputValue.OutputValues(0,XY,Rect(0,0,32,dimVolCost.y));
    hPath.OutputValues(0,XY,Rect(0,0,dimVolCost.z,dimVolCost.y));
    hOutputValue.OutputValues(0,XY,Rect(0,0,dimVolCost.z,dimVolCost.y),4);
    GpGpuTools::Array1DtoImageFile(GpGpuTools::MultArray(hMinCostId.pData(),dimTer,1.0f/32.0f),"ZMap.pgm",dimTer);

*/

    hOutputValue.Dealloc();
    hPath.Dealloc();
    hMinCostId.Dealloc();
    dInputData.Dealloc();
    dOutputData.Dealloc();
    dPath.Dealloc();
    dMinCostId.Dealloc();

}

/// \brief Appel exterieur du kernel d optimisation
extern "C" void OptimisationOneDirection(CuHostData3D<float> &data, uint3 dimVolCost, float defaultValue)
{
    LaunchKernelOptOneDirection(data,dimVolCost,defaultValue);
}

/// \brief Appel exterieur du kernel
extern "C" void Launch()
{
    uint3 dimVolCost  = make_uint3(128,256,32);

    CuHostData3D<int> streamCost(make_uint2(dimVolCost.x * dimVolCost.z,dimVolCost.y));
    CuHostData3D<int> streamIndex(make_uint2(dimVolCost.y*2,dimVolCost.x));

    streamCost.SetName("streamCost");
    streamIndex.SetName("streamIndex");

    uint si = 0 , sizeStreamCost = 0;

    srand (time(NULL));

    for(int i = 0 ; i < dimVolCost.x ; i++)
    {
        int pit         = i * dimVolCost.y;
        int pitLine     = pit * dimVolCost.z;
        int pitIndex    = pit * 2;

        while (si < dimVolCost.y){

            int min                         =  -CData<int>::GetRandomValue(10,16);
            int max                         =   CData<int>::GetRandomValue(10,16);
            int dim                         =   max - min + 1;
            streamIndex[pitIndex + si*2]    =   min;
            streamIndex[pitIndex + si*2+1]  =   max;

            for(int i = 0 ; i < dim; i++)
                streamCost[pitLine + sizeStreamCost+i] = CData<int>::GetRandomValue(4,10);

            si++;
            sizeStreamCost += dim;

        }
    }
    //    streamCost.OutputValues();

    LaunchKernelOptOneDirection2(streamCost,streamIndex,dimVolCost,5.0f, sizeStreamCost);

    streamCost.Dealloc();
    streamIndex.Dealloc();

}

#endif
