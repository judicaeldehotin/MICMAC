#include "hip/hip_runtime.h"
#include "gpu/cudaAppliMicMac.cuh"

#include <iostream>
#include <string>
using namespace std;

#ifdef _WIN32
  #include <windows.h>
  #include <Lmcons.h>
#endif

#ifdef _DEBUG
	#define   BLOCKDIM	16
	#define   SBLOCKDIM 10
#else
	#define   BLOCKDIM	32
	#define   SBLOCKDIM 16
#endif

/*
//------------------------------------------------------------------------------------------
// Non utilis�
texture<float, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Image;
texture<bool, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Cache;
texture<float2, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Project;
hipArray* dev_Img;				// Tableau des valeurs de l'image
hipArray* dev_CubeProjImg;		// Declaration du cube de projection pour le device
hipArray* dev_ArrayProjImg;	// Declaration du tableau de projection pour le device
//------------------------------------------------------------------------------------------
*/
//------------------------------------------------------------------------------------------
// ATTENTION : erreur de compilation avec l'option hipReadModeNormalizedFloat
// et l'utilisation de la fonction tex2DLayered
texture< bool,	hipTextureType2D >			TexMaskTer;
texture< float2,hipTextureType2DLayered >	TexLay_Proj;
texture< float,	hipTextureType2DLayered >	refTex_ImagesLayered;
hipArray* dev_ImgLd;		//
hipArray* dev_ProjLr;		//
hipArray* dev_MaskTer;		//

//------------------------------------------------------------------------------------------
//float*	host_SimpCor;
float*	host_Cache;
//float*	dev_SimpCor;
float*	dev_Cost;
float*	dev_Cache;
float*	dev_NbImgOk;

paramGPU h;

extern "C" void allocMemory(void)
{

	//std::cout << "allocMemory\n";

	if (dev_NbImgOk	!= NULL) checkCudaErrors( hipFree(dev_NbImgOk));
	//if (dev_SimpCor != NULL) checkCudaErrors( hipFree(dev_SimpCor));
	if (dev_Cache	!= NULL) checkCudaErrors( hipFree(dev_Cache));
	if (dev_Cost	!= NULL) checkCudaErrors( hipFree(dev_Cost));


	//int sCorMemSize = h.sizeTer * sizeof(float);
	int costMemSize = h.rSiTer	* sizeof(float);
	int nBI_MemSize = h.rSiTer	* sizeof(float);
	int cac_MemSize = h.sizeCach* sizeof(float)* h.nLayer;
	
	// Allocation m�moire
	//host_SimpCor	= (float*)	malloc(sCorMemSize);
	host_Cache		= (float*)	malloc(cac_MemSize);
	
	//checkCudaErrors( hipMalloc((void **) &dev_SimpCor	, sCorMemSize) );	
	checkCudaErrors( hipMalloc((void **) &dev_Cache	, cac_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_NbImgOk	, nBI_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_Cost		, costMemSize ) );

	// Texture des projections
	TexLay_Proj.addressMode[0]	= hipAddressModeClamp;
	TexLay_Proj.addressMode[1]	= hipAddressModeClamp;	
	TexLay_Proj.filterMode		= hipFilterModePoint; //hipFilterModePoint hipFilterModeLinear
	TexLay_Proj.normalized		= true;

}

extern "C" paramGPU updateSizeBlock(  uint2 ter0, uint2 ter1 )
{

	uint oldSizeTer = h.sizeTer;

	h.pUTer0.x	= (int)ter0.x - (int)h.rVig.x;
	h.pUTer0.y	= (int)ter0.y - (int)h.rVig.y;
	h.pUTer1.x	= (int)ter1.x + (int)h.rVig.x;
	h.pUTer1.y	= (int)ter1.y + (int)h.rVig.y;
	
	h.rDiTer	= make_uint2(ter1.x - ter0.x, ter1.y - ter0.y);
	h.dimTer	= make_uint2(h.pUTer1.x - h.pUTer0.x, h.pUTer1.y - h.pUTer0.y);
	h.dimSTer	= iDivUp(h.dimTer,h.sampTer);	// Dimension du bloque terrain sous echantillon�
	h.sizeTer	= size(h.dimTer);				// Taille du bloque terrain
	h.sizeSTer  = size(h.dimSTer);				// Taille du bloque terrain sous echantillon�
	h.rSiTer	= size(h.rDiTer);

	h.dimCach	= h.rDiTer * h.dimVig;
	h.sizeCach	= size(h.dimCach);
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cH), &h, sizeof(paramGPU)));

	if (oldSizeTer < h.sizeTer)
		allocMemory();

	return h;
}

static void correlOptionsGPU( uint2 ter0, uint2 ter1, uint2 dV,uint2 dRV, uint2 dI, float mAhEpsilon, uint samplingZ, float uvDef, uint nLayer )
{

	h.nLayer	= nLayer;
	h.dimVig	= dV;							// Dimension de la vignette
	h.dimImg	= dI;							// Dimension des images
	h.rVig		= dRV;							// Rayon de la vignette
	h.sizeVig	= size(dV);						// Taille de la vignette en pixel 
	h.sampTer	= samplingZ;					// Pas echantillonage du terrain
	h.UVDefValue= uvDef;						// UV Terrain incorrect
	h.badVig	= -4.0f;
	h.mAhEpsilon= mAhEpsilon;

	updateSizeBlock( ter0, ter1 );
}

extern "C" void imagesToLayers(float *fdataImg1D, uint2 dimImage, int nbLayer)
{

	//std::cout << "imagesToLayers\n";
	hipExtent sizeImgsLay = make_hipExtent( dimImage.x, dimImage.y, nbLayer );

	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// Allocation memoire GPU du tableau des calques d'images
	checkCudaErrors( hipMalloc3DArray(&dev_ImgLd,&channelDesc,sizeImgsLay,hipArrayLayered) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms	p	= { 0 };
	hipPitchedPtr		pit = make_hipPitchedPtr(fdataImg1D, sizeImgsLay.width * sizeof(float), sizeImgsLay.width, sizeImgsLay.height);

	p.dstArray	= dev_ImgLd;		// Pointeur du tableau de destination
	p.srcPtr	= pit;						// Pitch
	p.extent	= sizeImgsLay;				// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des images du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

	// Li� � la texture
	refTex_ImagesLayered.addressMode[0]	= hipAddressModeWrap;
    refTex_ImagesLayered.addressMode[1]	= hipAddressModeWrap;
    refTex_ImagesLayered.filterMode		= hipFilterModePoint; //hipFilterModeLinear hipFilterModePoint
    refTex_ImagesLayered.normalized		= true;
	checkCudaErrors( hipBindTextureToArray(refTex_ImagesLayered,dev_ImgLd) );

};

extern "C" void  allocMemoryTabProj(uint2 dimTer, int nbLayer)
{

	//std::cout << "projectionsToLayers\n";
	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Taille du tableau des calques 
	hipExtent siz_PL = make_hipExtent( dimTer.x, dimTer.y, nbLayer);

	// Allocation memoire GPU du tableau des calques de projections
	if (dev_ProjLr != NULL) hipFreeArray(dev_ProjLr);

	checkCudaErrors( hipMalloc3DArray(&dev_ProjLr,&channelDesc,siz_PL,hipArrayLayered ));
/*
	hipError_t eC =  hipMalloc3DArray(&dev_ProjLr,&channelDesc,siz_PL,hipArrayLayered );
	if (eC != hipSuccess)
	{
		std::cout << "Erreur cuda malloc\n";
		std::cout << "Dimension du tableau des Images : " << h.dimImg.x << ","<< h.dimImg.x << "," << nbLayer  << "\n";
		std::cout << "Dimension du tableau des projections : " << dimTer.x << ","<< dimTer.x << "," << nbLayer  << "\n";
		checkCudaErrors(eC);

	}
*/


}

extern "C" void  CopyProjToLayers(float *h_TabProj, uint2 dimTer, int nbLayer)
{
	hipExtent siz_PL = make_hipExtent( dimTer.x, dimTer.y, nbLayer);

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };

	p.dstArray	= dev_ProjLr;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(h_TabProj, siz_PL.width * sizeof(float2), siz_PL.width, siz_PL.height);
	p.extent	= siz_PL;
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des projections du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

};

__device__  inline float2 simpleProjection( uint2 size, uint2 ssize/*, uint2 sizeImg*/ ,uint2 coord, int L)
{
	const float2 cf = make_float2(ssize) * make_float2(coord) / make_float2(size) ;
	const int2	 a	= make_int2(cf);
	const float2 uva = (make_float2(a) + 0.5f) / (make_float2(ssize));
	const float2 uvb = (make_float2(a+1) + 0.5f) / (make_float2(ssize));
	float2 ra, rb, Iaa;

	ra	= tex2DLayered( TexLay_Proj, uva.x, uva.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uva.y, L);
	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(cH.badVig);

	Iaa	= ((float)(a.x + 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	ra	= tex2DLayered( TexLay_Proj, uva.x, uvb.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uvb.y, L);

	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(cH.badVig);

	ra	= ((float)(a.x+ 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	ra = ((float)(a.y+ 1.0f) - cf.y) * Iaa + (cf.y - (float)(a.y)) * ra;
	/*ra = (ra + 0.5f) / (make_float2(sizeImg));*/

	return ra;
}

__global__ void correlationKernel( float *dev_NbImgOk, float* cachVig/*, float *siCor*/, uint2 nbActThrd ) //__global__ void correlationKernel( int *dev_NbImgOk, float* cachVig)
{
	__shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];

	// Coordonn�es du terrain global avec bordure
	const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);
	
	// Si le processus est hors du terrain, nous sortons du kernel
	if (oSE(ptHTer,cH.dimTer)) return;

	//float2 PtTProj = tex2DLayered(TexLay_Proj, ((float)ghTer.x / (float)cDimTer.x * (float)cSDimTer.x + 0.5f) /(float)cSDimTer.x, ((float)ghTer.y/ (float)cDimTer.y * (float)cSDimTer.y + 0.5f) /(float)cSDimTer.y ,blockIdx.z) ;
	//const float2 PtTProj = simpleProjection( cDimTer, cSDimTer/*, cDimImg*/, ptHTer, blockIdx.z);
	const float2 PtTProj = tex2DLayered(TexLay_Proj, ((float)ptHTer.x  + 0.5f) /(float)cH.dimTer.x, ((float)ptHTer.y + 0.5f) /(float)cH.dimTer.y ,blockIdx.z) ;
	
	const int2 ptTer	= make_int2(ptHTer) - make_int2(cH.rVig);
	const int2 caVig	= ptTer * make_int2(cH.dimVig);
	const int  iC		= blockIdx.z * cH.sizeCach +  to1D( caVig, cH.dimCach );

	if (oEq(PtTProj, cH.UVDefValue))
	{
		cacheImg[threadIdx.y][threadIdx.x]  = cH.badVig;
		if (!( oSE( caVig, cH.dimCach ) || oI(caVig, 0))) //if (blockIdx.z	== iDI) siCor[iTer2] = 2*cH.badVig; 
			cachVig[iC]		= cH.badVig;
		return;
	}
 	else
		// !!! ATTENTION Modification pour simplification du debug !!!!
		//cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( refTex_ImagesLayered, (PtTProj.x + 0.5f) / (float)cDimImg.x, (PtTProj.y + 0.5f) / (float)cDimImg.y,blockIdx.z);
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( refTex_ImagesLayered, (((int)PtTProj.x )+ 0.5f) / (float)cH.dimImg.x, (((int)(PtTProj.y) )+ 0.5f) / (float)cH.dimImg.y,blockIdx.z);

	__syncthreads();

	// Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel
	if ( oSE(threadIdx, nbActThrd + cH.rVig) || oI(threadIdx , cH.rVig) || oSE( ptTer, cH.rDiTer) || oI(ptTer, 0))
		return;
	
	const short2 c0	= make_short2(threadIdx) - cH.rVig;
	const short2 c1	= make_short2(threadIdx) + cH.rVig;

	// Si le parcours de la vignette est hors du terrain, nous sortons!!! Sinon crash GPU!!!!
// 	if ( (c1.x >= blockDim.x) || (c1.y >= blockDim.y) || (c0.x < 0) || (c0.y < 0) )	//if (blockIdx.z == iDI) siCor[iTer] = 3*cH.badVig; // ## z ##
// 	{
// 		cachVig[iC] = cH.badVig;
// 		return;
// 	}

	// Intialisation des valeurs de calcul 
	float aSV = 0.0f, aSVV	= 0.0f;
	short x,y;
	
	#pragma unroll // ATTENTION PRAGMA FAIT AUGNENTER LA quantit� MEMOIRE des registres!!!
	for (y = c0.y ; y <= c1.y; y++)
	{
		#pragma unroll
		for (x = c0.x ; x <= c1.x; x++)
		{	
			const float val = cacheImg[y][x];	// Valeur de l'image

			if (val ==  cH.badVig)
			{
				cachVig[iC] = cH.badVig; 
				return;
			}
			aSV  += val;		// Somme des valeurs de l'image cte 
			aSVV += (val*val);	// Somme des carr�s des vals image cte
		}
	}

	aSV	 /=	cH.sizeVig;
	aSVV /=	cH.sizeVig;
	aSVV -=	(aSV * aSV);
	
	if ( aSVV <= cH.mAhEpsilon) //
	{
		cachVig[iC] = cH.badVig;
		return;
	}

	aSVV =	sqrt(aSVV);

	#pragma unroll
	for ( y = c0.y ; y <= c1.y; y++)
	{
		const int _cy	= ptTer.y * cH.dimVig.y + (y - c0.y);
		#pragma unroll
		for ( x = c0.x ; x <= c1.x; x++)					
// 			if (cacheImg[y][x] == cH.badVig)
// 			{
// 				cachVig[iC] = cH.badVig;
// 				return;
// 			}
// 			const int _cx	= ter.x * cDimVig.x + (x - c0.x);
// 			const int _iC   = (blockIdx.z * cH.sizeCach) + _cy * cH.dimCach.x + _cx;
			cachVig[(blockIdx.z * cH.sizeCach) + _cy * cH.dimCach.x + ptTer.x * cH.dimVig.x + (x - c0.x)] = (cacheImg[y][x] -aSV)/aSVV;
		
	}

//  	if (blockIdx.z	== iDI)
// 		siCor[iTer] = (1.0f + cachVig[iC]) / 2.0f; //== 0.0f ? -9 * cH.badVig : cachVig[iC] ; // ## � ##

	// Coordonn�es 1D du terrain
	//const int iTer	= (cRDiTer.x * ter.y) + ter.x; // ne sert pas 
	// Nombre d'images correctes
	//atomicAdd( &dev_NbImgOk[iTer], 1.0f);
	atomicAdd( &dev_NbImgOk[to1D(ptTer,cH.rDiTer)], 1.0f);
};

///////////////////////////////////////////////////////////////////////////////////
//																				///
// Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens	///
//																				///
///////////////////////////////////////////////////////////////////////////////////

__global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, float * dev_NbImgOk, uint2 nbActThr)
{
	__shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float aSVV[ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];

	// coordonn�es des threads
	const uint2 t = make_uint2(threadIdx);

	if ( threadIdx.z == 0)
	{
		aSV [t.y][t.x]		= 0.0f;
		aSVV[t.y][t.x]		= 0.0f;
		resu[t.y/2][t.x/2]	= 0.0f;
	}
	
	__syncthreads();

	// si le thread est inactif, il sort
 	if ( oSE( t, nbActThr))
 		return;

	// Coordonn�es 2D du cache vignette
	const uint2 cCach = make_uint2(blockIdx) * nbActThr  + t;
	
	// Si le thread est en dehors du cache
	if ( oSE(cCach, cH.dimCach))
		return;
	
	const uint2 coorTer	= cCach / cH.dimVig;						// Coordonn�es 2D du terrain 
	const uint iTer		= to1D(coorTer, cH.rDiTer);					// Coordonn�es 1D dans le terrain
	const bool mThrd	= t.x % cH.dimVig.x == 0 &&  t.y% cH.dimVig.y == 0 && threadIdx.z == 0;
	const float aNbImOk = dev_NbImgOk[iTer];

	if (aNbImOk < 2)
	{
		if (mThrd) dTCost[iTer] = -1000.0f;
		return;
	}

	const uint pitCachLayer = threadIdx.z * cH.sizeCach;			// Taille du cache vignette pour une image
	const uint iCach	= pitCachLayer + to1D( cCach, cH.dimCach );	// Coordonn�es 1D du cache vignette
	const uint2 cc		= coorTer * cH.dimVig;						// coordonn�es 2D 1er pixel de la vignette
	const int iCC		= pitCachLayer + to1D( cc, cH.dimCach );	// coordonn�es 1D 1er pixel de la vignette
	
	float val = (cacheVign[iCC] != cH.badVig) ? cacheVign[iCach] : 0.0f; // sortir si bad vignette

	atomicAdd( &(aSV[t.y][t.x]), val);

	const float VV = val * val;
	atomicAdd(&(aSVV[t.y][t.x]), VV);
	__syncthreads();

	if ( threadIdx.z != 0) return;

	// Coordonn�es 2D du terrain dans le repere des threads
	const uint2 coorTTer = t / cH.dimVig;
	
	atomicAdd(&(resu[coorTTer.y][coorTTer.x]),aSVV[t.y][t.x] - ((aSV[t.y][t.x] * aSV[t.y][t.x])/ aNbImOk)); 
	__syncthreads();

	if ( !mThrd ) return;

	// Normalisation pour le ramener a un equivalent de 1-Correl 
	const float cost = resu[coorTTer.y][coorTTer.x]/ (( aNbImOk -1.0f) * ((float)cH.sizeVig));

	dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));
}

extern "C" paramGPU Init_Correlation_GPU(  uint2 ter0, uint2 ter1, int nbLayer , uint2 dRVig , uint2 dimImg, float mAhEpsilon, uint samplingZ, float uvDef )
{

	dev_NbImgOk		= NULL;
	//dev_SimpCor		= NULL;
	dev_Cache		= NULL;
	dev_Cost		= NULL;
	dev_ProjLr		= NULL;

	correlOptionsGPU( ter0, ter1, dRVig * 2 + 1,dRVig, dimImg,mAhEpsilon, samplingZ, uvDef,nbLayer);
	allocMemory();

	return h;
}

extern "C" void basic_Correlation_GPU( float* h_TabCost,  int nbLayer ){
	
	//////////////////////////////////////////////////////////////////////////
	 
	//int sCorMemSize = h.sizeTer  * sizeof(float);
	int nBI_MemSize = h.rSiTer	 * sizeof(float);
	int cac_MemSize = h.sizeCach * sizeof(float) * nbLayer;
	int costMemSize = h.rSiTer	 * sizeof(float);

	//////////////////////////////////////////////////////////////////////////

	//checkCudaErrors( hipMemset( dev_SimpCor,	0, sCorMemSize ));
	checkCudaErrors( hipMemset( dev_Cost,		0, costMemSize ));
	checkCudaErrors( hipMemset( dev_Cache,		0, cac_MemSize ));
	checkCudaErrors( hipMemset( dev_NbImgOk,	0, nBI_MemSize ));
	checkCudaErrors( hipBindTextureToArray(TexLay_Proj,dev_ProjLr) );

	//////////////////////////////////////////////////////////////////////////

	dim3 threads( BLOCKDIM, BLOCKDIM, 1);
	uint2 actiThsCo = make_uint2(threads.x - 2 *((int)(h.dimVig.x)), threads.y - 2 * ((int)(h.dimVig.y)));
	dim3 blocks(iDivUp((int)(h.dimTer.x),actiThsCo.x) , iDivUp((int)(h.dimTer.y), actiThsCo.y), nbLayer);
	
	uint2 actiThs = make_uint2(SBLOCKDIM - SBLOCKDIM % ((int)h.dimVig.x), SBLOCKDIM - SBLOCKDIM % ((int)h.dimVig.y));
	dim3 threads_mC(SBLOCKDIM, SBLOCKDIM, nbLayer);
	dim3 blocks_mC(iDivUp((int)(h.dimCach.x), actiThs.x) , iDivUp((int)(h.dimCach.y), actiThs.y));

	////////////////////--  KERNEL  Correlation  --//////////////////////////
	
	correlationKernel<<<blocks, threads>>>( dev_NbImgOk, dev_Cache, actiThsCo);
	getLastCudaError("Basic Correlation kernel failed");
	//hipDeviceSynchronize();
	
	//////////////////--  KERNEL  Multi Correlation  --///////////////////////

   	multiCorrelationKernel<<<blocks_mC, threads_mC>>>( dev_Cost, dev_Cache, dev_NbImgOk, actiThs);
   	getLastCudaError("Multi-Correlation kernel failed");

	//////////////////////////////////////////////////////////////////////////

	checkCudaErrors( hipUnbindTexture(TexLay_Proj) );
	checkCudaErrors( hipMemcpy( h_TabCost, dev_Cost, costMemSize, hipMemcpyDeviceToHost) );
	
	//hipDeviceSynchronize();
	//checkCudaErrors( hipMemcpy( h_TabCost, dev_NbImgOk, costMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_SimpCor, dev_SimpCor, sCorMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_Cache, dev_Cache,	  cac_MemSize, hipMemcpyDeviceToHost) );

	//////////////////////////////////////////////////////////////////////////
/*
	if(0)
	{
		//////////////////////////////////////////////////////////////////////////
		if (0)
		{
			//for (uint idii = 0 ; idii < 4 ; idii++)
			uint idii = 1;
			{
				std::cout << "CACHE IMAGE : " << idii << " --------------------------------\n";
				for (uint j = 0 ; j < h.dimCach.y / h.dimVig.y ; j++)
				{
					for (uint i = 0; i < h.dimCach.x / h.dimVig.x ; i++)
					{
						float off	= 10.0f;
						int ii = i * h.dimVig.x + h.rVig.x;
						int jj = j * h.dimVig.y + h.rVig.y;

						int id		= (idii * h.sizeCach + jj * h.dimCach.x + ii );
						float out	= host_Cache[id];
						out			= floor(out*off)/off;

						int bad = -4;

						std::string S2 = "   ";
						std::string ES = "";
						std::string S1 = "  ";

						std::string valS;
						stringstream sValS (stringstream::in | stringstream::out);
						sValS << abs(out);
						long sizeV = sValS.str().length();
						if (sizeV == 3) ES = ES + " ";
						else if (sizeV == 2) ES = ES + "  ";
						else if (sizeV == 1) ES = ES + "   ";

						if (out == bad)
							std::cout << S1 << "!" + S2;
						else if (out == -1000.0f)
							std::cout << S1 << "." << S2;
						else if (out == 2*bad)
							std::cout << S1 << "s" << S2;
						else if (out == 3*bad)
							std::cout << S1 << "z" << S2;
						else if (out == 4*bad)
							std::cout << S1 << "s" << S2;
						else if (out == 5*bad)
							std::cout << S1 << "v" << S2;
						else if (out == 6*bad)
							std::cout << S1 << "e" << S2;
						else if (out == 7*bad)
							std::cout << S1 << "c" << S2;
						else if (out == 8*bad)
							std::cout << S1 << "?" << S2;
						else if (out == 9*bad)
							std::cout << S1 << "�" << S2;
						else if (out == 0.0f)
							std::cout << S1 << "0" << S2;
						else if ( out < 0.0f)
							std::cout <<  out << ES;				
						else 
							std::cout << S1 << out << ES;

					}
					std::cout << "\n";	
				}
				std::cout << "------------------------------------------\n";
			}
		}

		if (0)
		{
			uint idImage = 0;

			uint2 dimCach = h.dimTer * h.dimVig;

			float* imageCache	= new float[h.sizeTer * h.sizeVig];
			for (uint j = 0; j < dimCach.y; j++)
				for (uint i = 0; i < dimCach.x ; i++)
				{
					int id = (j * dimCach.x + i );
					imageCache[id] = host_Cache[idImage * size(dimCach) + id]/7.0f + 3.5f;
				}

				TCHAR name [ UNLEN + 1 ];
				DWORD size = UNLEN + 1;
				GetUserName( (TCHAR*)name, &size );

				std::string suname = name;

				std::string fileImaCache = "C:\\Users\\" + suname + "\\Pictures\\imageCache.pgm";

				std::cout << suname << "\n";
				// save PGM
				if (sdkSavePGM<float>(fileImaCache.c_str(), imageCache, dimCach.x,dimCach.y))
					std::cout <<"success save image" << "\n";
				else
					std::cout <<"Failed save image" << "\n";

				delete[] imageCache;
		
			float* image	= new float[h.rSiTer];
			for (uint j = 0; j < h.rDiTer.y ; j++)
				for (uint i = 0; i < h.rDiTer.x ; i++)
				{
					int id = (j * h.rDiTer.x + i );
					if (host_SimpCor[id] == -8)
					{
						image[id] = 0;
					} 
					else
					{
						image[id] = host_SimpCor[id]/500.f;	
						//image[id] = host_SimpCor[id]/2.0f;	
					}
					
				}

			TCHAR name [ UNLEN + 1 ];
			DWORD size = UNLEN + 1;
			GetUserName( (TCHAR*)name, &size );

			std::string suname = name;
			std::string fileImage = "C:\\Users\\" + suname + "\\Pictures\\image.pgm";

			// save PGM
			if (sdkSavePGM<float>(fileImage.c_str(), image, h.rDiTer.x,h.rDiTer.y))
				std::cout <<"success save image" << "\n";
			else
				std::cout <<"Failed save image" << "\n";

			delete[] image;
		}
		

		if(0)
		{

			for (uint j = 0 ; j < h.dimTer.y; j+= h.sampTer)
			{
				for (uint i = 0; i < h.dimTer.x ; i+= h.sampTer)
				{
					float off = 10000.0f;
					int id = (j * h.dimTer.x + i );
					float out = host_SimpCor[id];
					std::cout << floor(out*off)/off << " ";
				}
				std::cout << "\n";	
			}
			std::cout << "------------------------------------------\n";
		}
		if (0)
		{
			for (uint j = 0 ; j < h.rDiTer.y; j++)
			{
				for (uint i = 0; i < h.rDiTer.x ; i++)
				{
					float off = 10.0f;
					int id = (j * h.rDiTer.x + i );
					float out = h_TabCost[id];
					if (out < 10)
						std::cout << out << "  ";
					else
						std::cout << out << " ";
				}
				std::cout << "\n";	
			}
			std::cout << "------------------------------------------\n";

		}

		if (0)
		{

			for (uint j = 0 ; j < h.rDiTer.y; j+= h.sampTer)
			{
				for (uint i = 0; i < h.rDiTer.x ; i+= h.sampTer)
				{
					float off = 1.0f;

					int id = (j * h.rDiTer.x + i );
					float out = h_TabCost[id];
					if (out == -1000)
						std::cout << ".  ";
					else if (out >= 10 )
						std::cout << floor(out*off)/off  << " ";
					else
						std::cout << floor(out*off)/off  << "  ";
				}

				std::cout << "\n";	
			}

			std::cout << "------------------------------------------\n";
		}


		//if (0)
		

		{
			int bad = -4;
			for (uint j = 0 ; j < h.rDiTer.y; j+= h.sampTer)
			{
				for (uint i = 0; i < h.rDiTer.x ; i+= h.sampTer)
				{

					float off = 100.0f;
					int id = (j * h.rDiTer.x + i );

					std::string S2 = "    ";
					std::string ES = "";
					std::string S1 = " ";

					//float out = host_SimpCor[id];// 500.0f;
					float out = h_TabCost[id];
					out = floor(out*off)/off ;

					std::string valS;
					stringstream sValS (stringstream::in | stringstream::out);
					sValS << abs(out);
					long sizeV = sValS.str().length();

					if (sizeV == 5) ES = ES + "";
					else if (sizeV == 4) ES = ES + " ";
					else if (sizeV == 3) ES = ES + "  ";
					else if (sizeV == 2) ES = ES + "   ";
					else if (sizeV == 1) ES = ES + "    ";

					if (out == bad)
						std::cout << S1 << "!" + S2;
					else if (out == -1000.0f)
						std::cout << S1 << "." << S2;
					else if (out == 2*bad)
						std::cout << S1 << "s" << S2;
					else if (out == 3*bad)
						std::cout << S1 << "z" << S2;
					else if (out == 4*bad)
						std::cout << S1 << "s" << S2;
					else if (out == 5*bad)
						std::cout << S1 << "v" << S2;
					else if (out == 6*bad)
						std::cout << S1 << "e" << S2;
					else if (out == 7*bad)
						std::cout << S1 << "c" << S2;
					else if (out == 8*bad)
						std::cout << S1 << "?" << S2;
					else if (out == 9*bad)
						std::cout << S1 << "�" << S2;
					else if (out == 0.0f)
						std::cout << S1 << "0" << S2;
					else if ( out < 0.0f)
						std::cout << out << ES;				
					else 
						std::cout << S1 << out << ES;

				//////////////////////////////////////////////////////////////////////////
// 					else if ( out < 0.0f && out > -1.0f)
// 					{
// 						std::cout << " " << out << ES;
// 						//std::cout << "|\\|";
// 					}
// 					else if ( out > 0.0f && out < 1.0f)
// 						std::cout << S1 << out << ES;
// 						//std::cout << " *" << S1;
// 					else
// 						std::cout << S1 << "H" << S2;

				}
				std::cout << "\n";	
			}
			std::cout << "------------------------------------------\n";
		}	
	}
	*/
}

extern "C" void freeGpuMemory()
{
	//checkCudaErrors( hipUnbindTexture(refTex_Image) );
	//checkCudaErrors( hipFreeArray(dev_Img) );
	//checkCudaErrors( hipFreeArray(dev_CubeProjImg) );
	//checkCudaErrors( hipFreeArray(dev_ArrayProjImg) );

	checkCudaErrors( hipUnbindTexture(refTex_ImagesLayered) );	


	if(dev_ImgLd	!= NULL) checkCudaErrors( hipFreeArray( dev_ImgLd) );
	if(dev_ProjLr	!= NULL) checkCudaErrors( hipFreeArray( dev_ProjLr) );
	if(dev_NbImgOk	!= NULL) checkCudaErrors( hipFree( dev_NbImgOk));
	//if(dev_SimpCor	!= NULL) checkCudaErrors( hipFree( dev_SimpCor));
	if(dev_Cache	!= NULL) checkCudaErrors( hipFree( dev_Cache));
	if(dev_Cost		!= NULL) checkCudaErrors( hipFree( dev_Cost));

	dev_NbImgOk	= NULL;
	//dev_SimpCor = NULL;
	dev_Cache	= NULL;
	dev_ImgLd	= NULL;
	dev_Cost	= NULL;

	// DEBUG 
	//free(host_SimpCor); 
	free(host_Cache);
}

extern "C" void  FreeLayers()
{
	checkCudaErrors( hipFreeArray(dev_ImgLd));

};

extern "C" void  projToDevice(hipArray_t *dev_ArrayProjImg,texture<float2, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Project, float* aProj,  int sXImg, int sYImg)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(dev_ArrayProjImg,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(*dev_ArrayProjImg,0,0,aProj, sYImg*sizeof(float2),sYImg*sizeof(float2), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Project,*dev_ArrayProjImg) );

}

extern "C" void cubeProjToDevice(hipArray_t *dev_CubeProjImg,float* cubeProjPIm, hipExtent dimCube)
{

	// Format des canaux 
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
			
	// Taille du cube
	hipExtent sizeCube = dimCube;
			
	// Allocation memoire GPU du cube de projection
	checkCudaErrors( hipMalloc3DArray(dev_CubeProjImg,&channelDesc,sizeCube) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };
			
	p.dstArray	= *dev_CubeProjImg;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(cubeProjPIm, dimCube.width * 2 * sizeof(float), dimCube.width, dimCube.height);
	p.extent	= dimCube;					// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie du cube de projection du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );
		
}

extern "C" void  imageToDevice(hipArray_t *dev_Img, texture<float, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Image, float** aDataIm,  int sXImg, int sYImg)
{
	float *dataImg1D	= new float[sXImg*sYImg];
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	// TACHE : changer la structuration des donnees pour le stockage des images 
	// Tableau 2D  --->> tableau lin�aire
	for (int i = 0; i < sXImg ; i++)
		for (int j = 0; j < sYImg ; j++)
			dataImg1D[i*sYImg+j] = aDataIm[j][i];

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(dev_Img,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(*dev_Img,0,0,dataImg1D, sYImg*sizeof(float),sYImg*sizeof(float), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Image,*dev_Img) );

	delete dataImg1D;

}
