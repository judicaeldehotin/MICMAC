#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

texture<float, 1, hipReadModeNormalizedFloat> refDeTex;

hipArray* dev_Img;
// Declaration du cube de projection pour le device
hipArray* dev_CubeProjImg;
	
// Creation des tableaux de resultats de corr�lation
/*

> Cube des corr�lations
	- channel : somme des valeurs, somme des valeur


*/

extern "C" void  imageToDevice(float** aDataIm,  int sXImg, int sYImg)
{
	float *dataImg1D	= new float[sXImg*sYImg];
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();


	// TACHE : changer la structuration des donnees pour le stockage des images 
	// Tableau 2D  --->> tableau lin�aire
	for (int i = 0; i < sXImg ; i++)
		for (int j = 0; j < sYImg ; j++)
			dataImg1D[i*sYImg+j] = aDataIm[j][i];

	hipError_t cudaERROR;

	// Allocation m�moire du tableau cuda
	cudaERROR = hipMallocArray(&dev_Img,&channelDesc,sYImg,sXImg);

	// Copie des donn�es du Host dans le tableau Cuda
	cudaERROR = hipMemcpy2DToArray(dev_Img,0,0,dataImg1D, sYImg*sizeof(float),sYImg*sizeof(float), sXImg, hipMemcpyHostToDevice);

	// Lier la texture au tableau Cuda
	cudaERROR = hipBindTextureToArray(refDeTex,dev_Img);

}

extern "C" void cubeProjToDevice(float* cubeProjPIm, hipExtent dimCube)
{


		// Variable erreur cuda
		hipError_t cudaERROR;

		// Format des canaux 
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
		//hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
			
		// Taille du cube
		hipExtent sizeCube = dimCube;
			
		// Allocation memoire GPU du cube de projection
		cudaERROR = hipMalloc3DArray(&dev_CubeProjImg,&channelDesc,sizeCube);

		// D�claration des parametres de copie 3D
		hipMemcpy3DParms p = { 0 };
			
		// Pointeur du tableau de destination
		p.dstArray	= dev_CubeProjImg;
		// Pas du cube
		p.srcPtr	= make_hipPitchedPtr(cubeProjPIm, dimCube.width * 2 * sizeof(float), dimCube.width, dimCube.height);
		// Taille du cube
		p.extent	= dimCube;
		// Type de copie
		p.kind		= hipMemcpyHostToDevice;

		// Copie du cube de projection du Host vers le Device
		cudaERROR	= hipMemcpy3D(&p);
		// Sortie console : Statut de la copie 3D
		
}

extern "C" void correlation(){

	


}

extern "C" void freeTexture()
{
	hipUnbindTexture(refDeTex);
	hipFreeArray(dev_Img);
	hipFreeArray(dev_CubeProjImg);
}