#include "hip/hip_runtime.h"
#include "gpu/cudaAppliMicMac.cuh"

#ifdef _DEBUG
	#define   BLOCKDIM	8
	#define   SBLOCKDIM 10
#else
	#define   BLOCKDIM	32
	#define   SBLOCKDIM 16
#endif

//------------------------------------------------------------------------------------------
// Non utilis�
texture<float, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Image;
texture<bool, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Cache;
texture<float2, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Project;
hipArray* dev_Img;				// Tableau des valeurs de l'image
hipArray* dev_CubeProjImg;		// Declaration du cube de projection pour le device
hipArray* dev_ArrayProjImg;	// Declaration du tableau de projection pour le device
//------------------------------------------------------------------------------------------


//------------------------------------------------------------------------------------------
// ATTENTION : erreur de compilation avec l'option hipReadModeNormalizedFloat
// et l'utilisation de la fonction tex2DLayered
texture<float2,	hipTextureType2DLayered > TexLay_Proj;
texture<float,	hipTextureType2DLayered > refTex_ImagesLayered;
hipArray* dev_ImgLd;	//
hipArray* dev_ProjLr;		//

//------------------------------------------------------------------------------------------
//float*	host_Corr_Out;
float*	host_Cache;
int*	host_NbImgOk;
float*	dev_SimpCor;
float*	dev_Cost;
float*	dev_Cache;
int*	dev_NbImgOk;

paramGPU h;

extern "C" void allocMemory(void)
{

	if (dev_NbImgOk	!= NULL) checkCudaErrors( hipFree(dev_NbImgOk));
	if (dev_SimpCor != NULL) checkCudaErrors( hipFree(dev_SimpCor));
	if (dev_Cache	!= NULL) checkCudaErrors( hipFree(dev_Cache));
	if (dev_Cost	!= NULL) checkCudaErrors( hipFree(dev_Cost));


	int out_MemSize = h.sizeTer * sizeof(float);
	int costMemSize = h.rSiTer	* sizeof(float);
	int nBI_MemSize = h.sizeTer	* sizeof(int);
	int cac_MemSize = h.sizeCach* sizeof(float)* h.nLayer;

	
	// Allocation m�moire
	//host_Corr_Out	= (float*)	malloc(out_MemSize);
	//host_Cache		= (float*)	malloc(cac_MemSize);
	//host_NbImgOk	= (int*)	malloc(nBI_MemSize);

	
	checkCudaErrors( hipMalloc((void **) &dev_SimpCor	, out_MemSize) );	
	checkCudaErrors( hipMalloc((void **) &dev_Cache	, cac_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_NbImgOk	, nBI_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_Cost		, costMemSize ) );

	
	// Texture des projections
	TexLay_Proj.addressMode[0]	= hipAddressModeClamp;
	TexLay_Proj.addressMode[1]	= hipAddressModeClamp;	
	TexLay_Proj.filterMode		= hipFilterModePoint; //hipFilterModePoint 
	TexLay_Proj.normalized		= true;

}

extern "C" paramGPU updateSizeBlock( int x0, int x1, int y0, int y1 )
{

	uint oldSizeTer = h.sizeTer;

	h.pUTer0.x	= x0 - h.rVig.x;
	h.pUTer0.y	= y0 - h.rVig.y;
	h.pUTer1.x	= x1 + h.rVig.x;
	h.pUTer1.y	= y1 + h.rVig.y;
	
	h.rDiTer	= make_uint2(x1 - x0, y1 - y0);
	h.dimTer	= make_int2(h.pUTer1.x - h.pUTer0.x, h.pUTer1.y - h.pUTer0.y);
	h.dimSTer	= iDivUp(h.dimTer,h.sampTer);	// Dimension du bloque terrain sous echantillon�
	h.sizeTer	= size(h.dimTer);				// Taille du bloque terrain
	h.sizeSTer  = size(h.dimSTer);				// Taille du bloque terrain sous echantillon�
	h.rSiTer	= size(h.rDiTer);

	h.dimCach	= h.rDiTer * h.dimVig;
	h.sizeCach	= size(h.dimCach);
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cRDiTer), &h.rDiTer, sizeof(uint2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSDimTer), &h.dimSTer, sizeof(int2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cDimTer), &h.dimTer, sizeof(int2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSizeTer), &h.sizeTer, sizeof(uint)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSizeSTer), &h.sizeSTer, sizeof(uint)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cDimCach), &h.dimCach, sizeof(uint2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSizeCach), &h.sizeCach, sizeof(uint)));

	if (oldSizeTer < h.sizeTer)
		allocMemory();

	return h;
}

static void correlOptionsGPU(int x0, int x1, int y0, int y1, uint2 dV,uint2 dRV, uint2 dI, float mAhEpsilon, uint samplingZ, float uvDef, uint nLayer )
{

	h.nLayer	= nLayer;
	h.dimVig	= dV;							// Dimension de la vignette
	h.dimImg	= dI;							// Dimension des images
	h.rVig		= dRV;							// Rayon de la vignette
	h.sizeVig	= size(dV);						// Taille de la vignette en pixel 
	h.sampTer	= samplingZ;					// Pas echantillonage du terrain
	h.UVDefValue= uvDef;						// UV Terrain incorrect
	int badVi	= -4;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cRVig), &dRV, sizeof(uint2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cDimVig), &h.dimVig, sizeof(uint2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cDimImg), &dI, sizeof(uint2)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cMAhEpsilon), &mAhEpsilon, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSizeVig), &h.sizeVig, sizeof(uint)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cSampTer), &h.sampTer, sizeof(uint)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cUVDefValue), &h.UVDefValue, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cBadVignet), &badVi, sizeof(int)));
	
	updateSizeBlock( x0, x1, y0, y1 );
}

extern "C" void imagesToLayers(float *fdataImg1D, uint2 dimTer, int nbLayer)
{
	hipExtent sizeImgsLay = make_hipExtent( dimTer.x, dimTer.y, nbLayer );

	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// Allocation memoire GPU du tableau des calques d'images
	checkCudaErrors( hipMalloc3DArray(&dev_ImgLd,&channelDesc,sizeImgsLay,hipArrayLayered) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms	p	= { 0 };
	hipPitchedPtr		pit = make_hipPitchedPtr(fdataImg1D, sizeImgsLay.width * sizeof(float), sizeImgsLay.width, sizeImgsLay.height);

	p.dstArray	= dev_ImgLd;		// Pointeur du tableau de destination
	p.srcPtr	= pit;						// Pitch
	p.extent	= sizeImgsLay;				// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des images du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

	// Li� � la texture
	refTex_ImagesLayered.addressMode[0]	= hipAddressModeWrap;
    refTex_ImagesLayered.addressMode[1]	= hipAddressModeWrap;
    refTex_ImagesLayered.filterMode		= hipFilterModeLinear; //hipFilterModeLinear hipFilterModePoint
    refTex_ImagesLayered.normalized		= true;
	checkCudaErrors( hipBindTextureToArray(refTex_ImagesLayered,dev_ImgLd) );

};

extern "C" void  projectionsToLayers(float *h_TabProj, int2 dimTer, int nbLayer)
{
	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Taille du tableau des calques 
	hipExtent siz_PL = make_hipExtent( dimTer.x, dimTer.y, nbLayer);

	// Allocation memoire GPU du tableau des calques d'images
	checkCudaErrors( hipMalloc3DArray(&dev_ProjLr,&channelDesc,siz_PL,hipArrayLayered ));

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };

	p.dstArray	= dev_ProjLr;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(h_TabProj, siz_PL.width * sizeof(float2), siz_PL.width, siz_PL.height);
	p.extent	= siz_PL;
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des projections du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

};

__device__  inline float2 simpleProjection( uint2 size, uint2 ssize, uint2 sizeImg ,int2 coord, int L)
{
	const float bad = -1.0f;
	const float2 cf = make_float2(ssize) * make_float2(coord) / make_float2(size) ;
	const int2	 a	= make_int2(cf);
	const float2 uva = (make_float2(a) + 0.5f) / (make_float2(ssize));
	const float2 uvb = (make_float2(a+1) + 0.5f) / (make_float2(ssize));
	float2 ra, rb, Iaa;

	ra	= tex2DLayered( TexLay_Proj, uva.x, uva.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uva.y, L);
	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(bad);

	Iaa	= ((float)(a.x + 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	ra	= tex2DLayered( TexLay_Proj, uva.x, uvb.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uvb.y, L);

	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(bad);

	ra	= ((float)(a.x+ 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	
	ra = ((float)(a.y+ 1.0f) - cf.y) * Iaa + (cf.y - (float)(a.y)) * ra;
	ra = (ra + 0.5f) / (make_float2(sizeImg));

	return ra;
}

__global__ void correlationKernel( int *dev_NbImgOk, float* cache, float *dest )
//__global__ void correlationKernel( int *dev_NbImgOk, float* cache)
{
	__shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];
	const int outMask	= -1;
	const int iDI		= 1;

	// Se placer dans l'espace terrain
	//const int2	coorTer = make_int2(blockIdx) * (make_int2(blockDim) - 2 * make_int2(cRVig)) + make_int2(threadIdx) - make_int2(cRVig);

	const int2	coorTer = make_int2(blockIdx.x * (blockDim.x - 2 * ((int)cRVig.x)) + threadIdx.x - cRVig.x ,   blockIdx.y * (blockDim.y - 2 * ((int)cRVig.y)) + threadIdx.y - cRVig.y  );
	const uint	iTer	= coorTer.y * cDimTer.x + coorTer.x;

	// Si le processus est hors du terrain, nous sortons du kernel
	if ( coorTer.x >= cDimTer.x || coorTer.y >= cDimTer.y || coorTer.x < 0 || coorTer.y < 0) 
		return;

	//const float2 PtTProj = simpleProjection( cDimTer, cSDimTer, cDimImg, coorTer, blockIdx.z);
	//const float2 PtTProj = tex2DLayered(TexLay_Proj, ((float)coorTer.x / cDimTer.x * cSDimTer.x + 0.5f) /(float)cSDimTer.x, ((float)coorTer.y/ cDimTer.y * cSDimTer.y + 0.5f) /(float)cSDimTer.y ,blockIdx.z) ;
	const float2 PtTProj = tex2DLayered(TexLay_Proj, (float)coorTer.x / cDimTer.x , (float)coorTer.y/ cDimTer.y,blockIdx.z) ;

	if ( PtTProj.x == outMask ||  PtTProj.y == outMask )
	{
		cacheImg[threadIdx.y][threadIdx.x]  = cBadVignet;
		if (blockIdx.z == iDI)
			dest[iTer] = 2*cBadVignet; // ## . ##
		return;
	}
	else
	{
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( refTex_ImagesLayered, PtTProj.x, PtTProj.y,blockIdx.z);
		//dest[iTer] = cacheImg[threadIdx.y][threadIdx.x] ;
	}

	__syncthreads();

	// Intialisation des valeurs de calcul 
	float		aSV	= 0.0f;
	float	   aSVV	= 0.0f;
	const int2 c0	= make_int2(threadIdx.x - cRVig.x,threadIdx.y - cRVig.y);
	const int2 c1	= make_int2(threadIdx.x + cRVig.x,threadIdx.y + cRVig.y);

	if ( c1.x >= blockDim.x || c1.y >= blockDim.y || c0.x < 0 || c0.y < 0 )
	{
		//if (blockIdx.z == iDI)
			//dest[iTer] = cacheImg[threadIdx.y][threadIdx.x] ;
		if (blockIdx.z == iDI)
			dest[iTer] = 3*cBadVignet; // ## z ##
		return;
	}

	const uint cx	= cRVig.x + coorTer.x * cDimVig.x;
	const uint cy	= cRVig.y + coorTer.y * cDimVig.y;
	const uint iC   = (blockIdx.z * cSizeCach) + cy * cDimCach.x + cx;

	//#pragma unroll
	for (int y = c0.y ; y <= c1.y; y++)
	{
		//#pragma unroll
		for (int x = c0.x ; x <= c1.x; x++)
		{	
			const float val = cacheImg[y][x];	// Valeur de l'image

			if (val ==  cBadVignet)
			{
				cache[iC] = cBadVignet; 
				if (blockIdx.z == iDI)
					dest[iTer] = 5*cBadVignet; // ## v ##
				return;
			}
			aSV  += val;		// Somme des valeurs de l'image cte 
			aSVV += (val*val);	// Somme des carr�s des vals image cte
		}
	}
	
	aSV	 /=	cSizeVig;
	aSVV /=	cSizeVig;
	aSVV -=	(aSV * aSV);
	
	if ( aSVV <= cMAhEpsilon)
	{
		cache[iC] = cBadVignet;
		if (blockIdx.z == iDI)
			dest[iTer] = 6*cBadVignet; // ## e ##
		return;
	}

	aSVV =	sqrt(aSVV);

	//#pragma unroll
	for (int y = c0.y ; y <= c1.y; y++)
	{
		//const uint pCach = cDimCach.x * (y - c0.y);
		const uint _cy	= coorTer.y * cDimVig.y + (y - c0.y);

		//#pragma unroll
		for (int x = c0.x ; x <= c1.x; x++)
		{			

			if (cacheImg[y][x]  ==  cBadVignet) // A priori Inutile
			{
				cache[iC] = cBadVignet;
				if (blockIdx.z == iDI)
					dest[iTer] = 7*cBadVignet; // ## c ##
				return;
			}			
			const uint _cx	= coorTer.x * cDimVig.x + (x - c0.x);
			const uint _iC   = (blockIdx.z * cSizeCach) + _cy * cDimCach.x + _cx;			
			cache[_iC] = (cacheImg[y][x] -aSV)/aSVV;
		}
	}

	if (blockIdx.z == iDI)
	{
		const uint _cx	= cRVig.x + coorTer.x * cDimVig.x + 1;
		const uint _cy	= cRVig.y + coorTer.y * cDimVig.y + 1;
		const uint _iC   = (blockIdx.z * cSizeCach) + _cy * cDimCach.x + _cx;

		float tr = cache[_iC];
		
		if (tr == 0.0f)
			dest[iTer] = 9*cBadVignet; // ## � ##
		else
			dest[iTer] = tr;
	}
		//dest[iTer] = cSizeVig / 100.0f;

	// Nombre d'images correctes
	atomicAdd( &dev_NbImgOk[iTer], 1);
};

// ---------------------------------------------------------------------------
// Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens
// ---------------------------------------------------------------------------
__global__ void multiCorrelationKernel(float *dest, float* cache, int * dev_NbImgOk)
{

	__shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float aSVV[ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];

	// coordonn�es des threads
	const uint2 t = make_uint2(threadIdx);

	if ( threadIdx.z == 0)
	{
		aSV [t.y][t.x]		= 0.0f;
		aSVV[t.y][t.x]		= 0.0f;
		resu[t.y/2][t.x/2]	= 0.0f;
	}
	
	__syncthreads();

	// Threads utilis�es dans le bloque
	const uint2 activThds = make_uint2(blockDim.x - blockDim.x % cDimVig.x, blockDim.y - blockDim.y % cDimVig.y);

	// si le thread est inactif, il sort
	if ( t.x >=  activThds.x || t.y >=  activThds.y )
		return;

	// Coordonn�es 3D du cache
	const uint2 coordCach		= make_uint2(blockIdx.x * activThds.x  + t.x, blockIdx.y * activThds.y  + t.y);
	
	// Si le thread est en dehors du cache
	if ( coordCach.x >= cDimCach.x || coordCach.y >= cDimCach.y )
		return;
	
	// Coordonn�es 1D du cache
	const unsigned int iCach	= threadIdx.z * cSizeCach + coordCach.y * cDimCach.x + coordCach.x ;
	
	// Coordonn�es 2D du terrain 
	const uint2 coordTer		= coordCach / cDimVig;

	// coordonn�es central de la vignette
	const uint cx	= cRVig.x + coordTer.x * cDimVig.x;
	const uint cy	= cRVig.y + coordTer.y * cDimVig.y;
	const uint iCC	= threadIdx.z * cSizeCach + cy * cDimCach.x + cx;

	if (cache[iCC] == cBadVignet)
		return;
	
	// Coordonn�es 1D dans le cache
	const uint iTer		= coordTer.y * cRDiTer.x  + coordTer.x;

	const bool mainThread	= ((t.x % cDimVig.x)== 0) && ((t.y % cDimVig.y) == 0) && (threadIdx.z == 0);

	const uint aNbImOk		= dev_NbImgOk[iTer];
	if ( aNbImOk < 2)
	{
		if (mainThread)
			dest[iTer] = -1000.0f;
		return;
	}

	// Coordonn�es 2D du terrain dans le repere des threads
	const uint2 coorTTer	= t / cDimVig;

	const float val	= cache[iCach];
	if (val == cBadVignet ) // a virer
		return;
	
	__syncthreads();

	atomicAdd( &aSV[t.y][t.x], val);
	__syncthreads();

	atomicAdd( &aSVV[t.y][t.x], val * val);
	__syncthreads();

	atomicAdd(&resu[coorTTer.y][coorTTer.x],aSVV[t.y][t.x] - ( aSV[t.y][t.x] * aSV[t.y][t.x] / aNbImOk)); 
	__syncthreads();

	if ( !mainThread ) return;

	// Normalisation pour le ramener a un equivalent de 1-Correl 
	const float cost = resu[coorTTer.y][coorTTer.x] / (( aNbImOk-1) * cSizeVig);

	dest[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));

	//dest[iTer] = resu[coorTTer.y][coorTTer.x] / (( aNbImOk-1) * cSizeVig);
}

extern "C" paramGPU Init_Correlation_GPU( int x0, int x1, int y0, int y1, int nbLayer , uint2 dRVig , uint2 dimImg, float mAhEpsilon, uint samplingZ, float uvDef )
{
	dev_NbImgOk		= NULL;
	dev_SimpCor		= NULL;
	dev_Cache		= NULL;
	dev_Cost		= NULL;

	correlOptionsGPU(x0, x1, y0, y1, dRVig * 2 + 1,dRVig, dimImg,mAhEpsilon, samplingZ, uvDef,nbLayer);
	allocMemory();

	return h;
}

extern "C" void basic_Correlation_GPU( float* h_TabCost,  int nbLayer ){
	
	//////////////////////////////////////////////////////////////////////////
	
	int sCorMemSize = h.sizeTer  * sizeof(float);
	int nBI_MemSize = h.sizeTer	 * sizeof(int);
	int cac_MemSize = h.sizeCach * sizeof(float) * nbLayer;
	int costMemSize = h.rSiTer	 * sizeof(float);

	//////////////////////////////////////////////////////////////////////////

	checkCudaErrors( hipMemset( dev_SimpCor, 0, sCorMemSize ));
	checkCudaErrors( hipMemset( dev_Cost, 0, costMemSize ));
	checkCudaErrors( hipMemset( dev_Cache, 0, cac_MemSize ));
	checkCudaErrors( hipMemset( dev_NbImgOk, 0, nBI_MemSize ));
	checkCudaErrors( hipBindTextureToArray(TexLay_Proj,dev_ProjLr) );

	//////////////////////////////////////////////////////////////////////////

	dim3 threads( BLOCKDIM, BLOCKDIM, 1);
	dim3 blocks(iDivUp(h.dimTer.x,threads.x - 2 * h.dimVig.x) , iDivUp(h.dimTer.y,threads.y - 2 * h.dimVig.y), nbLayer);

	int actiThs_X = SBLOCKDIM - SBLOCKDIM % h.dimVig.x;
	int actiThs_Y = SBLOCKDIM - SBLOCKDIM % h.dimVig.y;

	dim3 threads_mC(SBLOCKDIM, SBLOCKDIM, nbLayer);
	dim3 blocks_mC(iDivUp(h.dimCach.x, actiThs_X) , iDivUp(h.dimCach.y, actiThs_Y));

	//////////////////--  KERNEL  Correlation  --//////////////////////
	
	correlationKernel<<<blocks, threads>>>( dev_NbImgOk, dev_Cache , dev_SimpCor);
	getLastCudaError("Basic Correlation kernel failed");
	
	////////////////--  KERNEL  Multi Correlation  --//////////////////////

	multiCorrelationKernel<<<blocks_mC, threads_mC>>>( dev_Cost, dev_Cache, dev_NbImgOk);
	getLastCudaError("Multi-Correlation kernel failed");

	//////////////////////////////////////////////////////////////////////////

	checkCudaErrors( hipUnbindTexture(TexLay_Proj) );
	//checkCudaErrors( hipMemcpy( host_Corr_Out,	dev_Corr_Out, out_MemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( h_TabCorre, dev_SimpCor, out_SimpCor, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy( h_TabCost, dev_Cost, costMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_NbImgOk,	dev_NbImgOk,  nBI_MemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_Cache,	dev_Cache,	  cac_MemSize, hipMemcpyDeviceToHost) );
	//--------------------------------------------------------

	if(0)
	{
		/*if (0)
		{
			uint idImage = 0;

			uint2 dimCach = h.dimTer * h.dimVig;

			float* imageCache	= new float[h.sizeTer * h.sizeVig];
			for (uint j = 0; j < dimCach.y; j++)
				for (uint i = 0; i < dimCach.x ; i++)
				{
					int id = (j * dimCach.x + i );
					imageCache[id] = host_Cache[idImage * size(dimCach) + id]/7.0f + 3.5f;
				}

				std::string fileImaCache = "C:\\Users\\gchoqueux\\Pictures\\imageCache.pgm";
				// save PGM
				if (sdkSavePGM<float>(fileImaCache.c_str(), imageCache, dimCach.x,dimCach.y))
					std::cout <<"success save image" << "\n";
				else
					std::cout <<"Failed save image" << "\n";

				delete[] imageCache;
		
			float* image	= new float[h.sizeTer];
			for (uint j = 0; j < h.dimTer.y ; j++)
				for (uint i = 0; i < h.dimTer.x ; i++)
				{
					int id = (j * h.dimTer.x + i );
					//image[id] = h_TabCorre[id]/500.f;	
					image[id] = h_TabCorre[id]/2.0f;	
				}

			std::string file = "C:\\Users\\gchoqueux\\Pictures\\image.pgm";
			// save PGM
			if (sdkSavePGM<float>(file.c_str(), image, h.dimTer.x,h.dimTer.y))
				std::cout <<"success save image" << "\n";
			else
				std::cout <<"Failed save image" << "\n";

			delete[] image;
		}*/

		int bad = -4;
		/*for (uint j = h.rVig.y ; j < h.dimTer.y - h.rVig.y; j+= h.sampTer)
		{
			for (uint i = h.rVig.x; i < h.dimTer.x - h.rVig.x ; i+= h.sampTer)
			{*/
		for (uint j = 0 ; j < h.dimTer.y; j+= h.sampTer)
		{
			for (uint i = 0; i < h.dimTer.x ; i+= h.sampTer)
			{
				float off = 10000000.0f;
				int id = (j * h.dimTer.x + i );

				float out = h_TabCost[id];

				if (out == bad)
					std::cout << " ! ";
				else if (out == 2*bad)
					std::cout << " . ";
				else if (out == 3*bad)
					std::cout << " z ";
				else if (out == 4*bad)
					std::cout << " s ";
				else if (out == 5*bad)
					std::cout << " v ";
				else if (out == 6*bad)
					std::cout << " e ";
				else if (out == 7*bad)
					std::cout << " c ";
				else if (out == 8*bad)
					std::cout << " ? ";
				else if (out == 9*bad)
					std::cout << " � ";
				else if (out == 0.0f)
 					std::cout << " 0 ";
				else if ( out < 0.0f && out > -1.0f)
				{
					//std::cout << floor(out*off)/off  << " ";
					std::cout << "|\\|";
				}
				else if ( out < 1.0f  && out > 0.0f)
					//std::cout << " "  <<  floor(out*off)/off  << " ";
					std::cout << "|/|";
				else
					//std::cout << floor(out*off)/off  << " ";
					std::cout << " * ";
			}

				std::cout << "\n";	
		}

		std::cout << "------------------------------------------\n";	
		
	}
	if(0)
	{

		float minCache =  1e10;
		float maxCache = -1e10;
		int step = 3;
		for (uint j = 0; j < h.dimTer.y * h.dimVig.y ; j+=step)
		{
			for (uint i = 0; i < h.dimTer.x * h.dimVig.x ; i+=step)
			{
				int id = (j * h.dimTer.x * h.dimVig.x + i );
				float c = host_Cache[id];

				if ( c < minCache || c > maxCache )
				{
					minCache = min( minCache, c);
					maxCache = max( maxCache, c);
					if(c!=0.0f)
					std::cout << minCache << " / " << maxCache << std::endl;
				}
				//float c = host_NbImgOk[id];
				//std::cout << c << " ";
			}
			//std::cout << std::endl; 
		}
	}

}

extern "C" void freeGpuMemory()
{
	checkCudaErrors( hipUnbindTexture(refTex_Image) );
	checkCudaErrors( hipUnbindTexture(refTex_ImagesLayered) );
	checkCudaErrors( hipFreeArray(dev_Img) );	
	checkCudaErrors( hipFreeArray(dev_CubeProjImg) );
	checkCudaErrors( hipFreeArray(dev_ArrayProjImg) );


	if(dev_ImgLd	!= NULL) checkCudaErrors( hipFreeArray( dev_ImgLd) );
	if(dev_ProjLr	!= NULL) checkCudaErrors( hipFreeArray( dev_ProjLr) );
	if(dev_NbImgOk	!= NULL) checkCudaErrors( hipFree( dev_NbImgOk));
	if(dev_SimpCor	!= NULL) checkCudaErrors( hipFree( dev_SimpCor));
	if(dev_Cache	!= NULL) checkCudaErrors( hipFree( dev_Cache));
	if(dev_Cost		!= NULL) checkCudaErrors( hipFree( dev_Cost));


	dev_NbImgOk	= NULL;
	dev_SimpCor = NULL;
	dev_Cache	= NULL;
	dev_ImgLd	= NULL;
	dev_Cost	= NULL;

	//free(host_Corr_Out);
	//free(host_NbImgOk); 
	//free(host_Cache);
}

extern "C" void  FreeLayers()
{
	checkCudaErrors( hipFreeArray(dev_ImgLd));

};

extern "C" void  projToDevice(float* aProj,  int sXImg, int sYImg)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(&dev_ArrayProjImg,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(dev_ArrayProjImg,0,0,aProj, sYImg*sizeof(float2),sYImg*sizeof(float2), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Project,dev_ArrayProjImg) );

}

extern "C" void cubeProjToDevice(float* cubeProjPIm, hipExtent dimCube)
{

	// Format des canaux 
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
			
	// Taille du cube
	hipExtent sizeCube = dimCube;
			
	// Allocation memoire GPU du cube de projection
	checkCudaErrors( hipMalloc3DArray(&dev_CubeProjImg,&channelDesc,sizeCube) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };
			
	p.dstArray	= dev_CubeProjImg;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(cubeProjPIm, dimCube.width * 2 * sizeof(float), dimCube.width, dimCube.height);
	p.extent	= dimCube;					// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie du cube de projection du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );
		
}

extern "C" void  imageToDevice(float** aDataIm,  int sXImg, int sYImg)
{
	float *dataImg1D	= new float[sXImg*sYImg];
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	// TACHE : changer la structuration des donnees pour le stockage des images 
	// Tableau 2D  --->> tableau lin�aire
	for (int i = 0; i < sXImg ; i++)
		for (int j = 0; j < sYImg ; j++)
			dataImg1D[i*sYImg+j] = aDataIm[j][i];

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(&dev_Img,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(dev_Img,0,0,dataImg1D, sYImg*sizeof(float),sYImg*sizeof(float), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Image,dev_Img) );

	delete dataImg1D;

}
