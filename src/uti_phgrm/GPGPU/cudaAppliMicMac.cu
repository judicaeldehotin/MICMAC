#include "hip/hip_runtime.h"
#include "GpGpu/cudaAppliMicMac.cuh"
#include "GpGpu/cudaTextureTools.cuh"

// ATTENTION : erreur de compilation avec l'option hipReadModeNormalizedFloat et l'utilisation de la fonction tex2DLayered
texture< pixel,	hipTextureType2D >	TexMaskTer;
texture< float2,hipTextureType2DLayered >	TexL_Proj;
texture< float,	hipTextureType2DLayered >	TexL_Images;

ImageCuda<pixel>			mask;
ImageLayeredCuda<float>		LayeredImages;
ImageLayeredCuda<float2>	LayeredProjection;

//float*	host_Cache;
float*	dev_Cost;
float*	dev_Cache;
float*	dev_NbImgOk;
static __constant__ paramGPU cH;
paramGPU h;

//------------------------------------------------------------------------------------------
extern "C" void SetMask(pixel* dataMask, uint2 dimMask)
{
	mask.InitImage(dimMask,dataMask);

	hipBindTextureToArray(TexMaskTer,mask.GetCudaArray());
}

extern "C" void allocMemory(void)
{
	if (dev_NbImgOk	!= NULL) checkCudaErrors( hipFree(dev_NbImgOk));
	if (dev_Cache	!= NULL) checkCudaErrors( hipFree(dev_Cache));
	if (dev_Cost	!= NULL) checkCudaErrors( hipFree(dev_Cost));

	int costMemSize = h.rSiTer	* sizeof(float) * h.ZInter;
	int nBI_MemSize = h.rSiTer	* sizeof(float) * h.ZInter;
	int cac_MemSize = h.sizeCach* sizeof(float)* h.nbImages * h.ZInter;
	
	// Allocation m�moire
	checkCudaErrors( hipMalloc((void **) &dev_Cache	, cac_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_NbImgOk	, nBI_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_Cost		, costMemSize ) );

	// Texture des projections
	TexL_Proj.addressMode[0]	= hipAddressModeClamp;
	TexL_Proj.addressMode[1]	= hipAddressModeClamp;	
	TexL_Proj.filterMode		= hipFilterModeLinear; //hipFilterModePoint hipFilterModeLinear
	TexL_Proj.normalized		= true;

}

extern "C" paramGPU updateSizeBlock( uint2 ter0, uint2 ter1, uint Zinter )
{

	uint oldSizeTer = h.sizeTer;

	h.ZInter	= Zinter;
	h.ptMask0	= make_int2(ter0);
	h.ptMask1	= make_int2(ter1);
	h.pUTer0.x	= (int)ter0.x - (int)h.rVig.x;
	h.pUTer0.y	= (int)ter0.y - (int)h.rVig.y;
	h.pUTer1.x	= (int)ter1.x + (int)h.rVig.x;
	h.pUTer1.y	= (int)ter1.y + (int)h.rVig.y;
	h.rDiTer	= make_uint2(ter1.x - ter0.x, ter1.y - ter0.y);
	h.dimTer	= make_uint2(h.pUTer1.x - h.pUTer0.x, h.pUTer1.y - h.pUTer0.y);
	h.dimSTer	= iDivUp(h.dimTer,h.sampTer);	// Dimension du bloque terrain sous echantillon�
	h.sizeTer	= size(h.dimTer);				// Taille du bloque terrain
	h.sizeSTer  = size(h.dimSTer);				// Taille du bloque terrain sous echantillon�
	h.rSiTer	= size(h.rDiTer);
	h.dimCach	= h.rDiTer * h.dimVig;
	h.sizeCach	= size(h.dimCach);
	h.restTer	= h.dimSTer * h.sampTer - h.dimTer;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cH), &h, sizeof(paramGPU)));

	if (oldSizeTer < h.sizeTer)
		allocMemory();

	return h;
}

static void correlOptionsGPU( uint2 ter0, uint2 ter1, uint2 dV,uint2 dRV, uint2 dI, float mAhEpsilon, uint samplingZ, int uvINTDef, uint nLayer, uint interZ )
{

	float uvDef;
	memset(&uvDef,uvINTDef,sizeof(float));

	h.nbImages	= nLayer;
	h.dimVig	= dV;							// Dimension de la vignette
	h.dimImg	= dI;							// Dimension des images
	h.rVig		= dRV;							// Rayon de la vignette
	h.sizeVig	= size(dV);						// Taille de la vignette en pixel 
	h.sampTer	= samplingZ;					// Pas echantillonage du terrain
	h.DefaultVal= uvDef;						// UV Terrain incorrect
	h.IntDefault	= uvINTDef;
	h.badVig	= -4.0f;
	h.mAhEpsilon= mAhEpsilon;

	updateSizeBlock( ter0, ter1, interZ );
}

extern "C" void imagesToLayers(float *fdataImg1D, uint2 dimImage, int nbLayer)
{

	LayeredImages.SetDimension(dimImage,nbLayer);
	LayeredImages.AllocMemory();
	LayeredImages.copyHostToDevice(fdataImg1D);

	// Li� � la texture
	TexL_Images.addressMode[0]	= hipAddressModeWrap;
    TexL_Images.addressMode[1]	= hipAddressModeWrap;
    TexL_Images.filterMode		= hipFilterModePoint; //hipFilterModeLinear hipFilterModePoint
    TexL_Images.normalized		= true;
	
	checkCudaErrors( hipBindTextureToArray(TexL_Images,LayeredImages.GetCudaArray()) );

};

extern "C" void  allocMemoryTabProj(uint2 dimTer, int nbLayer)
{
	LayeredProjection.DeallocMemory();
	LayeredProjection.SetDimension(dimTer,nbLayer);
	LayeredProjection.AllocMemory();
}

extern "C" void  CopyProjToLayers(float2 *h_TabProj)
{
	LayeredProjection.copyHostToDevice(h_TabProj);
};

__global__ void correlationKernel( float *dev_NbImgOk, float* cachVig, uint2 nbActThrd )
{
	__shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];

	// Coordonn�es du terrain global avec bordure // __umul24!!!! A voir
	const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);
	
	// Si le processus est hors du terrain, nous sortons du kernel
	if (oSE(ptHTer,cH.dimTer)) return;

#if (SAMPLETERR == 1)
	const float2 ptProj = tex2DLayeredPt(TexL_Proj,ptHTer,cH.dimSTer,blockIdx.z);
#else
	const float2 ptProj = tex2DLayeredPt(TexL_Proj,ptHTer,cH.dimSTer,cH.sampTer,blockIdx.z);
#endif
	
	if (oI(ptProj,0))
	{
		cacheImg[threadIdx.y][threadIdx.x]  = cH.badVig;
		return;
	}
 	else
		//cacheImg[threadIdx.y][threadIdx.x] = tex2DFastBicubic<float,float>(TexL_Images, ptProj.x, ptProj.y, cH.dimImg,(int)(blockIdx.z % cH.nbImages));
		//cacheImg[threadIdx.y][threadIdx.x] = tex2DLayeredPt( TexL_Images, ptProj, cH.dimImg, (int)(blockIdx.z % cH.nbImages));
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( TexL_Images, (((int)ptProj.x )+ 0.5f) / (float)cH.dimImg.x, (((int)(ptProj.y) )+ 0.5f) / (float)cH.dimImg.y,(int)(blockIdx.z % cH.nbImages));
 
	__syncthreads();

	const int2 ptTer = make_int2(ptHTer) - make_int2(cH.rVig);
	// Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel
	if (oSE(threadIdx, nbActThrd + cH.rVig) || oI(threadIdx , cH.rVig) || oSE( ptTer, cH.rDiTer) || oI(ptTer, 0))
		return;

	if(tex2D(TexMaskTer, ptTer.x, ptTer.y) == 0) return;

	const short2 c0	= make_short2(threadIdx) - cH.rVig;
	const short2 c1	= make_short2(threadIdx) + cH.rVig;
	 
	// Intialisation des valeurs de calcul 
	float aSV = 0.0f, aSVV	= 0.0f;
	short2 pt;
	
	#pragma unroll // ATTENTION PRAGMA FAIT AUGMENTER LA quantit� MEMOIRE des registres!!!
	for (pt.y = c0.y ; pt.y <= c1.y; pt.y++)
		#pragma unroll
		for (pt.x = c0.x ; pt.x <= c1.x; pt.x++)
		{	
			const float val = cacheImg[pt.y][pt.x];	// Valeur de l'image

			if (val ==  cH.badVig) return;

			aSV  += val;		// Somme des valeurs de l'image cte 
			aSVV += (val*val);	// Somme des carr�s des vals image cte
		}
	
	aSV	 /=	cH.sizeVig;
	aSVV /=	cH.sizeVig;
	aSVV -=	(aSV * aSV);
	
	if ( aSVV <= cH.mAhEpsilon) return;

	aSVV =	sqrt(aSVV);

	const uint pitchCache = blockIdx.z * cH.sizeCach + ptTer.x * cH.dimVig.x;
	const uint pitchCachY = ptTer.y * cH.dimVig.y ;
	#pragma unroll
	for ( pt.y = c0.y ; pt.y <= c1.y; pt.y++)
	{
		const int _py	= (pitchCachY + (pt.y - c0.y))* cH.dimCach.x;
		#pragma unroll
		for ( pt.x = c0.x ; pt.x <= c1.x; pt.x++)					
			cachVig[ pitchCache + _py  + (pt.x - c0.x)] = (cacheImg[pt.y][pt.x] -aSV)/aSVV;
	}	

	const int ZPitch = (blockIdx.z / cH.nbImages) * cH.rSiTer;

	atomicAdd( &dev_NbImgOk[ZPitch + to1D(ptTer,cH.rDiTer)], 1.0f);
};

// Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens	///
__global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, float * dev_NbImgOk, uint2 nbActThr)
{
	__shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];		// Somme des valeurs
	__shared__ float aSVV[ SBLOCKDIM ][ SBLOCKDIM ];		// Somme des carr�s des valeurs
	__shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];	// resultat
	__shared__ ushort nbIm[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];	// nombre d'images correcte

	// coordonn�es des threads
	const uint2 t = make_uint2(threadIdx);

	if ( threadIdx.z == 0)
	{
		aSV [t.y][t.x]		= 0.0f;
		aSVV[t.y][t.x]		= 0.0f;
		resu[t.y/2][t.x/2]	= 0.0f;
		nbIm[t.y/2][t.x/2]	= 0;
	}
	
	__syncthreads();

 	if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

	// Coordonn�es 2D du cache vignette
	const uint2 ptCach = make_uint2(blockIdx) * nbActThr  + t;
	
	// Si le thread est en dehors du cache
	if ( oSE(ptCach, cH.dimCach))	return;
	
	const uint2	ptTer	= ptCach / cH.dimVig;						// Coordonn�es 2D du terrain

	if(tex2D(TexMaskTer, ptTer.x, ptTer.y) == 0) return;

	const uint	iTer	= blockIdx.z * cH.rSiTer + to1D(ptTer, cH.rDiTer);	// Coordonn�es 1D dans le terrain
	const bool	mThrd	= t.x % cH.dimVig.x == 0 &&  t.y % cH.dimVig.y == 0 && threadIdx.z == 0;
	const uint2 thTer	= t / cH.dimVig;									// Coordonn�es 2D du terrain dans le repere des threads
	
	if (mThrd)
		nbIm[thTer.y][thTer.x] = (ushort)dev_NbImgOk[iTer];

	__syncthreads();

	if (nbIm[thTer.y][thTer.x] < 2) return;
	
	const uint sizLayer = (blockIdx.z * cH.nbImages + threadIdx.z) * cH.sizeCach;	// Taille du cache vignette pour une image

	const uint2 cc		= ptTer * cH.dimVig;										// coordonn�es 2D 1er pixel de la vignette
	const int iCC		= sizLayer + to1D( cc, cH.dimCach );						// coordonn�es 1D 1er pixel de la vignette

	if (cacheVign[iCC] == cH.DefaultVal) return;									// sortir si la vignette incorrecte
	
	const uint iCach	= sizLayer + to1D( ptCach, cH.dimCach );					// Coordonn�es 1D du cache vignette
	const float val		= cacheVign[iCach]; 

	atomicAdd( &(aSV[t.y][t.x]), val);
	atomicAdd(&(aSVV[t.y][t.x]), val * val);
	__syncthreads();

	if ( threadIdx.z != 0) return;

	atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - ((aSV[t.y][t.x] * aSV[t.y][t.x])/ nbIm[thTer.y][thTer.x])); 

	if ( !mThrd ) return;
	__syncthreads();

	// Normalisation pour le ramener a un equivalent de 1-Correl 
	const float cost = resu[thTer.y][thTer.x]/ (( nbIm[thTer.y][thTer.x] -1.0f) * ((float)cH.sizeVig));

	dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));
}

extern "C" paramGPU Init_Correlation_GPU(  uint2 ter0, uint2 ter1, int nbLayer , uint2 dRVig , uint2 dimImg, float mAhEpsilon, uint samplingZ, int uvINTDef , uint interZ)
{
	dev_NbImgOk		= NULL;
	dev_Cache		= NULL;
	dev_Cost		= NULL;

	correlOptionsGPU( ter0, ter1, dRVig * 2 + 1,dRVig, dimImg,mAhEpsilon, samplingZ, uvINTDef,nbLayer, interZ);
	allocMemory();

	return h;
}

extern "C" void basic_Correlation_GPU( float* h_TabCost,  int nbLayer, uint interZ ){

	int nBI_MemSize = h.rSiTer	 * sizeof(float) * interZ;
	int cac_MemSize = h.sizeCach * sizeof(float) * nbLayer * interZ;
	int costMemSize = h.rSiTer	 * sizeof(float) * interZ;

	//----------------------------------------------------------------------------
	 
	checkCudaErrors( hipMemset( dev_Cost,	h.IntDefault, costMemSize ));
	checkCudaErrors( hipMemset( dev_Cache,	h.IntDefault, cac_MemSize ));
	checkCudaErrors( hipMemset( dev_NbImgOk,0, nBI_MemSize ));
	checkCudaErrors( hipBindTextureToArray(TexL_Proj,LayeredProjection.GetCudaArray()) );

	//----------------------------------------------------------------------------
	//				calcul de dimension du kernel de correlation

	dim3	threads( BLOCKDIM, BLOCKDIM, 1);
	uint2	thd2D		= make_uint2(threads);
	uint2	actiThsCo	= thd2D - 2 * h.dimVig;
	uint2	block2D		= iDivUp(h.dimTer,actiThsCo);
	dim3	blocks(block2D.x , block2D.y, nbLayer * interZ);

	//----------------------------------------------------------------------------
	//				calcul de dimension du kernel de multi-correlation

	uint2	actiThs		= SBLOCKDIM - make_uint2( SBLOCKDIM % h.dimVig.x, SBLOCKDIM % h.dimVig.y);
	dim3	threads_mC(SBLOCKDIM, SBLOCKDIM, nbLayer);
	uint2	block2D_mC	= iDivUp(h.dimCach,actiThs);
	dim3	blocks_mC(block2D_mC.x,block2D_mC.y,interZ);

	//-----------------------  KERNEL  Correlation  -------------------------------
	
	correlationKernel<<<blocks, threads>>>( dev_NbImgOk, dev_Cache, actiThsCo);
	getLastCudaError("Basic Correlation kernel failed");
	
	//-------------------  KERNEL  Multi Correlation  ------------------------------

    multiCorrelationKernel<<<blocks_mC, threads_mC>>>( dev_Cost, dev_Cache, dev_NbImgOk, actiThs);
    getLastCudaError("Multi-Correlation kernel failed");

	//----------------------------------------------------------------------------

	checkCudaErrors( hipUnbindTexture(TexL_Proj) );
	checkCudaErrors( hipMemcpy( h_TabCost, dev_Cost, costMemSize, hipMemcpyDeviceToHost) );
	
	//----------------------------------------------------------------------------
	//checkCudaErrors( hipMemcpy( h_TabCost, dev_NbImgOk, costMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_Cache, dev_Cache,	  cac_MemSize, hipMemcpyDeviceToHost) );
	//GpGpuTools::OutputArray(h_TabCost,h.rDiTer,11.0f,h.DefaultVal);
	//----------------------------------------------------------------------------

}

extern "C" void freeGpuMemory()
{
	checkCudaErrors( hipUnbindTexture(TexL_Images) );	
	checkCudaErrors( hipUnbindTexture(TexMaskTer) );	

	if(dev_NbImgOk	!= NULL) checkCudaErrors( hipFree( dev_NbImgOk));
	if(dev_Cache	!= NULL) checkCudaErrors( hipFree( dev_Cache));
	if(dev_Cost		!= NULL) checkCudaErrors( hipFree( dev_Cost));

	dev_NbImgOk	= NULL;
	dev_Cache	= NULL;
	dev_Cost	= NULL;

	mask.DeallocMemory();
	LayeredImages.DeallocMemory();
	LayeredProjection.DeallocMemory();
}
