#include "hip/hip_runtime.h"
#include "GpGpu/cudaAppliMicMac.cuh"
#include "GpGpu/helper_math_extented.cuh"


#include <iostream>
#include <string>
using namespace std;

#ifdef _WIN32
  #include <windows.h>
  #include <Lmcons.h>
#endif

#ifdef _DEBUG
	#define   BLOCKDIM	16
	#define   SBLOCKDIM 10
#else
	#define   BLOCKDIM	32
	#define   SBLOCKDIM 16
#endif

/* Non utilis�
texture<float, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Image;
texture<bool, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Cache;
texture<float2, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Project;
hipArray* dev_Img;				// Tableau des valeurs de l'image
hipArray* dev_CubeProjImg;		// Declaration du cube de projection pour le device
hipArray* dev_ArrayProjImg;	// Declaration du tableau de projection pour le device
*/

//------------------------------------------------------------------------------------------
// ATTENTION : erreur de compilation avec l'option hipReadModeNormalizedFloat
// et l'utilisation de la fonction tex2DLayered
texture< bool,	hipTextureType2D >			TexMaskTer;
texture< float2,hipTextureType2DLayered >	TexLay_Proj;
texture< float,	hipTextureType2DLayered >	refTex_ImagesLayered;
hipArray* dev_ImgLd;		//
hipArray* dev_ProjLr;		//
hipArray* dev_MaskTer;		//

//------------------------------------------------------------------------------------------
//float*	host_SimpCor;
//float*	dev_SimpCor;
float*	host_Cache;
float*	dev_Cost;
float*	dev_Cache;
float*	dev_NbImgOk;

paramGPU h;
static __constant__ paramGPU cH;

extern "C" void allocMemory(void)
{
	//if (dev_SimpCor != NULL) checkCudaErrors( hipFree(dev_SimpCor));
	//int sCorMemSize = h.sizeTer * sizeof(float);
	//host_SimpCor	= (float*)	malloc(sCorMemSize);
	//checkCudaErrors( hipMalloc((void **) &dev_SimpCor	, sCorMemSize) );
	//host_Cache		= (float*)	malloc(cac_MemSize);

	if (dev_NbImgOk	!= NULL) checkCudaErrors( hipFree(dev_NbImgOk));
	if (dev_Cache	!= NULL) checkCudaErrors( hipFree(dev_Cache));
	if (dev_Cost	!= NULL) checkCudaErrors( hipFree(dev_Cost));

	int costMemSize = h.rSiTer	* sizeof(float);
	int nBI_MemSize = h.rSiTer	* sizeof(float);
	int cac_MemSize = h.sizeCach* sizeof(float)* h.nLayer;
	
	// Allocation m�moire
	checkCudaErrors( hipMalloc((void **) &dev_Cache	, cac_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_NbImgOk	, nBI_MemSize ) );
	checkCudaErrors( hipMalloc((void **) &dev_Cost		, costMemSize ) );

	// Texture des projections
	TexLay_Proj.addressMode[0]	= hipAddressModeClamp;
	TexLay_Proj.addressMode[1]	= hipAddressModeClamp;	
	TexLay_Proj.filterMode		= hipFilterModePoint; //hipFilterModePoint hipFilterModeLinear
	TexLay_Proj.normalized		= true;

}

extern "C" paramGPU updateSizeBlock( uint2 ter0, uint2 ter1 )
{

	uint oldSizeTer = h.sizeTer;

	h.pUTer0.x	= (int)ter0.x - (int)h.rVig.x;
	h.pUTer0.y	= (int)ter0.y - (int)h.rVig.y;
	h.pUTer1.x	= (int)ter1.x + (int)h.rVig.x;
	h.pUTer1.y	= (int)ter1.y + (int)h.rVig.y;
	h.rDiTer	= make_uint2(ter1.x - ter0.x, ter1.y - ter0.y);
	h.dimTer	= make_uint2(h.pUTer1.x - h.pUTer0.x, h.pUTer1.y - h.pUTer0.y);
	h.dimSTer	= iDivUp(h.dimTer,h.sampTer);	// Dimension du bloque terrain sous echantillon�
	h.sizeTer	= size(h.dimTer);				// Taille du bloque terrain
	h.sizeSTer  = size(h.dimSTer);				// Taille du bloque terrain sous echantillon�
	h.rSiTer	= size(h.rDiTer);
	h.dimCach	= h.rDiTer * h.dimVig;
	h.sizeCach	= size(h.dimCach);
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cH), &h, sizeof(paramGPU)));

	if (oldSizeTer < h.sizeTer)
		allocMemory();

	return h;
}

static void correlOptionsGPU( uint2 ter0, uint2 ter1, uint2 dV,uint2 dRV, uint2 dI, float mAhEpsilon, uint samplingZ, int uvINTDef, uint nLayer )
{

	float uvDef;
	memset(&uvDef,uvINTDef,sizeof(float));

	h.nLayer	= nLayer;
	h.dimVig	= dV;							// Dimension de la vignette
	h.dimImg	= dI;							// Dimension des images
	h.rVig		= dRV;							// Rayon de la vignette
	h.sizeVig	= size(dV);						// Taille de la vignette en pixel 
	h.sampTer	= samplingZ;					// Pas echantillonage du terrain
	h.UVDefValue= uvDef;						// UV Terrain incorrect
	h.UVIntDef	= uvINTDef;
	h.badVig	= -4.0f;
	h.mAhEpsilon= mAhEpsilon;

	updateSizeBlock( ter0, ter1 );
}

extern "C" void imagesToLayers(float *fdataImg1D, uint2 dimImage, int nbLayer)
{

	hipExtent sizeImgsLay = make_hipExtent( dimImage.x, dimImage.y, nbLayer );

	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// Allocation memoire GPU du tableau des calques d'images
	checkCudaErrors( hipMalloc3DArray(&dev_ImgLd,&channelDesc,sizeImgsLay,hipArrayLayered) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms	p	= { 0 };
	hipPitchedPtr		pit = make_hipPitchedPtr(fdataImg1D, sizeImgsLay.width * sizeof(float), sizeImgsLay.width, sizeImgsLay.height);

	p.dstArray	= dev_ImgLd;				// Pointeur du tableau de destination
	p.srcPtr	= pit;						// Pitch
	p.extent	= sizeImgsLay;				// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des images du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

	// Li� � la texture
	refTex_ImagesLayered.addressMode[0]	= hipAddressModeWrap;
    refTex_ImagesLayered.addressMode[1]	= hipAddressModeWrap;
    refTex_ImagesLayered.filterMode		= hipFilterModePoint; //hipFilterModeLinear hipFilterModePoint
    refTex_ImagesLayered.normalized		= true;
	checkCudaErrors( hipBindTextureToArray(refTex_ImagesLayered,dev_ImgLd) );

};

extern "C" void  allocMemoryTabProj(uint2 dimTer, int nbLayer)
{

	// D�finition du format des canaux d'images
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Taille du tableau des calques 
	hipExtent siz_PL = make_hipExtent( dimTer.x, dimTer.y, nbLayer);

	// Allocation memoire GPU du tableau des calques de projections
	if (dev_ProjLr != NULL) hipFreeArray(dev_ProjLr);

	checkCudaErrors( hipMalloc3DArray(&dev_ProjLr,&channelDesc,siz_PL,hipArrayLayered ));

}

extern "C" void  CopyProjToLayers(float *h_TabProj, uint2 dimTer, int nbLayer)
{
	hipExtent siz_PL = make_hipExtent( dimTer.x, dimTer.y, nbLayer);

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };

	p.dstArray	= dev_ProjLr;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(h_TabProj, siz_PL.width * sizeof(float2), siz_PL.width, siz_PL.height);
	p.extent	= siz_PL;
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie des projections du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );

};

__device__  inline float2 simpleProjection( uint2 size, uint2 ssize/*, uint2 sizeImg*/ ,uint2 coord, int L)
{
	const float2 cf		= make_float2(ssize) * make_float2(coord) / make_float2(size) ;
	const int2	 a		= make_int2(cf);
	const float2 uva	= (make_float2(a) + 0.5f) / (make_float2(ssize));
	const float2 uvb	= (make_float2(a+1) + 0.5f) / (make_float2(ssize));
	float2 ra, rb, Iaa;

	ra	= tex2DLayered( TexLay_Proj, uva.x, uva.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uva.y, L);
	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(cH.badVig);

	Iaa	= ((float)(a.x + 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	ra	= tex2DLayered( TexLay_Proj, uva.x, uvb.y, L);
	rb	= tex2DLayered( TexLay_Proj, uvb.x, uvb.y, L);

	if (ra.x < 0.0f || ra.y < 0.0f || rb.x < 0.0f || rb.y < 0.0f)
		return make_float2(cH.badVig);

	ra	= ((float)(a.x+ 1.0f) - cf.x) * ra + (cf.x - (float)(a.x)) * rb;
	ra = ((float)(a.y+ 1.0f) - cf.y) * Iaa + (cf.y - (float)(a.y)) * ra;
	/*ra = (ra + 0.5f) / (make_float2(sizeImg));*/

	return ra;
}

__global__ void correlationKernel( float *dev_NbImgOk, float* cachVig/*, float *siCor*/, uint2 nbActThrd ) //__global__ void correlationKernel( int *dev_NbImgOk, float* cachVig)
{
	__shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];

	// Coordonn�es du terrain global avec bordure
	const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);
	
	// Si le processus est hors du terrain, nous sortons du kernel
	if (oSE(ptHTer,cH.dimTer)) return;

	//float2 PtTProj = tex2DLayered(TexLay_Proj, ((float)ghTer.x / (float)cDimTer.x * (float)cSDimTer.x + 0.5f) /(float)cSDimTer.x, ((float)ghTer.y/ (float)cDimTer.y * (float)cSDimTer.y + 0.5f) /(float)cSDimTer.y ,blockIdx.z) ;
	//const float2 PtTProj = simpleProjection( cDimTer, cSDimTer/*, cDimImg*/, ptHTer, blockIdx.z);
	const float2 PtTProj = tex2DLayered(TexLay_Proj, ((float)ptHTer.x  + 0.5f) /(float)cH.dimTer.x, ((float)ptHTer.y + 0.5f) /(float)cH.dimTer.y ,blockIdx.z) ;

	if (oEq(PtTProj, cH.UVDefValue))
	{
		cacheImg[threadIdx.y][threadIdx.x]  = cH.badVig;
		return;
	}
 	else
		// !!! ATTENTION Modification pour simplification du debug !!!!
		//cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( refTex_ImagesLayered, (PtTProj.x + 0.5f) / (float)cDimImg.x, (PtTProj.y + 0.5f) / (float)cDimImg.y,blockIdx.z);
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( refTex_ImagesLayered, (((int)PtTProj.x )+ 0.5f) / (float)cH.dimImg.x, (((int)(PtTProj.y) )+ 0.5f) / (float)cH.dimImg.y,blockIdx.z);

	__syncthreads();

	const int2 ptTer = make_int2(ptHTer) - make_int2(cH.rVig);

	// Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel
	if ( oSE(threadIdx, nbActThrd + cH.rVig) || oI(threadIdx , cH.rVig) || oSE( ptTer, cH.rDiTer) || oI(ptTer, 0))
		return;
	
	const short2 c0	= make_short2(threadIdx) - cH.rVig;
	const short2 c1	= make_short2(threadIdx) + cH.rVig;
	 
	// Intialisation des valeurs de calcul 
	float aSV = 0.0f, aSVV	= 0.0f;
	short2 pt;
	
	#pragma unroll // ATTENTION PRAGMA FAIT AUGMENTER LA quantit� MEMOIRE des registres!!!
	for (pt.y = c0.y ; pt.y <= c1.y; pt.y++)
		#pragma unroll
		for (pt.x = c0.x ; pt.x <= c1.x; pt.x++)
		{	
			const float val = cacheImg[pt.y][pt.x];	// Valeur de l'image

			if (val ==  cH.badVig) return;

			aSV  += val;		// Somme des valeurs de l'image cte 
			aSVV += (val*val);	// Somme des carr�s des vals image cte
		}
	
	aSV	 /=	cH.sizeVig;
	aSVV /=	cH.sizeVig;
	aSVV -=	(aSV * aSV);
	
	if ( aSVV <= cH.mAhEpsilon) return;

	aSVV =	sqrt(aSVV);

	#pragma unroll
	for ( pt.y = c0.y ; pt.y <= c1.y; pt.y++)
	{
		const int _cy	= ptTer.y * cH.dimVig.y + (pt.y - c0.y);
		#pragma unroll
		for ( pt.x = c0.x ; pt.x <= c1.x; pt.x++)					
			cachVig[(blockIdx.z * cH.sizeCach) + _cy * cH.dimCach.x + ptTer.x * cH.dimVig.x + (pt.x - c0.x)] = (cacheImg[pt.y][pt.x] -aSV)/aSVV;
	}	

	atomicAdd( &dev_NbImgOk[to1D(ptTer,cH.rDiTer)], 1.0f);
};

///////////////////////////////////////////////////////////////////////////////////
//																				///
// Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens	///
//																				///
///////////////////////////////////////////////////////////////////////////////////

__global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, float * dev_NbImgOk, uint2 nbActThr)
{
	__shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float aSVV[ SBLOCKDIM ][ SBLOCKDIM ];
	__shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];

	// coordonn�es des threads
	const uint2 t = make_uint2(threadIdx);

	if ( threadIdx.z == 0)
	{
		aSV [t.y][t.x]		= 0.0f;
		aSVV[t.y][t.x]		= 0.0f;
		resu[t.y/2][t.x/2]	= 0.0f;
	}
	
	__syncthreads();

 	if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

	// Coordonn�es 2D du cache vignette
	const uint2 ptCach = make_uint2(blockIdx) * nbActThr  + t;
	
	// Si le thread est en dehors du cache
	if ( oSE(ptCach, cH.dimCach))	return;
	
	const uint2	ptTer	= ptCach / cH.dimVig;					// Coordonn�es 2D du terrain 
	const uint	iTer	= to1D(ptTer, cH.rDiTer);				// Coordonn�es 1D dans le terrain
	const bool	mThrd	= t.x % cH.dimVig.x == 0 &&  t.y % cH.dimVig.y == 0 && threadIdx.z == 0;
	const float aNbImOk = dev_NbImgOk[iTer];					// Nombre vignettes correctes

	if (aNbImOk < 2) return;
	
	const uint sizLayer = threadIdx.z * cH.sizeCach;			// Taille du cache vignette pour une image
	const uint iCach	= sizLayer + to1D( ptCach, cH.dimCach );	// Coordonn�es 1D du cache vignette
	const uint2 cc		= ptTer * cH.dimVig;						// coordonn�es 2D 1er pixel de la vignette
	const int iCC		= sizLayer + to1D( cc, cH.dimCach );	// coordonn�es 1D 1er pixel de la vignette
	
	const float val = (cacheVign[iCC] != cH.UVDefValue) ? cacheVign[iCach] : 0.0f; // sortir si bad vignette

	atomicAdd( &(aSV[t.y][t.x]), val);

	atomicAdd(&(aSVV[t.y][t.x]), val * val);
	__syncthreads();

	if ( threadIdx.z != 0) return;

	const uint2 thTer = t / cH.dimVig;	// Coordonn�es 2D du terrain dans le repere des threads
	
	atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - ((aSV[t.y][t.x] * aSV[t.y][t.x])/ aNbImOk)); 

	if ( !mThrd ) return;
	__syncthreads();

	// Normalisation pour le ramener a un equivalent de 1-Correl 
	const float cost = resu[thTer.y][thTer.x]/ (( aNbImOk -1.0f) * ((float)cH.sizeVig));

	dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));
}

extern "C" paramGPU Init_Correlation_GPU(  uint2 ter0, uint2 ter1, int nbLayer , uint2 dRVig , uint2 dimImg, float mAhEpsilon, uint samplingZ, int uvINTDef )
{
	dev_NbImgOk		= NULL;
	dev_Cache		= NULL;
	dev_Cost		= NULL;
	dev_ProjLr		= NULL;

	correlOptionsGPU( ter0, ter1, dRVig * 2 + 1,dRVig, dimImg,mAhEpsilon, samplingZ, uvINTDef,nbLayer);
	allocMemory();

	return h;
}

extern "C" void basic_Correlation_GPU( float* h_TabCost,  int nbLayer ){

	//////////////////////////////////////////////////////////////////////////
	//int sCorMemSize = h.sizeTer  * sizeof(float);
	//checkCudaErrors( hipMemset( dev_SimpCor,	0, sCorMemSize ));
	////////////////////////////////////////////////////////////////////////// 
	
	int nBI_MemSize = h.rSiTer	 * sizeof(float);
	int cac_MemSize = h.sizeCach * sizeof(float) * nbLayer;
	int costMemSize = h.rSiTer	 * sizeof(float);

	//////////////////////////////////////////////////////////////////////////
	 
	checkCudaErrors( hipMemset( dev_Cost,	h.UVIntDef, costMemSize ));
	checkCudaErrors( hipMemset( dev_Cache,	h.UVIntDef, cac_MemSize ));
	checkCudaErrors( hipMemset( dev_NbImgOk,0, nBI_MemSize ));
	checkCudaErrors( hipBindTextureToArray(TexLay_Proj,dev_ProjLr) );

	//////////////////////////////////////////////////////////////////////////

	dim3 threads( BLOCKDIM, BLOCKDIM, 1);
	uint2 actiThsCo = make_uint2(threads.x - 2 *((int)(h.dimVig.x)), threads.y - 2 * ((int)(h.dimVig.y)));
	dim3 blocks(iDivUp((int)(h.dimTer.x),actiThsCo.x) , iDivUp((int)(h.dimTer.y), actiThsCo.y), nbLayer);
	
	uint2 actiThs = make_uint2(SBLOCKDIM - SBLOCKDIM % ((int)h.dimVig.x), SBLOCKDIM - SBLOCKDIM % ((int)h.dimVig.y));
	dim3 threads_mC(SBLOCKDIM, SBLOCKDIM, nbLayer);
	dim3 blocks_mC(iDivUp((int)(h.dimCach.x), actiThs.x) , iDivUp((int)(h.dimCach.y), actiThs.y));

	////////////////////--  KERNEL  Correlation  --//////////////////////////
	
	correlationKernel<<<blocks, threads>>>( dev_NbImgOk, dev_Cache, actiThsCo);
	getLastCudaError("Basic Correlation kernel failed");
	
	//////////////////--  KERNEL  Multi Correlation  --///////////////////////

   	multiCorrelationKernel<<<blocks_mC, threads_mC>>>( dev_Cost, dev_Cache, dev_NbImgOk, actiThs);
   	getLastCudaError("Multi-Correlation kernel failed");

	//////////////////////////////////////////////////////////////////////////

	checkCudaErrors( hipUnbindTexture(TexLay_Proj) );
	checkCudaErrors( hipMemcpy( h_TabCost, dev_Cost, costMemSize, hipMemcpyDeviceToHost) );
	
	//hipDeviceSynchronize();
	//checkCudaErrors( hipMemcpy( h_TabCost, dev_NbImgOk, costMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_SimpCor, dev_SimpCor, sCorMemSize, hipMemcpyDeviceToHost) );
	//checkCudaErrors( hipMemcpy( host_Cache, dev_Cache,	  cac_MemSize, hipMemcpyDeviceToHost) );

	//GpGpuTools::OutputArray(h_TabCost,h.rDiTer,100.0f,h.UVDefValue);

	//////////////////////////////////////////////////////////////////////////

}

extern "C" void freeGpuMemory()
{
	//checkCudaErrors( hipUnbindTexture(refTex_Image) );
	//checkCudaErrors( hipFreeArray(dev_Img) );
	//checkCudaErrors( hipFreeArray(dev_CubeProjImg) );
	//checkCudaErrors( hipFreeArray(dev_ArrayProjImg) );

	//if(dev_SimpCor	!= NULL) checkCudaErrors( hipFree( dev_SimpCor));
	
	checkCudaErrors( hipUnbindTexture(refTex_ImagesLayered) );	

	if(dev_ImgLd	!= NULL) checkCudaErrors( hipFreeArray( dev_ImgLd) );
	if(dev_ProjLr	!= NULL) checkCudaErrors( hipFreeArray( dev_ProjLr) );
	if(dev_NbImgOk	!= NULL) checkCudaErrors( hipFree( dev_NbImgOk));
	if(dev_Cache	!= NULL) checkCudaErrors( hipFree( dev_Cache));
	if(dev_Cost		!= NULL) checkCudaErrors( hipFree( dev_Cost));

	dev_NbImgOk	= NULL;
	dev_Cache	= NULL;
	dev_ImgLd	= NULL;
	dev_Cost	= NULL;

	// DEBUG
	//dev_SimpCor = NULL;
	//free(host_SimpCor); 
	//free(host_Cache);
}

extern "C" void  projToDevice(hipArray_t *dev_ArrayProjImg,texture<float2, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Project, float* aProj,  int sXImg, int sYImg)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(dev_ArrayProjImg,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(*dev_ArrayProjImg,0,0,aProj, sYImg*sizeof(float2),sYImg*sizeof(float2), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Project,*dev_ArrayProjImg) );

}

extern "C" void cubeProjToDevice(hipArray_t *dev_CubeProjImg,float* cubeProjPIm, hipExtent dimCube)
{

	// Format des canaux 
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
			
	// Taille du cube
	hipExtent sizeCube = dimCube;
			
	// Allocation memoire GPU du cube de projection
	checkCudaErrors( hipMalloc3DArray(dev_CubeProjImg,&channelDesc,sizeCube) );

	// D�claration des parametres de copie 3D
	hipMemcpy3DParms p = { 0 };
			
	p.dstArray	= *dev_CubeProjImg;			// Pointeur du tableau de destination
	p.srcPtr	= make_hipPitchedPtr(cubeProjPIm, dimCube.width * 2 * sizeof(float), dimCube.width, dimCube.height);
	p.extent	= dimCube;					// Taille du cube
	p.kind		= hipMemcpyHostToDevice;	// Type de copie

	// Copie du cube de projection du Host vers le Device
	checkCudaErrors( hipMemcpy3D(&p) );
		
}

extern "C" void  imageToDevice(hipArray_t *dev_Img, texture<float, hipTextureType2D, hipReadModeNormalizedFloat> refTex_Image, float** aDataIm,  int sXImg, int sYImg)
{
	float *dataImg1D	= new float[sXImg*sYImg];
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	// TACHE : changer la structuration des donnees pour le stockage des images 
	// Tableau 2D  --->> tableau lin�aire
	for (int i = 0; i < sXImg ; i++)
		for (int j = 0; j < sYImg ; j++)
			dataImg1D[i*sYImg+j] = aDataIm[j][i];

	// Allocation m�moire du tableau cuda
	checkCudaErrors( hipMallocArray(dev_Img,&channelDesc,sYImg,sXImg) );

	// Copie des donn�es du Host dans le tableau Cuda
	checkCudaErrors( hipMemcpy2DToArray(*dev_Img,0,0,dataImg1D, sYImg*sizeof(float),sYImg*sizeof(float), sXImg, hipMemcpyHostToDevice) );

	// Lier la texture au tableau Cuda
	checkCudaErrors( hipBindTextureToArray(refTex_Image,*dev_Img) );

	delete dataImg1D;

}
