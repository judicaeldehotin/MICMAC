#include "hip/hip_runtime.h"
#include "GpGpu/cudaAppliMicMac.cuh"
#include "GpGpu/cudaTextureTools.cuh"

static __constant__ paramMicMacGpGpu cH;

// ATTENTION : erreur de compilation avec l'option hipReadModeNormalizedFloat et l'utilisation de la fonction tex2DLayered
texture< pixel,	hipTextureType2D >			TexS_MaskTer;
texture< float,	hipTextureType2DLayered >	TexL_Images;
TexFloat2Layered							TexL_Proj_01;
TexFloat2Layered							TexL_Proj_02;

template<int TexSel> __device__ __host__ TexFloat2Layered TexFloat2L();

template<> __device__ __host__ TexFloat2Layered TexFloat2L<1>() { return TexL_Proj_01; };
template<> __device__ __host__ TexFloat2Layered TexFloat2L<2>() { return TexL_Proj_02; };

//------------------------------------------------------------------------------------------

extern "C" textureReference& getMask(){	return TexS_MaskTer;}
extern "C" textureReference& getImage(){ return TexL_Images;}
extern "C" textureReference& getProjection(){return TexL_Proj_01;}

extern "C" void CopyParamTodevice( paramMicMacGpGpu param )
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cH), &param, sizeof(paramMicMacGpGpu)));
}

template<int TexSel> __global__ void correlationKernel( float *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
{
	__shared__ float cacheImg[ BLOCKDIM ][ BLOCKDIM ];

	// Coordonn�es du terrain global avec bordure // __umul24!!!! A voir
	const uint2 ptHTer = make_uint2(blockIdx) * nbActThrd + make_uint2(threadIdx);
	
	// Si le processus est hors du terrain, nous sortons du kernel
	if (oSE(ptHTer,cH.dimTer)) return;

#if (SAMPLETERR == 1)
	const float2 ptProj = tex2DLayeredPt(TexFloat2L<TexSel>(),ptHTer,cH.dimSTer,blockIdx.z);
#else
	const float2 ptProj = tex2DLayeredPt(TexFloat2L<TexSel>(),ptHTer,cH.dimSTer,cH.sampTer,blockIdx.z);
#endif

	if (oI(ptProj,0))
	{
		cacheImg[threadIdx.y][threadIdx.x]  = cH.badVig;
		return;
	}
 	else
#if		INTERPOLA == NEAREST
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayered( TexL_Images, (((int)ptProj.x )+ 0.5f) / (float)cH.dimImg.x, (((int)(ptProj.y) )+ 0.5f) / (float)cH.dimImg.y,(int)(blockIdx.z % cH.nbImages));
#elif	INTERPOLA == LINEARINTER
		cacheImg[threadIdx.y][threadIdx.x] = tex2DLayeredPt( TexL_Images, ptProj, cH.dimImg, (int)(blockIdx.z % cH.nbImages));
#elif	INTERPOLA == BICUBIC
		cacheImg[threadIdx.y][threadIdx.x] = tex2DFastBicubic<float,float>(TexL_Images, ptProj.x, ptProj.y, cH.dimImg,(int)(blockIdx.z % cH.nbImages));
#endif
		
	__syncthreads();

	const int2 ptTer = make_int2(ptHTer) - make_int2(cH.rVig);
	// Nous traitons uniquement les points du terrain du bloque ou Si le processus est hors du terrain global, nous sortons du kernel
	if (oSE(threadIdx, nbActThrd + cH.rVig) || oI(threadIdx , cH.rVig) || oSE( ptTer, cH.rDiTer) || oI(ptTer, 0))
		return;

	if(tex2D(TexS_MaskTer, ptTer.x, ptTer.y) == 0) return;

	const short2 c0	= make_short2(threadIdx) - cH.rVig;
	const short2 c1	= make_short2(threadIdx) + cH.rVig;
	 
	// Intialisation des valeurs de calcul 
	float aSV = 0.0f, aSVV	= 0.0f;
	short2 pt;
	
	#pragma unroll // ATTENTION PRAGMA FAIT AUGMENTER LA quantit� MEMOIRE des registres!!!
	for (pt.y = c0.y ; pt.y <= c1.y; pt.y++)
		#pragma unroll
		for (pt.x = c0.x ; pt.x <= c1.x; pt.x++)
		{	
			const float val = cacheImg[pt.y][pt.x];	// Valeur de l'image

			if (val ==  cH.badVig) return;

			aSV  += val;		// Somme des valeurs de l'image cte 
			aSVV += (val*val);	// Somme des carr�s des vals image cte
		}

#ifdef FLOATMATH
		aSV	 = fdividef(aSV,(float)cH.sizeVig );
		aSVV = fdividef(aSVV,(float)cH.sizeVig );
		aSVV -=	(aSV * aSV);
#else
		aSV	 /=	cH.sizeVig;
		aSVV /=	cH.sizeVig;
		aSVV -=	(aSV * aSV);
#endif
	
	if ( aSVV <= cH.mAhEpsilon) return;

	aSVV =	rsqrtf(aSVV); // racine carre inverse

	const uint pitchCache = blockIdx.z * cH.sizeCach + ptTer.x * cH.dimVig.x;
	const uint pitchCachY = ptTer.y * cH.dimVig.y ;
	#pragma unroll
	for ( pt.y = c0.y ; pt.y <= c1.y; pt.y++)
	{
		const int _py	= (pitchCachY + (pt.y - c0.y))* cH.dimCach.x;
		#pragma unroll
		for ( pt.x = c0.x ; pt.x <= c1.x; pt.x++)		
			cachVig[ pitchCache + _py  + (pt.x - c0.x)] = (cacheImg[pt.y][pt.x] -aSV)*aSVV;

	}	

	const int ZPitch = (blockIdx.z / cH.nbImages) * cH.rSiTer;

	atomicAdd( &dev_NbImgOk[ZPitch + to1D(ptTer,cH.rDiTer)], 1.0f);
};

extern "C" void	 KernelCorrelation(dim3 blocks, dim3 threads, float *dev_NbImgOk, float* cachVig, uint2 nbActThrd)
{
	correlationKernel<1><<<blocks, threads>>>( dev_NbImgOk, cachVig, nbActThrd);
	getLastCudaError("Basic Correlation kernel failed");
}

// Calcul "rapide"  de la multi-correlation en utilisant la formule de Huygens	///
__global__ void multiCorrelationKernel(float *dTCost, float* cacheVign, float * dev_NbImgOk, uint2 nbActThr)
{
	__shared__ float aSV [ SBLOCKDIM ][ SBLOCKDIM ];		// Somme des valeurs
	__shared__ float aSVV[ SBLOCKDIM ][ SBLOCKDIM ];		// Somme des carr�s des valeurs
	__shared__ float resu[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];	// resultat
	__shared__ ushort nbIm[ SBLOCKDIM/2 ][ SBLOCKDIM/2 ];	// nombre d'images correcte

	// coordonn�es des threads
	const uint2 t = make_uint2(threadIdx);

	if ( threadIdx.z == 0)
	{
		aSV [t.y][t.x]		= 0.0f;
		aSVV[t.y][t.x]		= 0.0f;
		resu[t.y/2][t.x/2]	= 0.0f;
		nbIm[t.y/2][t.x/2]	= 0;
	}
	
	__syncthreads();

 	if ( oSE( t, nbActThr))	return; // si le thread est inactif, il sort

	// Coordonn�es 2D du cache vignette
	const uint2 ptCach = make_uint2(blockIdx) * nbActThr  + t;
	
	// Si le thread est en dehors du cache
	if ( oSE(ptCach, cH.dimCach))	return;
	
	const uint2	ptTer	= ptCach / cH.dimVig;						// Coordonn�es 2D du terrain

	if(tex2D(TexS_MaskTer, ptTer.x, ptTer.y) == 0) return;

	const uint	iTer	= blockIdx.z * cH.rSiTer + to1D(ptTer, cH.rDiTer);	// Coordonn�es 1D dans le terrain
	const bool	mThrd	= t.x % cH.dimVig.x == 0 &&  t.y % cH.dimVig.y == 0 && threadIdx.z == 0;
	const uint2 thTer	= t / cH.dimVig;									// Coordonn�es 2D du terrain dans le repere des threads
	
	if (mThrd)
		nbIm[thTer.y][thTer.x] = (ushort)dev_NbImgOk[iTer];

	__syncthreads();

	if (nbIm[thTer.y][thTer.x] < 2) return;
	
	const uint sizLayer = (blockIdx.z * cH.nbImages + threadIdx.z) * cH.sizeCach;	// Taille du cache vignette pour une image

	const uint2 cc		= ptTer * cH.dimVig;										// coordonn�es 2D 1er pixel de la vignette
	const int iCC		= sizLayer + to1D( cc, cH.dimCach );						// coordonn�es 1D 1er pixel de la vignette

	if (cacheVign[iCC] == cH.DefaultVal) return;									// sortir si la vignette incorrecte
	
	const uint iCach	= sizLayer + to1D( ptCach, cH.dimCach );					// Coordonn�es 1D du cache vignette
	const float val		= cacheVign[iCach]; 

	atomicAdd( &(aSV[t.y][t.x]), val);
	atomicAdd(&(aSVV[t.y][t.x]), val * val);
	__syncthreads();

	if ( threadIdx.z != 0) return;

#ifdef FLOATMATH
	atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - fdividef(aSV[t.y][t.x] * aSV[t.y][t.x],(float)nbIm[thTer.y][thTer.x])); 
#else
	atomicAdd(&(resu[thTer.y][thTer.x]),aSVV[t.y][t.x] - ((aSV[t.y][t.x] * aSV[t.y][t.x])/ nbIm[thTer.y][thTer.x])); 
#endif
	
	if ( !mThrd ) return;
	__syncthreads();

	// Normalisation pour le ramener a un equivalent de 1-Correl 
#ifdef FLOATMATH
	const float cost = fdividef( resu[thTer.y][thTer.x], (float)( nbIm[thTer.y][thTer.x] -1.0f) * (cH.sizeVig));
#else
	const float cost = resu[thTer.y][thTer.x]/ (( nbIm[thTer.y][thTer.x] -1.0f) * ((float)cH.sizeVig));
#endif

	dTCost[iTer] = 1.0f - max (-1.0, min(1.0f,1.0f - cost));
}

extern "C" void KernelmultiCorrelation(dim3 blocks, dim3 threads, float *dTCost, float* cacheVign, float * dev_NbImgOk, uint2 nbActThr)
{
	multiCorrelationKernel<<<blocks, threads>>>(dTCost, cacheVign, dev_NbImgOk, nbActThr);
	getLastCudaError("Multi-Correlation kernel failed");

}